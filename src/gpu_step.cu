#include "hip/hip_runtime.h"
#define _GNU_SOURCE
#include <iostream>
#include <fstream>
#include <vector>
#include <sstream>
#include <stdexcept>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <chrono>
#include <cstring> // For strerror
#include <sys/time.h>
#include <thread>
#include <omp.h>
#include <unistd.h>
#include "kseq++/kseq++.hpp"



#include "index.hpp"
#include "indexparameters.hpp"
#include "cmdline.hpp"
#include "exceptions.hpp"
#include "io.hpp"
#include "randstrobes.hpp"
#include "refs.hpp"
#include "logger.hpp"
#include "pc.hpp"
#include "readlen.hpp"
#include "my_struct.hpp"
#include "hash.hpp"
#include "timer.hpp"

//#define assert(x) ((void)0)


#define my_bucket_index_t StrobemerIndex::bucket_index_t

#define rescue_threshold 100


inline double GetTime() {
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return (double) tv.tv_sec + (double) tv.tv_usec / 1000000;
}

__host__ __device__ char get_base(const char* h_seq, const int* h_pre_sum, int i, int j) {
    static const char code2nt[4] = {'A', 'C', 'G', 'T'};

    int compressed_index = h_pre_sum[i] + (j / 4);
    int pos_in_byte = j % 4;

    uint8_t packed = h_seq[compressed_index];
    int base_code = (packed >> (pos_in_byte * 2)) & 0x3;
    //int base_code = (packed >> (6 - pos_in_byte * 2)) & 0x3;

    return code2nt[base_code];
}

__host__ __device__ char get_base(const char* h_seq, int i) {
    static const char code2nt[4] = {'A', 'C', 'G', 'T'};

    int compressed_index = i / 4;
    int pos_in_byte = i % 4;

    uint8_t packed = h_seq[compressed_index];
    int base_code = (packed >> (pos_in_byte * 2)) & 0x3;
    //int base_code = (packed >> (6 - pos_in_byte * 2)) & 0x3;
    //printf("packed %u - %d %d %c\n", packed, pos_in_byte, base_code, code2nt[base_code]);

    return code2nt[base_code];
}

__host__ __device__ char get_base_code(const char* h_seq, const int* h_pre_sum, int i, int j) {
    int compressed_index = h_pre_sum[i] + (j / 4);
    int pos_in_byte = j % 4;

    uint8_t packed = h_seq[compressed_index];
    int base_code = (packed >> (pos_in_byte * 2)) & 0x3;
    //int base_code = (packed >> (6 - pos_in_byte * 2)) & 0x3;

    return base_code;
}

__host__ __device__ char get_base_code(const char* h_seq,int i) {
    int compressed_index = i / 4;
    int pos_in_byte = i % 4;

    uint8_t packed = h_seq[compressed_index];
    int base_code = (packed >> (pos_in_byte * 2)) & 0x3;
    //int base_code = (packed >> (6 - pos_in_byte * 2)) & 0x3;

    return base_code;
}

__host__ __device__ void bit2char(const char* bit_seq, char* char_seq, int len) {
    for (int i = 0; i < len; i++) char_seq[i] = get_base(bit_seq, i);
}


__device__ static inline syncmer_hash_t gpu_syncmer_kmer_hash(uint64_t packed) {
    return xxh64(packed);
}

__device__ static inline syncmer_hash_t gpu_syncmer_smer_hash(uint64_t packed) {
    return xxh64(packed);
}

__device__  static inline randstrobe_hash_t gpu_randstrobe_hash(syncmer_hash_t hash1, syncmer_hash_t hash2) {
    return hash1 + hash2;
}


__device__ inline randstrobe_hash_t gpu_get_hash(const RefRandstrobe *d_randstrobes, size_t d_randstrobes_size, my_bucket_index_t position) {
    if (position < d_randstrobes_size) {
        return d_randstrobes[position].hash;
    } else {
        return static_cast<randstrobe_hash_t>(-1);
    }
}

__device__ inline bool gpu_is_filtered(const RefRandstrobe *d_randstrobes, size_t d_randstrobes_size, my_bucket_index_t position,
                                       unsigned int filter_cutoff) {
    return gpu_get_hash(d_randstrobes, d_randstrobes_size, position) ==
           gpu_get_hash(d_randstrobes, d_randstrobes_size, position + filter_cutoff);
}

__device__ int gpu_get_count(
    const RefRandstrobe *d_randstrobes,
    const my_bucket_index_t *d_randstrobe_start_indices,
    my_bucket_index_t position,
    int bits
) {
    const auto key = d_randstrobes[position].hash;
    const unsigned int top_N = key >> (64 - bits);
    int64_t position_end = d_randstrobe_start_indices[top_N + 1];
    int64_t position_start = position;

    if(position_end == 0) return 0;
    int64_t low = position_start, high = position_end - 1, ans = 0;
    while (low <= high) {
        int64_t mid = (low + high) / 2;
        if (d_randstrobes[mid].hash == key) {
            low = mid + 1;
            ans = mid;
        } else {
            high = mid - 1;
        }
    }
    return ans - position_start + 1;
}

__device__ inline size_t gpu_find(
    const RefRandstrobe *d_randstrobes,
    const my_bucket_index_t *d_randstrobe_start_indices,
    const randstrobe_hash_t key,
    int bits
) {
    const unsigned int top_N = key >> (64 - bits);
    my_bucket_index_t position_start = d_randstrobe_start_indices[top_N];
    my_bucket_index_t position_end = d_randstrobe_start_indices[top_N + 1];
    if(position_end - position_start < 64) {
        for (my_bucket_index_t i = position_start; i < position_end; ++i) {
            if (d_randstrobes[i].hash == key) {
                return i;
            }
        }
        return static_cast<size_t>(-1); // No match
    } else {
        my_bucket_index_t low = position_start, high = position_end;
        while (low < high) {
            my_bucket_index_t mid = low + (high - low) / 2;
            if (d_randstrobes[mid].hash < key) {
                low = mid + 1;
            } else {
                high = mid;
            }
        }
        if (d_randstrobes[low].hash == key) {
            return low;
        } else return static_cast<size_t>(-1); // No match
    }
}


template <typename T>
struct DefaultCompare {
    __device__ __forceinline__
        bool operator()(const T& a, const T& b) const {
        return a < b;
    }
};

template <typename T, typename Compare = DefaultCompare<T>>
__device__ void quick_sort_iterative(
    T* data,
    int low,
    int high,
    Compare comp = DefaultCompare<T>()
) {
    if (low > high) return;
    int vec_size = high - low + 1;
    my_vector<int>stack_vec(vec_size * 2);
    int* stack = stack_vec.data;
    int top = -1;
    stack[++top] = low;
    stack[++top] = high;
    while (top >= 0) {
        high = stack[top--];
        low = stack[top--];
        // Partition
        T pivot = data[high];
        int i = low - 1;
        for (int j = low; j < high; ++j) {
            //if (data[j] < pivot) {
            if (comp(data[j], pivot)) {
                ++i;
                T temp = data[i];
                data[i] = data[j];
                data[j] = temp;
            }
        }
        T temp = data[i + 1];
        data[i + 1] = data[high];
        data[high] = temp;
        int pivot_index = i + 1;
        if (pivot_index - 1 > low) {
            stack[++top] = low;
            stack[++top] = pivot_index - 1;
        }
        if (pivot_index + 1 < high) {
            stack[++top] = pivot_index + 1;
            stack[++top] = high;
        }
    }
}


template <typename T>
__device__ void bubble_sort(T* data, int size) {
    for (int i = 0; i < size - 1; ++i) {
        for (int j = 0; j < size - i - 1; ++j) {
            if (data[j + 1] < data[j]) {
                T temp = data[j];
                data[j] = data[j + 1];
                data[j + 1] = temp;
            }
        }
    }
}

template <typename T>
__device__ void quick_sort(T* data, int size) {
    quick_sort_iterative(data, 0, size - 1);
}

struct Rescue_Seeds {
    int read_id;
    int read_fr;
    int seeds_num;
    QueryRandstrobe* seeds;
};

__device__ int lock = 0;

__device__ void acquire_lock() {
    while (atomicCAS(&lock, 0, 1) != 0) {
    }
}

__device__ void release_lock() {
    atomicExch(&lock, 0);
}

__device__ void print_nam(Nam nam) {
    printf("nam_id: %d, ref_id: %d, ref_start: %d, ref_end: %d, query_start: %d, query_end: %d, n_hits: %d, is_rc: %d\n",
           nam.nam_id, nam.ref_id, nam.ref_start, nam.ref_end, nam.query_start, nam.query_end, nam.n_hits, nam.is_rc);
}

__device__ void print_str(my_string str) {
    for(int i = 0; i < str.size(); i++) {
        printf("%c", str[i]);
    }
    printf("\n");
}


#define MAX_TRIES_LIMIT (mapping_parameters->max_tries * 2 + 2)
#define MAX_TRIES_LIMIT2 (map_param.max_tries * 2 + 2)



struct GPURead {
    char* seq;
    char* rc;
    int length;
    __device__ int size() const { return length; }
};


struct TODOInfos {
    uint32_t read_info;
    int ref_id;
    int r_begin;
    int r_len;
    char *seq, *ref;
};


struct GPUAlignTmpRes {
    int type;
    // type 0 : size1 == 0 size2 == 0, unmapped_pair
    // type 1 : size1 == 0, rescue read1
    // type 2 : size2 == 0, rescue read2
    // type 3 : good pair
    // type 4 : for loop
    int mapq1;
    int mapq2;
    int type4_loop_size;
    my_vector<int> is_extend_seed;
    my_vector<int> consistent_nam;
    my_vector<int> is_read1;
    my_vector<Nam> type4_nams;
    my_vector<Nam> todo_nams;
    my_vector<int> done_align;
    // if done_align, align_res is the alignment results
    my_vector<GPUAlignment> align_res;
    my_vector<CigarData> cigar_info;
    my_vector<TODOInfos> todo_infos;
};

struct GPUReferences {
    my_vector<my_string> sequences;
    my_vector<int> lengths;
    int num_refs;
};

struct GPUAlignmentInfo {
    my_vector<uint32_t> cigar;
    unsigned int edit_distance{0};
    unsigned int ref_start{0};
    unsigned int ref_end{0};
    unsigned int query_start{0};
    unsigned int query_end{0};
    int sw_score{0};

    __device__ int ref_span() const { return ref_end - ref_start; }
};

__device__ void cigar_push(my_vector<uint32_t>& m_ops, uint8_t op, int len) {
    assert(op < 16);
    if (m_ops.empty() || (m_ops.back() & 0xf) != op) {
        m_ops.push_back(len << 4 | op);
    } else {
        m_ops.back() += len << 4;
    }
}

__device__ void my_hamming_align(const my_string &query, const my_string &ref, int match, int mismatch, int end_bonus, GPUAlignmentInfo& aln) {
    if (query.length() != ref.length()) {
        return;
    }
    size_t n = query.length();

    size_t start = 0; // start of the current segment
    int score = end_bonus; // accumulated score so far in the current segment

    size_t best_start = 0;
    size_t best_end = 0;
    int best_score = 0;
    for (size_t i = 0; i < n; ++i) {
        if (query[i] == ref[i]) {
            score += match;
        } else {
            score -= mismatch;
        }
        if (score < 0) {
            start = i + 1;
            score = 0;
        }
        if (score > best_score) {
            best_start = start;
            best_score = score;
            best_end = i + 1;
        }
    }
    if (score + end_bonus > best_score) {
        best_score = score + end_bonus;
        best_end = query.length();
        best_start = start;
    }

    size_t segment_start = best_start;
    size_t segment_end = best_end;
    score = best_score;

    if (segment_start > 0) {
        cigar_push(aln.cigar, CIGAR_SOFTCLIP, segment_start);
    }

    // Create CIGAR string and count mismatches
    int counter = 0;
    bool prev_is_match = false;
    int mismatches = 0;
    bool first = true;
    for (size_t i = segment_start; i < segment_end; i++) {
        bool is_match = query[i] == ref[i];
        mismatches += is_match ? 0 : 1;
        if (!first && is_match != prev_is_match) {
            cigar_push(aln.cigar, prev_is_match ? CIGAR_EQ : CIGAR_X, counter);
            counter = 0;
        }
        counter++;
        prev_is_match = is_match;
        first = false;
    }
    if (!first) {
        cigar_push(aln.cigar, prev_is_match ? CIGAR_EQ : CIGAR_X, counter);
    }

    int soft_right = query.length() - segment_end;
    if (soft_right > 0) {
        cigar_push(aln.cigar, CIGAR_SOFTCLIP, soft_right);
    }

    aln.sw_score = score;
    aln.edit_distance = mismatches;
    aln.ref_start = segment_start;
    aln.ref_end = segment_end;
    aln.query_start = segment_start;
    aln.query_end = segment_end;
    return;
}

__device__ bool gpu_extend_seed_part(
    GPUAlignTmpRes& align_tmp_res,
    const AlignmentParameters& aligner_parameters,
    const Nam& nam,
    const GPUReferences& references,
    const GPURead& read,
    bool consistent_nam
) {
    const my_string query(nam.is_rc ? read.rc : read.seq, read.length);
    const my_string ref = references.sequences[nam.ref_id];

    const auto projected_ref_start = my_max(0, nam.ref_start - nam.query_start);
    const auto projected_ref_end = my_min(nam.ref_end + query.size() - nam.query_end, ref.size());

    GPUAlignmentInfo info;
    int result_ref_start;
    bool gapped = true;
    if (projected_ref_end - projected_ref_start == query.size() && consistent_nam) {
        my_string ref_segm_ham = ref.substr(projected_ref_start, query.size());
        int hamming_dist = 0;
        int limit_error = query.size() * 0.05;
        for (size_t i = 0; i < query.size() && hamming_dist <= limit_error; ++i) {
            if (query[i] != ref_segm_ham[i]) {
                ++hamming_dist;
            }
        }
        if (hamming_dist >= 0 && (((float) hamming_dist / query.size()) < 0.05)) {  //Hamming distance worked fine, no need to ksw align
            my_hamming_align(
                query, ref_segm_ham, aligner_parameters.match, aligner_parameters.mismatch,
                aligner_parameters.end_bonus, info
            );
            result_ref_start = projected_ref_start + info.ref_start;
            gapped = false;
        }
    }

    align_tmp_res.todo_nams.push_back(nam);
    align_tmp_res.is_extend_seed.push_back(true);
    if (gapped) {
        // not pass hamming, pending to do align on GPU, tag is false
        GPUAlignment alignment;
        align_tmp_res.done_align.push_back(false);
        align_tmp_res.align_res.push_back(alignment);
        align_tmp_res.cigar_info.length++;
        align_tmp_res.cigar_info.back().cigar[0] = 0;
    } else {
        // pass hamming, store result, tag is true
        align_tmp_res.done_align.push_back(true);
        int softclipped = info.query_start + (query.size() - info.query_end);
        GPUAlignment alignment;
        //alignment.cigar.move_from(info.cigar);
        alignment.edit_distance = info.edit_distance;
        alignment.global_ed = info.edit_distance + softclipped;
        alignment.score = info.sw_score;
        alignment.ref_start = result_ref_start;
        alignment.length = info.ref_span();
        alignment.is_rc = nam.is_rc;
        alignment.is_unaligned = false;
        alignment.ref_id = nam.ref_id;
        alignment.gapped = gapped;
        align_tmp_res.align_res.push_back(alignment);
        assert(info.cigar.size() + 1 <= MAX_CIGAR_ITEM);
        align_tmp_res.cigar_info.length++;
        align_tmp_res.cigar_info.back().cigar[0] = info.cigar.size();
        for (int i = 0; i < info.cigar.size(); i++) {
            align_tmp_res.cigar_info.back().cigar[i + 1] = info.cigar[i];
        }
    }
    return gapped;
}


__device__ bool gpu_has_shared_substring(const my_string& read_seq, const my_string& ref_seq, int k) {
    int sub_size = 2 * k / 3;
    int step_size = k / 3;
    //my_vector<uint32_t> hash0;
    __shared__ uint32_t g_hash0[50 * 32];
    uint32_t *hash0 = &(g_hash0[threadIdx.x * 50]);
    int N = 0;
    for (int i = 0; i + sub_size < read_seq.size(); i += step_size) {
        uint32_t h = 0;
        for (int j = 0; j < sub_size; ++j) {
            unsigned char base = read_seq[i + j];
            uint8_t code = gpu_nt2int_mod8[base % 8];
            h = (h << 2) | code;
        }
        hash0[N++] = h;
        //assert(N <= 50);
        //hash0.push_back(h);
        //N++;
    }
    quick_sort(&(hash0[0]), N);
    for (int i = 0; i + sub_size < ref_seq.size(); i++) {
        uint32_t h = 0;
        for (int j = 0; j < sub_size; ++j) {
            unsigned char base = ref_seq[i + j];
            uint8_t code = gpu_nt2int_mod8[base % 8];
            h = (h << 2) | code;
        }
        int left = 0, right = N - 1;
        while (left <= right) {
            int mid = (left + right) / 2;
            if (hash0[mid] == h) return true;
            else if (hash0[mid] < h) left = mid + 1;
            else right = mid - 1;
        }
    }
    return false;
}

__device__ bool gpu_rescue_mate_part(
    GPUAlignTmpRes& align_tmp_res,
    const AlignmentParameters& aligner_parameters,
    const Nam& nam,
    const GPUReferences& references,
    const GPURead& read,
    float mu,
    float sigma,
    int k
) {
    GPUAlignment alignment;
    int a, b;
    my_string r_tmp;
    auto read_len = read.size();

    if (nam.is_rc) {
        r_tmp = my_string(read.seq, read_len);
        a = nam.ref_start - nam.query_start - (mu + 5 * sigma);
        b = nam.ref_start - nam.query_start + read_len / 2;  // at most half read overlap
    } else {
        r_tmp = my_string(read.rc, read_len);                                             // mate is rc since fr orientation
        a = nam.ref_end + (read_len - nam.query_end) - read_len / 2;  // at most half read overlap
        b = nam.ref_end + (read_len - nam.query_end) + (mu + 5 * sigma);
    }

    auto ref_len = references.lengths[nam.ref_id];
    auto ref_start = my_max(0, my_min(a, ref_len));
    auto ref_end = my_min(ref_len, my_max(0, b));

    align_tmp_res.todo_nams.push_back(nam);
    align_tmp_res.is_extend_seed.push_back(false);
    if (ref_end < ref_start + k) {
        //        alignment.cigar = Cigar();
        alignment.edit_distance = read_len;
        alignment.score = 0;
        alignment.ref_start = 0;
        alignment.is_rc = nam.is_rc;
        alignment.ref_id = nam.ref_id;
        alignment.is_unaligned = true;
        align_tmp_res.done_align.push_back(true);
        align_tmp_res.align_res.push_back(alignment);
        align_tmp_res.cigar_info.length++;
        align_tmp_res.cigar_info.back().cigar[0] = 0;
        return true;
    }
    my_string ref_segm = references.sequences[nam.ref_id].substr(ref_start, ref_end - ref_start);

    if (!gpu_has_shared_substring(r_tmp, ref_segm, k)) {
        //        alignment.cigar = Cigar();
        alignment.edit_distance = read_len;
        alignment.score = 0;
        alignment.ref_start = 0;
        alignment.is_rc = nam.is_rc;
        alignment.ref_id = nam.ref_id;
        alignment.is_unaligned = true;
        align_tmp_res.done_align.push_back(true);
        align_tmp_res.align_res.push_back(alignment);
        align_tmp_res.cigar_info.length++;
        align_tmp_res.cigar_info.back().cigar[0] = 0;
        return true;
    }

    align_tmp_res.done_align.push_back(false);
    align_tmp_res.align_res.push_back(alignment);
    align_tmp_res.cigar_info.length++;
    align_tmp_res.cigar_info.back().cigar[0] = 0;
    return false;
}


__device__ bool gpu_reverse_nam_if_needed(Nam& nam, const GPURead& read, const GPUReferences& references, int k) {
    auto read_len = read.size();
    my_string ref_start_kmer = references.sequences[nam.ref_id].substr(nam.ref_start, k);
    my_string ref_end_kmer = references.sequences[nam.ref_id].substr(nam.ref_end - k, k);


    my_string seq, seq_rc;
    if (nam.is_rc) {
        seq = my_string(read.rc, read_len);
        seq_rc = my_string(read.seq, read_len);
    } else {
        seq = my_string(read.seq, read_len);
        seq_rc = my_string(read.rc, read_len);
    }
    my_string read_start_kmer = seq.substr(nam.query_start, k);
    my_string read_end_kmer = seq.substr(nam.query_end - k, k);
    if (ref_start_kmer == read_start_kmer && ref_end_kmer == read_end_kmer) {
        return true;
    }

    // False forward or false reverse (possible due to symmetrical hash values)
    //    we need two extra checks for this - hopefully this will remove all the false hits we see (true hash collisions should be very few)
    int q_start_tmp = read_len - nam.query_end;
    int q_end_tmp = read_len - nam.query_start;
    // false reverse hit, change coordinates in nam to forward
    read_start_kmer = seq_rc.substr(q_start_tmp, k);
    read_end_kmer = seq_rc.substr(q_end_tmp - k, k);
    if (ref_start_kmer == read_start_kmer && ref_end_kmer == read_end_kmer) {
        nam.is_rc = !nam.is_rc;
        nam.query_start = q_start_tmp;
        nam.query_end = q_end_tmp;
        return true;
    }
    return false;
}


__device__ void gpu_part2_extend_seed_get_str(
    GPUAlignTmpRes& align_tmp_res,
    int j,
    const GPURead& read1,
    const GPURead& read2,
    const GPUReferences& references,
    int read_id
) {
    Nam nam = align_tmp_res.todo_nams[j];
    GPURead read = align_tmp_res.is_read1[j] ? read1 : read2;
    const my_string query = nam.is_rc ? my_string(read.rc, read.length) : my_string(read.seq, read.length);
    const my_string ref = references.sequences[nam.ref_id];

    const auto projected_ref_start = my_max(0, nam.ref_start - nam.query_start);
    const auto projected_ref_end = my_max(nam.ref_end + query.size() - nam.query_end, ref.size());

    const int diff = my_abs(nam.ref_span() - nam.query_span());
    const int ext_left = my_min(50, projected_ref_start);
    const int ref_start = projected_ref_start - ext_left;
    const int ext_right = my_min(50, ref.size() - nam.ref_end);
    auto ref_segm_size = read.size() + diff + ext_left + ext_right;
    if (ref_start + ref_segm_size > references.lengths[nam.ref_id]) ref_segm_size = references.lengths[nam.ref_id] - ref_start;
    uint32_t packed = (static_cast<uint32_t>(align_tmp_res.is_read1[j]) << 31) |
                      (static_cast<uint32_t>(nam.is_rc) << 30) |
                      (static_cast<uint32_t>(0) << 15) |
                      (static_cast<uint32_t>(read.length));
    align_tmp_res.todo_infos.push_back({packed, nam.ref_id, ref_start, ref_segm_size, query.data, ref.data});
}


__device__ void gpu_part2_rescue_mate_get_str(
    GPUAlignTmpRes& align_tmp_res,
    int j,
    const GPURead& read1,
    const GPURead& read2,
    const GPUReferences& references,
    float mu,
    float sigma,
    int read_id
) {
    Nam nam = align_tmp_res.todo_nams[j];
    GPURead read = align_tmp_res.is_read1[j] ? read1 : read2;
    int a, b;
    my_string r_tmp;
    auto read_len = read.size();

    if (nam.is_rc) {
        r_tmp = my_string(read.seq, read.length);
        a = nam.ref_start - nam.query_start - (mu + 5 * sigma);
        b = nam.ref_start - nam.query_start + read_len / 2;  // at most half read overlap
    } else {
        r_tmp = my_string(read.rc, read.length);                                              // mate is rc since fr orientation
        a = nam.ref_end + (read_len - nam.query_end) - read_len / 2;  // at most half read overlap
        b = nam.ref_end + (read_len - nam.query_end) + (mu + 5 * sigma);
    }

    auto ref_len = references.lengths[nam.ref_id];
    auto ref_start = my_max(0, my_min(a, ref_len));
    auto ref_end = my_min(ref_len, my_max(0, b));
    uint32_t packed = (static_cast<uint32_t>(align_tmp_res.is_read1[j]) << 31) |
                      (static_cast<uint32_t>(!nam.is_rc) << 30) |
                      (static_cast<uint32_t>(0) << 15) |
                      (static_cast<uint32_t>(read.length));
    align_tmp_res.todo_infos.push_back({packed, nam.ref_id, ref_start, ref_end - ref_start, r_tmp.data, references.sequences[nam.ref_id].data});
}

__device__ void gpu_rescue_read_part(
    int flag,
    GPUAlignTmpRes& align_tmp_res,
    const GPURead& read2,  // read to be rescued
    const GPURead& read1,  // read that has NAMs
    const AlignmentParameters& aligner_parameters,
    const GPUReferences& references,
    my_vector<Nam>& nams1,
    int max_tries,
    float dropoff,
    int k,
    float mu,
    float sigma,
    size_t max_secondary,
    double secondary_dropoff,
    bool swap_r1r2
) {
    //align_tmp_res.type = flag;
    Nam n_max1 = nams1[0];
    int tries = 0;
    // this loop is safe, loop size is stable
    for (int i = 0; i < nams1.size(); i++) {
        Nam &nam = nams1[i];
        float score_dropoff1 = (float) nam.n_hits / n_max1.n_hits;
        // only consider top hits (as minimap2 does) and break if below dropoff cutoff.
        if (tries >= max_tries || score_dropoff1 < dropoff) {
            break;
        }

        const bool consistent_nam = gpu_reverse_nam_if_needed(nam, read1, references, k);
        // reserve extend and store info
        if(flag == 1) align_tmp_res.is_read1.push_back(true);
        else align_tmp_res.is_read1.push_back(false);
        bool gapped = gpu_extend_seed_part(align_tmp_res, aligner_parameters, nam, references, read1, consistent_nam);

        // Force SW alignment to rescue mate
        if(flag == 1) align_tmp_res.is_read1.push_back(false);
        else align_tmp_res.is_read1.push_back(true);
        bool is_unaligned = gpu_rescue_mate_part(align_tmp_res, aligner_parameters, nam, references, read2, mu, sigma, k);
        tries++;
    }
}

__device__ inline bool gpu_is_proper_nam_pair3(const Nam nam1, const Nam nam2, float mu, float sigma) {
    int a = my_max(0, nam1.ref_start - nam1.query_start);
    int b = my_max(0, nam2.ref_start - nam2.query_start);

    // r1 ---> <---- r2
    bool r1_r2 = nam2.is_rc && (a <= b) && (b - a < mu + 10 * sigma);
    if(r1_r2) return true;

    // r2 ---> <---- r1
    bool r2_r1 = nam1.is_rc && (b <= a) && (a - b < mu + 10 * sigma);
    if(r2_r1) return true;
    return false;
}


__device__ inline bool gpu_is_proper_nam_pair2(const Nam nam1, const Nam nam2, float mu, float sigma) {
    if (nam1.is_rc == nam2.is_rc) {
        return false;
    }
    int a = my_max(0, nam1.ref_start - nam1.query_start);
    int b = my_max(0, nam2.ref_start - nam2.query_start);

    // r1 ---> <---- r2
    bool r1_r2 = nam2.is_rc && (a <= b) && (b - a < mu + 10 * sigma);
    if(r1_r2) return true;

    // r2 ---> <---- r1
    bool r2_r1 = nam1.is_rc && (b <= a) && (a - b < mu + 10 * sigma);
    if(r2_r1) return true;
    return false;
}

__device__ inline bool gpu_is_proper_nam_pair(const Nam nam1, const Nam nam2, float mu, float sigma) {
    if (nam1.ref_id != nam2.ref_id || nam1.is_rc == nam2.is_rc) {
        return false;
    }
    int a = my_max(0, nam1.ref_start - nam1.query_start);
    int b = my_max(0, nam2.ref_start - nam2.query_start);

    // r1 ---> <---- r2
    bool r1_r2 = nam2.is_rc && (a <= b) && (b - a < mu + 10 * sigma);
    //    if(r1_r2) return 1;

    // r2 ---> <---- r1
    bool r2_r1 = nam1.is_rc && (b <= a) && (a - b < mu + 10 * sigma);
    //    if(r2_r1) return 1;
    //    return 0;

    return r1_r2 || r2_r1;
}

__device__ float gpu_top_dropoff(my_vector<Nam>& nams) {
    auto& n_max = nams[0];
    if (n_max.n_hits <= 2) {
        return 1.0;
    }
    if (nams.size() > 1) {
        return (float) nams[1].n_hits / n_max.n_hits;
    }
    return 0.0;
}

__device__ uint8_t gpu_get_mapq(const my_vector<Nam>& nams, const Nam& n_max) {
    if (nams.size() <= 1) {
        return 60;
    }
    const float s1 = n_max.score;
    const float s2 = nams[1].score;
    // from minimap2: MAPQ = 40(1−s2/s1) ·min{1,|M|/10} · log s1
    const float min_matches = my_min(n_max.n_hits / 10.0, 1.0);
    const int uncapped_mapq = 40 * (1 - s2 / s1) * min_matches * log(s1);
    return my_min(uncapped_mapq, 60);
}

__device__ bool gpu_is_proper_pair(const GPUAlignment& alignment1, const GPUAlignment& alignment2, float mu, float sigma) {
    const int dist = alignment2.ref_start - alignment1.ref_start;
    const bool same_reference = alignment1.ref_id == alignment2.ref_id;
    const bool both_aligned = same_reference && !alignment1.is_unaligned && !alignment2.is_unaligned;
    const bool r1_r2 = !alignment1.is_rc && alignment2.is_rc && dist >= 0; // r1 ---> <---- r2
    const bool r2_r1 = !alignment2.is_rc && alignment1.is_rc && dist <= 0; // r2 ---> <---- r1
    const bool rel_orientation_good = r1_r2 || r2_r1;
    const bool insert_good = std::abs(dist) <= mu + 6 * sigma;

    return both_aligned && insert_good && rel_orientation_good;
}

struct GPUInsertSizeDistribution {
    float sample_size = 1;
    float mu = 300;
    float sigma = 100;
    float V = 10000;
    float SSE = 10000;

    // Add a new observation
    __device__ void update(int dist) {
        if (dist >= 2000) {
            return;
        }
        const float e = dist - mu;
        mu += e / sample_size;  // (1.0/(sample_size +1.0)) * (sample_size*mu + d);
        SSE += e * (dist - mu);
        if (sample_size > 1) {
            //d < 1000 ? ((sample_size +1.0)/sample_size) * ( (V*sample_size/(sample_size +1)) + ((mu-d)*(mu-d))/sample_size ) : V;
            V = SSE / (sample_size - 1.0);
        } else {
            V = SSE;
        }
        sigma = sqrtf(V);
        sample_size = sample_size + 1.0;
        if (mu < 0) {
            printf("mu negative, mu: %f sigma: %f SSE: %f sample size: %f\n", mu, sigma, SSE, sample_size);
            assert(false);
        }
        if (SSE < 0) {
            printf("SSE negative, mu: %f sigma: %f SSE: %f sample size: %f\n", mu, sigma, SSE, sample_size);
            assert(false);
        }
    }
};


struct gpu_NamPair {
    int score;
    my_vector<Nam> *nams1;
    my_vector<Nam> *nams2;
    int i1, i2;
    //Nam nam1;
    //Nam nam2;
};

struct ref_ids_edge {
    int pre;
    int ref_id;
};

#define key_mod_val 29

__device__ void get_best_scoring_nam_pairs_sort1(
    my_vector<gpu_NamPair>& joint_nam_scores,
    my_vector<Nam>& nams1,
    my_vector<Nam>& nams2,
    float mu,
    float sigma,
    int max_tries,
    int tid
) {
    int nams1_len = nams1.size();
    int nams2_len = nams2.size();
    my_vector<bool> added_n1(nams1_len);
    my_vector<bool> added_n2(nams2_len);
    for(int i = 0; i < nams1_len; i++) added_n1.push_back(false);
    for(int i = 0; i < nams2_len; i++) added_n2.push_back(false);
    int pre_ref_id;

    my_vector<int> ref_ids1(nams1_len);
    pre_ref_id = -1;
    for (int i = 0; i < nams1.size(); i++) {
        if (nams1[i].ref_id != pre_ref_id) {
            pre_ref_id = nams1[i].ref_id;
            ref_ids1.push_back(nams1[i].ref_id);
        }
    }

    my_vector<int> ref_ids2(nams2_len);
    pre_ref_id = -1;
    for (int i = 0; i < nams2.size(); i++) {
        if (nams2[i].ref_id != pre_ref_id) {
            pre_ref_id = nams2[i].ref_id;
            ref_ids2.push_back(nams2[i].ref_id);
        }
    }

    my_vector<int> result(ref_ids1.size() + ref_ids2.size());
    int p1 = 0, p2 = 0;

    while (p1 < ref_ids1.size() && p2 < ref_ids2.size()) {
        if (ref_ids1[p1] < ref_ids2[p2]) {
            result.push_back(ref_ids1[p1++]);
        } else if (ref_ids1[p1] > ref_ids2[p2]) {
            result.push_back(ref_ids2[p2++]);
        } else {
            result.push_back(ref_ids1[p1]);
            ++p1;
            ++p2;
        }
    }
    while (p1 < ref_ids1.size()) result.push_back(ref_ids1[p1++]);
    while (p2 < ref_ids2.size()) result.push_back(ref_ids2[p2++]);


    int best_joint_hits = 0;
    int pos1 = 0, pos2 = 0;
    //for (int ref_id = 0; ref_id < 3000; ref_id++) {
    for (int gid = 0; gid < result.size(); gid++) {
        int ref_id = result[gid];
        while (pos1 < nams1_len && nams1[pos1].ref_id < ref_id) pos1++;
        while (pos2 < nams2_len && nams2[pos2].ref_id < ref_id) pos2++;
        int end1 = pos1, end2 = pos2;
        while (end1 < nams1_len && nams1[end1].ref_id == ref_id) end1++;
        while (end2 < nams2_len && nams2[end2].ref_id == ref_id) end2++;
        //if (pos1 == nams1_len || pos2 == nams2_len) break;
        int round_size = 0;
        //for (int i = pos1, k = 0; i < end1 && k < max_tries; i++, k++) {
        for (int i = pos1, k = 0; i < end1; i++, k++) {
            const Nam &nam1 = nams1[i];
            //for (int j = pos2, p = 0; j < end2 && p < max_tries; j++, p++) {
            for (int j = pos2, p = 0; j < end2; j++, p++) {
                const Nam &nam2 = nams2[j];
                int joint_hits = nam1.n_hits + nam2.n_hits;
                if (joint_hits < 0.5 * best_joint_hits || round_size > max_tries * 2) {
                    //if (joint_hits < best_joint_hits / 2) {
                    break;
                }
                //assert(nam1.ref_id == ref_id && nam1.ref_id == nam2.ref_id);
                if (gpu_is_proper_nam_pair2(nam1, nam2, mu, sigma)) {
                    joint_nam_scores.push_back(gpu_NamPair{joint_hits, &nams1, &nams2, i, j});
                    //joint_nam_scores.push_back(gpu_NamPair{joint_hits, nam1, nam2});
                    added_n1[i] = 1;
                    added_n2[j] = 1;
                    best_joint_hits = my_max(joint_hits, best_joint_hits);
                    round_size++;
                }
            }
            if (round_size > max_tries * 2) break;
        }
    }

    // Find high-scoring R1 NAMs that are not part of a proper pair
    Nam dummy_nam;
    dummy_nam.ref_start = -1;
    int best_joint_hits1 = best_joint_hits > 0 ? best_joint_hits : nams1[0].n_hits;
    //for(int i = 0; i < my_min(nams1.size(), max_tries); i++) {
    int now_cnt = 0;
    pre_ref_id = nams1[0].ref_id;
    for(int i = 0; i < nams1.size(); i++) {
        int ref_id = nams1[i].ref_id;
        if (ref_id == pre_ref_id) now_cnt++;
        else {
            now_cnt = 1;
            pre_ref_id = ref_id;
        }
        if (now_cnt > max_tries) continue;
        Nam nam1 = nams1[i];
        if (nam1.n_hits < best_joint_hits1 / 2) {
            //break;
            continue;
        }
        if (added_n1[i]) {
            continue;
        }
        joint_nam_scores.push_back(gpu_NamPair{nam1.n_hits, &nams1, &nams2, i, -1});
        //joint_nam_scores.push_back(gpu_NamPair{nam1.n_hits, nam1, dummy_nam});
    }

    // Find high-scoring R2 NAMs that are not part of a proper pair
    int best_joint_hits2 = best_joint_hits > 0 ? best_joint_hits : nams2[0].n_hits;
    //for(int i = 0; i < my_min(nams2.size(), max_tries); i++) {
    now_cnt = 0;
    pre_ref_id = nams2[0].ref_id;
    for(int i = 0; i < nams2.size(); i++) {
        int ref_id = nams2[i].ref_id;
        if (ref_id == pre_ref_id) now_cnt++;
        else {
            now_cnt = 1;
            pre_ref_id = ref_id;
        }
        if (now_cnt > max_tries) continue;
        Nam nam2 = nams2[i];
        if (nam2.n_hits < best_joint_hits2 / 2) {
            //break;
            continue;
        }
        if (added_n2[i]) {
            continue;
        }
        joint_nam_scores.push_back(gpu_NamPair{nam2.n_hits, &nams1, &nams2, -1, i});
        //joint_nam_scores.push_back(gpu_NamPair{nam2.n_hits, dummy_nam, nam2});
    }

    quick_sort_iterative(&(joint_nam_scores[0]), 0, joint_nam_scores.size() - 1, [](const gpu_NamPair &n1, const gpu_NamPair &n2) {
        //if (n1.score != n2.score) return n1.score > n2.score;
        //if (n1.nam1.score != n2.nam1.score) return n1.nam1.score > n2.nam1.score;
        //if (n1.nam1.is_rc != n2.nam1.is_rc) return !n1.nam1.is_rc;  // false < true
        //if (n1.nam1.query_end != n2.nam1.query_end) return n1.nam1.query_end < n2.nam1.query_end;
        //if (n1.nam1.query_start != n2.nam1.query_start) return n1.nam1.query_start < n2.nam1.query_start;
        //if (n1.nam1.ref_end != n2.nam1.ref_end) return n1.nam1.ref_end < n2.nam1.ref_end;
        //if (n1.nam1.ref_start != n2.nam1.ref_start) return n1.nam1.ref_start < n2.nam1.ref_start;
        //if (n1.nam2.score != n2.nam2.score) return n1.nam2.score > n2.nam2.score;
        //if (n1.nam2.is_rc != n2.nam2.is_rc) return !n1.nam2.is_rc;
        //if (n1.nam2.query_end != n2.nam2.query_end) return n1.nam2.query_end < n2.nam2.query_end;
        //if (n1.nam2.query_start != n2.nam2.query_start) return n1.nam2.query_start < n2.nam2.query_start;
        //if (n1.nam2.ref_end != n2.nam2.ref_end) return n1.nam2.ref_end < n2.nam2.ref_end;
        //return n1.nam2.ref_start < n2.nam2.ref_start;

        Nam dummy_nam;
        dummy_nam.ref_start = -1;
        Nam nam1_1 = n1.i1 == -1 ? dummy_nam : (*n1.nams1)[n1.i1];
        Nam nam1_2 = n1.i2 == -1 ? dummy_nam : (*n1.nams2)[n1.i2];
        Nam nam2_1 = n2.i1 == -1 ? dummy_nam : (*n2.nams1)[n2.i1];
        Nam nam2_2 = n2.i2 == -1 ? dummy_nam : (*n2.nams2)[n2.i2];

        //return n1.score > n2.score;
        if (n1.score != n2.score) return n1.score > n2.score;
        if (nam1_1.score != nam2_1.score) return nam1_1.score > nam2_1.score;
        if (nam1_1.is_rc != nam2_1.is_rc) return !nam1_1.is_rc;  // false < true
        if (nam1_1.query_end != nam2_1.query_end) return nam1_1.query_end < nam2_1.query_end;
        if (nam1_1.query_start != nam2_1.query_start) return nam1_1.query_start < nam2_1.query_start;
        if (nam1_1.ref_end != nam2_1.ref_end) return nam1_1.ref_end < nam2_1.ref_end;
        if (nam1_1.ref_start != nam2_1.ref_start) return nam1_1.ref_start < nam2_1.ref_start;
        if (nam1_2.score != nam2_2.score) return nam1_2.score > nam2_2.score;
        if (nam1_2.is_rc != nam2_2.is_rc) return !nam1_2.is_rc;
        if (nam1_2.query_end != nam2_2.query_end) return nam1_2.query_end < nam2_2.query_end;
        if (nam1_2.query_start != nam2_2.query_start) return nam1_2.query_start < nam2_2.query_start;
        if (nam1_2.ref_end != nam2_2.ref_end) return nam1_2.ref_end < nam2_2.ref_end;
        return nam1_2.ref_start < nam2_2.ref_start;

    });

    return;
}

__device__ void get_best_scoring_nam_pairs_sort2(
    my_vector<gpu_NamPair>& joint_nam_scores,
    my_vector<Nam>& nams1,
    my_vector<Nam>& nams2,
    float mu,
    float sigma,
    int max_tries,
    int tid
) {
    int nams1_len = nams1.size();
    int nams2_len = nams2.size();
    my_vector<bool> added_n1(nams1_len);
    my_vector<bool> added_n2(nams2_len);
    for(int i = 0; i < nams1_len; i++) added_n1.push_back(false);
    for(int i = 0; i < nams2_len; i++) added_n2.push_back(false);

    // find is_rc split pos
    int mid_pos1 = nams1.size();
    for (int i = 0; i < nams1.size(); i++) {
        if (nams1[i].is_rc == 1) {
            mid_pos1 = i;
            break;
        }
    }
    int mid_pos2 = nams2.size();
    for (int i = 0; i < nams2.size(); i++) {
        if (nams2[i].is_rc == 1) {
            mid_pos2 = i;
            break;
        }
    }


    int pre_ref_id, p1, p2, pos1, pos2;

    //my_vector<int> ref_ids1(mid_pos1);
    //pre_ref_id = -1;
    //for (int i = 0; i < mid_pos1; i++) {
    //    if (nams1[i].ref_id != pre_ref_id) {
    //        pre_ref_id = nams1[i].ref_id;
    //        ref_ids1.push_back(nams1[i].ref_id);
    //    }
    //}
    //my_vector<int> ref_ids2(nams2.size() - mid_pos2);
    //pre_ref_id = -1;
    //for (int i = mid_pos2; i < nams2.size(); i++) {
    //    if (nams2[i].ref_id != pre_ref_id) {
    //        pre_ref_id = nams2[i].ref_id;
    //        ref_ids2.push_back(nams2[i].ref_id);
    //    }
    //}

    //my_vector<int> result(my_min(ref_ids1.size(), ref_ids2.size()));
    //p1 = 0, p2 = 0;

    //while (p1 < ref_ids1.size() && p2 < ref_ids2.size()) {
    //    if (ref_ids1[p1] < ref_ids2[p2]) {
    //        result.push_back(ref_ids1[p1++]);
    //    } else if (ref_ids1[p1] > ref_ids2[p2]) {
    //        result.push_back(ref_ids2[p2++]);
    //    } else {
    //        result.push_back(ref_ids1[p1]);
    //        ++p1;
    //        ++p2;
    //    }
    //}
    //while (p1 < ref_ids1.size()) result.push_back(ref_ids1[p1++]);
    //while (p2 < ref_ids2.size()) result.push_back(ref_ids2[p2++]);

    int best_joint_hits = 0;
    pos1 = 0;
    pos2 = mid_pos2;
    nams1_len = mid_pos1;
    nams2_len = nams2.size();
    for (int ref_id = 0; ref_id < 30; ref_id++) {
        //for (int gid = 0; gid < result.size(); gid++) {
        //int ref_id = result[gid];
        while (pos1 < nams1_len && nams1[pos1].ref_id < ref_id) pos1++;
        while (pos2 < nams2_len && nams2[pos2].ref_id < ref_id) pos2++;
        int end1 = pos1, end2 = pos2;
        while (end1 < nams1_len && nams1[end1].ref_id == ref_id) end1++;
        while (end2 < nams2_len && nams2[end2].ref_id == ref_id) end2++;
        //if (pos1 == nams1_len || pos2 == nams2_len) break;
        int round_size = 0;
        for (int i = pos1; i < end1; i++) {
            const Nam &nam1 = nams1[i];
            for (int j = pos2; j < end2; j++) {
                const Nam &nam2 = nams2[j];
                int joint_hits = nam1.n_hits + nam2.n_hits;
                //if (joint_hits < best_joint_hits / 2 || round_size > max_tries) {
                ////if (joint_hits < best_joint_hits / 2) {
                //    break;
                //}
                //assert(nam1.ref_id == ref_id && nam1.ref_id == nam2.ref_id && nam1.is_rc == 0 && nam2.is_rc == 1);
                if (gpu_is_proper_nam_pair3(nam1, nam2, mu, sigma)) {
                    joint_nam_scores.push_back(gpu_NamPair{joint_hits, &nams1, &nams2, i, j});
                    added_n1[i] = 1;
                    added_n2[j] = 1;
                    best_joint_hits = my_max(joint_hits, best_joint_hits);
                    round_size++;
                }
            }
            //if (round_size > max_tries) break;
        }
    }

    //ref_ids1.clear();
    //pre_ref_id = -1;
    //for (int i = mid_pos1; i < nams1.size(); i++) {
    //    if (nams1[i].ref_id != pre_ref_id) {
    //        pre_ref_id = nams1[i].ref_id;
    //        ref_ids1.push_back(nams1[i].ref_id);
    //    }
    //}
    //ref_ids2.clear();
    //pre_ref_id = -1;
    //for (int i = 0; i < mid_pos2; i++) {
    //    if (nams2[i].ref_id != pre_ref_id) {
    //        pre_ref_id = nams2[i].ref_id;
    //        ref_ids2.push_back(nams2[i].ref_id);
    //    }
    //}

    //result.clear();
    //p1 = 0, p2 = 0;
    //while (p1 < ref_ids1.size() && p2 < ref_ids2.size()) {
    //    if (ref_ids1[p1] < ref_ids2[p2]) {
    //        result.push_back(ref_ids1[p1++]);
    //    } else if (ref_ids1[p1] > ref_ids2[p2]) {
    //        result.push_back(ref_ids2[p2++]);
    //    } else {
    //        result.push_back(ref_ids1[p1]);
    //        ++p1;
    //        ++p2;
    //    }
    //}
    //while (p1 < ref_ids1.size()) result.push_back(ref_ids1[p1++]);
    //while (p2 < ref_ids2.size()) result.push_back(ref_ids2[p2++]);

    pos1 = mid_pos1;
    pos2 = 0;
    nams1_len = nams1.size();
    nams2_len = mid_pos2;
    for (int ref_id = 0; ref_id < 30; ref_id++) {
        //for (int gid = 0; gid < result.size(); gid++) {
        //int ref_id = result[gid];
        while (pos1 < nams1_len && nams1[pos1].ref_id < ref_id) pos1++;
        while (pos2 < nams2_len && nams2[pos2].ref_id < ref_id) pos2++;
        int end1 = pos1, end2 = pos2;
        while (end1 < nams1_len && nams1[end1].ref_id == ref_id) end1++;
        while (end2 < nams2_len && nams2[end2].ref_id == ref_id) end2++;
        if (pos1 == nams1_len || pos2 == nams2_len) break;
        int round_size = 0;
        for (int i = pos1; i < end1; i++) {
            const Nam &nam1 = nams1[i];
            for (int j = pos2; j < end2; j++) {
                const Nam &nam2 = nams2[j];
                int joint_hits = nam1.n_hits + nam2.n_hits;
                //if (joint_hits < best_joint_hits / 2 || round_size > max_tries) {
                ////if (joint_hits < best_joint_hits / 2) {
                //    break;
                //}
                //assert(nam1.ref_id == ref_id && nam1.ref_id == nam2.ref_id && nam1.is_rc == 1 && nam2.is_rc == 0);
                if (gpu_is_proper_nam_pair3(nam1, nam2, mu, sigma)) {
                    joint_nam_scores.push_back(gpu_NamPair{joint_hits, &nams1, &nams2, i, j});
                    added_n1[i] = 1;
                    added_n2[j] = 1;
                    best_joint_hits = my_max(joint_hits, best_joint_hits);
                    round_size++;
                }
            }
            //if (round_size > max_tries) break;
        }
    }

    // Find high-scoring R1 NAMs that are not part of a proper pair
    Nam dummy_nam;
    dummy_nam.ref_start = -1;
    int best_joint_hits1 = best_joint_hits > 0 ? best_joint_hits : nams1[0].n_hits;
    int now_cnt = 0;
    pre_ref_id = nams1[0].ref_id;
    for(int i = 0; i < nams1.size(); i++) {
        //for(int i = 0; i < my_min(nams1.size(), max_tries); i++) {
        int ref_id = nams1[i].ref_id;
        if (ref_id == pre_ref_id) now_cnt++;
        else {
            now_cnt = 1;
            pre_ref_id = ref_id;
        }
        //if (now_cnt > max_tries) continue;
        Nam nam1 = nams1[i];
        if (nam1.n_hits < best_joint_hits1 / 2) {
            //break;
            continue;
        }
        if (added_n1[i]) {
            continue;
        }
        joint_nam_scores.push_back(gpu_NamPair{nam1.n_hits, &nams1, &nams2, i, -1});
        //joint_nam_scores.push_back(gpu_NamPair{nam1.n_hits, nam1, dummy_nam});
    }

    // Find high-scoring R2 NAMs that are not part of a proper pair
    int best_joint_hits2 = best_joint_hits > 0 ? best_joint_hits : nams2[0].n_hits;
    now_cnt = 0;
    pre_ref_id = nams2[0].ref_id;
    for(int i = 0; i < nams2.size(); i++) {
        //for(int i = 0; i < my_min(nams2.size(), max_tries); i++) {
        int ref_id = nams2[i].ref_id;
        if (ref_id == pre_ref_id) now_cnt++;
        else {
            now_cnt = 1;
            pre_ref_id = ref_id;
        }
        //if (now_cnt > max_tries) continue;
        Nam nam2 = nams2[i];
        if (nam2.n_hits < best_joint_hits2 / 2) {
            //break;
            continue;
        }
        if (added_n2[i]) {
            continue;
        }
        joint_nam_scores.push_back(gpu_NamPair{nam2.n_hits, &nams1, &nams2, -1, i});
        //joint_nam_scores.push_back(gpu_NamPair{nam2.n_hits, dummy_nam, nam2});
    }

    quick_sort_iterative(&(joint_nam_scores[0]), 0, joint_nam_scores.size() - 1, [](const gpu_NamPair &n1, const gpu_NamPair &n2) {
        //if (n1.score != n2.score) return n1.score > n2.score;
        //if (n1.nam1.score != n2.nam1.score) return n1.nam1.score > n2.nam1.score;
        //if (n1.nam1.is_rc != n2.nam1.is_rc) return !n1.nam1.is_rc;  // false < true
        //if (n1.nam1.query_end != n2.nam1.query_end) return n1.nam1.query_end < n2.nam1.query_end;
        //if (n1.nam1.query_start != n2.nam1.query_start) return n1.nam1.query_start < n2.nam1.query_start;
        //if (n1.nam1.ref_end != n2.nam1.ref_end) return n1.nam1.ref_end < n2.nam1.ref_end;
        //if (n1.nam1.ref_start != n2.nam1.ref_start) return n1.nam1.ref_start < n2.nam1.ref_start;
        //if (n1.nam2.score != n2.nam2.score) return n1.nam2.score > n2.nam2.score;
        //if (n1.nam2.is_rc != n2.nam2.is_rc) return !n1.nam2.is_rc;
        //if (n1.nam2.query_end != n2.nam2.query_end) return n1.nam2.query_end < n2.nam2.query_end;
        //if (n1.nam2.query_start != n2.nam2.query_start) return n1.nam2.query_start < n2.nam2.query_start;
        //if (n1.nam2.ref_end != n2.nam2.ref_end) return n1.nam2.ref_end < n2.nam2.ref_end;
        //return n1.nam2.ref_start < n2.nam2.ref_start;

        Nam dummy_nam;
        dummy_nam.ref_start = -1;
        Nam nam1_1 = n1.i1 == -1 ? dummy_nam : (*n1.nams1)[n1.i1];
        Nam nam1_2 = n1.i2 == -1 ? dummy_nam : (*n1.nams2)[n1.i2];
        Nam nam2_1 = n2.i1 == -1 ? dummy_nam : (*n2.nams1)[n2.i1];
        Nam nam2_2 = n2.i2 == -1 ? dummy_nam : (*n2.nams2)[n2.i2];

        //return n1.score > n2.score;
        if (n1.score != n2.score) return n1.score > n2.score;
        if (nam1_1.score != nam2_1.score) return nam1_1.score > nam2_1.score;
        if (nam1_1.is_rc != nam2_1.is_rc) return !nam1_1.is_rc;  // false < true
        if (nam1_1.query_end != nam2_1.query_end) return nam1_1.query_end < nam2_1.query_end;
        if (nam1_1.query_start != nam2_1.query_start) return nam1_1.query_start < nam2_1.query_start;
        if (nam1_1.ref_end != nam2_1.ref_end) return nam1_1.ref_end < nam2_1.ref_end;
        if (nam1_1.ref_start != nam2_1.ref_start) return nam1_1.ref_start < nam2_1.ref_start;
        if (nam1_2.score != nam2_2.score) return nam1_2.score > nam2_2.score;
        if (nam1_2.is_rc != nam2_2.is_rc) return !nam1_2.is_rc;
        if (nam1_2.query_end != nam2_2.query_end) return nam1_2.query_end < nam2_2.query_end;
        if (nam1_2.query_start != nam2_2.query_start) return nam1_2.query_start < nam2_2.query_start;
        if (nam1_2.ref_end != nam2_2.ref_end) return nam1_2.ref_end < nam2_2.ref_end;
        return nam1_2.ref_start < nam2_2.ref_start;

    });

    return;
}

#define TOP_K 40

__device__ void heapify_down(gpu_NamPair heap[], int size, int i) {
    while (2 * i + 1 < size) {
        int smallest = i;
        int l = 2 * i + 1;
        int r = 2 * i + 2;

        if (l < size && heap[l].score < heap[smallest].score)
            smallest = l;
        if (r < size && heap[r].score < heap[smallest].score)
            smallest = r;

        if (smallest == i) break;

        gpu_NamPair tmp = heap[i];
        heap[i] = heap[smallest];
        heap[smallest] = tmp;

        i = smallest;
    }
}

__device__ void heapify_up(gpu_NamPair heap[], int i) {
    while (i > 0) {
        int parent = (i - 1) / 2;
        if (heap[parent].score <= heap[i].score) break;

        gpu_NamPair tmp = heap[i];
        heap[i] = heap[parent];
        heap[parent] = tmp;

        i = parent;
    }
}

__device__ void maintain_top_k(gpu_NamPair heap[], int *heap_size, gpu_NamPair new_pair) {
    if (*heap_size < TOP_K) {
        heap[*heap_size] = new_pair;
        heapify_up(heap, *heap_size);
        (*heap_size)++;
    } else if (new_pair.score > heap[0].score) {
        heap[0] = new_pair;
        heapify_down(heap, *heap_size, 0);
    }
}

__device__ void get_best_scoring_nam_pairs_sort3(
    my_vector<gpu_NamPair>& joint_nam_scores,
    my_vector<Nam>& nams1,
    my_vector<Nam>& nams2,
    float mu_f,
    float sigma_f,
    int max_tries,
    int tid
) {
    int mu = (int)mu_f;
    int sigma = (int)sigma_f;
    int nams1_len = nams1.size();
    int nams2_len = nams2.size();
    my_vector<bool> added_n1(nams1_len);
    my_vector<bool> added_n2(nams2_len);
    for(int i = 0; i < nams1_len; i++) added_n1.push_back(false);
    for(int i = 0; i < nams2_len; i++) added_n2.push_back(false);

    // find is_rc split pos
    int mid_pos1 = nams1.size();
    for (int i = 0; i < nams1.size(); i++) {
        if (nams1[i].is_rc == 1) {
            mid_pos1 = i;
            break;
        }
    }
    int mid_pos2 = nams2.size();
    for (int i = 0; i < nams2.size(); i++) {
        if (nams2[i].is_rc == 1) {
            mid_pos2 = i;
            break;
        }
    }

    gpu_NamPair heap[TOP_K];
    int heap_size = 0;


    int pre_ref_id, p1, p2, pos1, pos2, best_nam2_n_hits;

    int best_joint_hits = 0;
    pos1 = 0;
    pos2 = mid_pos2;
    nams1_len = mid_pos1;
    nams2_len = nams2.size();
    for (int ref_id = 0; ref_id < 3000; ref_id++) {
        while (pos1 < nams1_len && nams1[pos1].ref_id < ref_id) pos1++;
        while (pos2 < nams2_len && nams2[pos2].ref_id < ref_id) pos2++;
        int end1 = pos1, end2 = pos2;
        while (end1 < nams1_len && nams1[end1].ref_id == ref_id) end1++;
        while (end2 < nams2_len && nams2[end2].ref_id == ref_id) end2++;
        if (pos1 == nams1_len || pos2 == nams2_len) break;
        best_nam2_n_hits = -1;
        for (int i = pos2; i < end2; i++) best_nam2_n_hits = my_max(best_nam2_n_hits, nams2[i].n_hits);
        int round_size = 0;
        for (int i = pos1 + 1; i < end1; i++) {
            assert(nams1[i].n_hits <= nams1[i - 1].n_hits);
        }
        for (int i = pos1; i < end1; i++) {
            const Nam &nam1 = nams1[i];
            int round_best_score = nam1.n_hits + best_nam2_n_hits;
            if (round_best_score < best_joint_hits / 2) break;
            if (heap_size == TOP_K && round_best_score < heap[0].score) break;
            int val1 = my_max(0, nam1.ref_start - nam1.query_start);
            int l_pos = pos2, r_pos = end2 - 1, ans_pos = end2;
            while (l_pos <= r_pos) {
                int mid_pos = (l_pos + r_pos) / 2;
                int val2 = my_max(0, nams2[mid_pos].ref_start - nams2[mid_pos].query_start);
                if (val2 >= val1) {
                    ans_pos = mid_pos;
                    r_pos = mid_pos - 1;
                } else {
                    l_pos = mid_pos + 1;
                }
            }
            for (int j = ans_pos; j < end2; j++) {
                const Nam &nam2 = nams2[j];
                int val2 = my_max(0, nams2[j].ref_start - nams2[j].query_start);
                //assert(nam1.ref_id == ref_id && nam1.ref_id == nam2.ref_id && nam1.is_rc == 0 && nam2.is_rc == 1 && val2 >= val1);
                if (val2 >= val1 + mu + 10 * sigma) break;
                int joint_hits = nam1.n_hits + nam2.n_hits;
                if (joint_hits < best_joint_hits / 2) continue;
                //bool res = gpu_is_proper_nam_pair(nam1, nam2, mu, sigma);
                //if (res == false) continue;
                //joint_nam_scores.push_back(gpu_NamPair{joint_hits, &nams1, &nams2, i, j});
                maintain_top_k(heap, &heap_size, gpu_NamPair{joint_hits, &nams1, &nams2, i, j});
                added_n1[i] = 1;
                added_n2[j] = 1;
                best_joint_hits = my_max(joint_hits, best_joint_hits);
                round_size++;
            }
        }
    }

    pos1 = mid_pos1;
    pos2 = 0;
    nams1_len = nams1.size();
    nams2_len = mid_pos2;
    for (int ref_id = 0; ref_id < 3000; ref_id++) {
        while (pos1 < nams1_len && nams1[pos1].ref_id < ref_id) pos1++;
        while (pos2 < nams2_len && nams2[pos2].ref_id < ref_id) pos2++;
        int end1 = pos1, end2 = pos2;
        while (end1 < nams1_len && nams1[end1].ref_id == ref_id) end1++;
        while (end2 < nams2_len && nams2[end2].ref_id == ref_id) end2++;
        if (pos1 == nams1_len || pos2 == nams2_len) break;
        best_nam2_n_hits = -1;
        for (int i = pos2; i < end2; i++) best_nam2_n_hits = my_max(best_nam2_n_hits, nams2[i].n_hits);
        int round_size = 0;
        for (int i = pos1 + 1; i < end1; i++) {
            assert(nams1[i].n_hits <= nams1[i - 1].n_hits);
        }
        for (int i = pos1; i < end1; i++) {
            const Nam &nam1 = nams1[i];
            int round_best_score = nam1.n_hits + best_nam2_n_hits;
            if (round_best_score < best_joint_hits / 2) break;
            if (heap_size == TOP_K && round_best_score < heap[0].score) break;
            int val1 = my_max(0, nam1.ref_start - nam1.query_start);
            int l_pos = pos2, r_pos = end2 - 1, ans_pos = end2;
            while (l_pos <= r_pos) {
                int mid_pos = (l_pos + r_pos) / 2;
                int val2 = my_max(0, nams2[mid_pos].ref_start - nams2[mid_pos].query_start);
                if (val2 > val1 - (mu + 10 * sigma)) {
                    ans_pos = mid_pos;
                    r_pos = mid_pos - 1;
                } else {
                    l_pos = mid_pos + 1;
                }
            }
            for (int j = ans_pos; j < end2; j++) {
                const Nam &nam2 = nams2[j];
                int val2 = my_max(0, nams2[j].ref_start - nams2[j].query_start);
                //assert(nam1.ref_id == ref_id && nam1.ref_id == nam2.ref_id && nam1.is_rc == 1 && nam2.is_rc == 0 && val2 > val1 - (mu + 10 * sigma));
                if (val2 > val1) break;
                int joint_hits = nam1.n_hits + nam2.n_hits;
                if (joint_hits < best_joint_hits / 2) continue;
                //bool res = gpu_is_proper_nam_pair(nam1, nam2, mu, sigma);
                //if (res == false) continue;
                //joint_nam_scores.push_back(gpu_NamPair{joint_hits, &nams1, &nams2, i, j});
                maintain_top_k(heap, &heap_size, gpu_NamPair{joint_hits, &nams1, &nams2, i, j});
                added_n1[i] = 1;
                added_n2[j] = 1;
                best_joint_hits = my_max(joint_hits, best_joint_hits);
                round_size++;
            }
        }
    }

    // Find high-scoring R1 NAMs that are not part of a proper pair
    int best_joint_hits1 = best_joint_hits;
    if (best_joint_hits1 == 0) {
        for (int i = 0; i < nams1.size(); i++) {
            best_joint_hits1 = my_max(best_joint_hits1, nams1[i].n_hits);
        }
    }
    int now_cnt = 0;
    pre_ref_id = nams1[0].ref_id + 3000 * nams1[0].is_rc;
    for(int i = 0; i < nams1.size(); i++) {
        //for(int i = 0; i < my_min(nams1.size(), max_tries); i++) {
        int ref_id = nams1[i].ref_id + 3000 * nams1[i].is_rc;
        if (ref_id == pre_ref_id) now_cnt++;
        else {
            now_cnt = 1;
            pre_ref_id = ref_id;
        }
        if (now_cnt > max_tries) continue;
        Nam nam1 = nams1[i];
        if (nam1.n_hits < best_joint_hits1 / 2) {
            //break;
            continue;
        }
        if (added_n1[i]) {
            continue;
        }
        joint_nam_scores.push_back(gpu_NamPair{nam1.n_hits, &nams1, &nams2, i, -1});
        //maintain_top_k(heap, &heap_size, gpu_NamPair{nam1.n_hits, &nams1, &nams2, i, -1});
    }

    // Find high-scoring R2 NAMs that are not part of a proper pair
    int best_joint_hits2 = best_joint_hits;
    if (best_joint_hits2 == 0) {
        for (int i = 0; i < nams2.size(); i++) {
            best_joint_hits2 = my_max(best_joint_hits2, nams2[i].n_hits);
        }
    }
    for(int i = 0; i < nams2.size(); i++) {
        //for(int i = 0; i < my_min(nams2.size(), max_tries); i++) {
        Nam nam2 = nams2[i];
        if (nam2.n_hits < best_joint_hits2 / 2) {
            continue;
        }
        if (added_n2[i]) {
            continue;
        }
        //joint_nam_scores.push_back(gpu_NamPair{nam2.n_hits, &nams1, &nams2, -1, i});
        maintain_top_k(heap, &heap_size, gpu_NamPair{nam2.n_hits, &nams1, &nams2, -1, i});
    }

    for (int i = 0; i < heap_size; i++) joint_nam_scores.push_back(heap[i]);

    quick_sort_iterative(&(joint_nam_scores[0]), 0, joint_nam_scores.size() - 1, [](const gpu_NamPair &n1, const gpu_NamPair &n2) {
        //if (n1.score != n2.score) return n1.score > n2.score;
        //if (n1.nam1.score != n2.nam1.score) return n1.nam1.score > n2.nam1.score;
        //if (n1.nam1.is_rc != n2.nam1.is_rc) return !n1.nam1.is_rc;  // false < true
        //if (n1.nam1.query_end != n2.nam1.query_end) return n1.nam1.query_end < n2.nam1.query_end;
        //if (n1.nam1.query_start != n2.nam1.query_start) return n1.nam1.query_start < n2.nam1.query_start;
        //if (n1.nam1.ref_end != n2.nam1.ref_end) return n1.nam1.ref_end < n2.nam1.ref_end;
        //if (n1.nam1.ref_start != n2.nam1.ref_start) return n1.nam1.ref_start < n2.nam1.ref_start;
        //if (n1.nam2.score != n2.nam2.score) return n1.nam2.score > n2.nam2.score;
        //if (n1.nam2.is_rc != n2.nam2.is_rc) return !n1.nam2.is_rc;
        //if (n1.nam2.query_end != n2.nam2.query_end) return n1.nam2.query_end < n2.nam2.query_end;
        //if (n1.nam2.query_start != n2.nam2.query_start) return n1.nam2.query_start < n2.nam2.query_start;
        //if (n1.nam2.ref_end != n2.nam2.ref_end) return n1.nam2.ref_end < n2.nam2.ref_end;
        //return n1.nam2.ref_start < n2.nam2.ref_start;

        //return n1.score > n2.score;

        Nam dummy_nam;
        dummy_nam.ref_start = -1;
        Nam nam1_1 = n1.i1 == -1 ? dummy_nam : (*n1.nams1)[n1.i1];
        Nam nam1_2 = n1.i2 == -1 ? dummy_nam : (*n1.nams2)[n1.i2];
        Nam nam2_1 = n2.i1 == -1 ? dummy_nam : (*n2.nams1)[n2.i1];
        Nam nam2_2 = n2.i2 == -1 ? dummy_nam : (*n2.nams2)[n2.i2];

        if (n1.score != n2.score) return n1.score > n2.score;
        if (nam1_1.score != nam2_1.score) return nam1_1.score > nam2_1.score;
        if (nam1_1.is_rc != nam2_1.is_rc) return !nam1_1.is_rc;  // false < true
        if (nam1_1.query_end != nam2_1.query_end) return nam1_1.query_end < nam2_1.query_end;
        if (nam1_1.query_start != nam2_1.query_start) return nam1_1.query_start < nam2_1.query_start;
        if (nam1_1.ref_end != nam2_1.ref_end) return nam1_1.ref_end < nam2_1.ref_end;
        if (nam1_1.ref_start != nam2_1.ref_start) return nam1_1.ref_start < nam2_1.ref_start;
        if (nam1_2.score != nam2_2.score) return nam1_2.score > nam2_2.score;
        if (nam1_2.is_rc != nam2_2.is_rc) return !nam1_2.is_rc;
        if (nam1_2.query_end != nam2_2.query_end) return nam1_2.query_end < nam2_2.query_end;
        if (nam1_2.query_start != nam2_2.query_start) return nam1_2.query_start < nam2_2.query_start;
        if (nam1_2.ref_end != nam2_2.ref_end) return nam1_2.ref_end < nam2_2.ref_end;
        return nam1_2.ref_start < nam2_2.ref_start;

    });


    return;
}

__device__ void gpu_get_best_scoring_nam_pairs_check(
    my_vector<gpu_NamPair>& joint_nam_scores,
    my_vector<Nam>& nams1,
    my_vector<Nam>& nams2,
    float mu,
    float sigma,
    int max_tries,
    int tid
) {
    int nams1_len = nams1.size();
    int nams2_len = nams2.size();
    my_vector<bool> added_n1(nams1_len);
    my_vector<bool> added_n2(nams2_len);
    for(int i = 0; i < nams1_len; i++) added_n1.push_back(false);
    for(int i = 0; i < nams2_len; i++) added_n2.push_back(false);

    int best_joint_hits = 0;
    for (int i = 0; i < nams1_len; i++) {
        const Nam &nam1 = nams1[i];
        for (int j = 0; j < nams2_len; j++) {
            const Nam &nam2 = nams2[j];
            int joint_hits = nam1.n_hits + nam2.n_hits;
            //            if (joint_hits < 0.5 * best_joint_hits || joint_nam_scores.size() > max_tries * 2) {
            if (joint_hits < best_joint_hits / 2) {
                break;
            }
            if (gpu_is_proper_nam_pair(nam1, nam2, mu, sigma)) {
                joint_nam_scores.push_back(gpu_NamPair{joint_hits, &nams1, &nams2, i, j});
                added_n1[i] = 1;
                added_n2[j] = 1;
                best_joint_hits = my_max(joint_hits, best_joint_hits);
            }
        }
        //        if (joint_nam_scores.size() > max_tries * 2) break;
    }

    // Find high-scoring R1 NAMs that are not part of a proper pair
    Nam dummy_nam;
    dummy_nam.ref_start = -1;
    int best_joint_hits1 = best_joint_hits > 0 ? best_joint_hits : nams1[0].n_hits;
    //    for(int i = 0; i < my_min(nams1.size(), max_tries); i++) {
    for(int i = 0; i < nams1.size(); i++) {
        Nam nam1 = nams1[i];
        if (nam1.n_hits < best_joint_hits1 / 2) {
            break;
        }
        if (added_n1[i]) {
            continue;
        }
        joint_nam_scores.push_back(gpu_NamPair{nam1.n_hits, &nams1, &nams2, i, -1});
    }

    // Find high-scoring R2 NAMs that are not part of a proper pair
    int best_joint_hits2 = best_joint_hits > 0 ? best_joint_hits : nams2[0].n_hits;
    //    for(int i = 0; i < my_min(nams2.size(), max_tries); i++) {
    for(int i = 0; i < nams2.size(); i++) {
        Nam nam2 = nams2[i];
        if (nam2.n_hits < best_joint_hits2 / 2) {
            break;
        }
        if (added_n2[i]) {
            continue;
        }
        joint_nam_scores.push_back(gpu_NamPair{nam2.n_hits, &nams1, &nams2, -1, i});
    }

    quick_sort_iterative(&(joint_nam_scores[0]), 0, joint_nam_scores.size() - 1, [](const gpu_NamPair &n1, const gpu_NamPair &n2) {
        if (n1.score != n2.score) return n1.score > n2.score;
        Nam dummy_nam;
        dummy_nam.ref_start = -1;
        Nam nam1_1 = n1.i1 == -1 ? dummy_nam : (*n1.nams1)[n1.i1];
        Nam nam1_2 = n1.i2 == -1 ? dummy_nam : (*n1.nams2)[n1.i2];
        Nam nam2_1 = n2.i1 == -1 ? dummy_nam : (*n2.nams1)[n2.i1];
        Nam nam2_2 = n2.i2 == -1 ? dummy_nam : (*n2.nams2)[n2.i2];

        if (nam1_1.score != nam2_1.score) return nam1_1.score > nam2_1.score;
        if (nam1_1.is_rc != nam2_1.is_rc) return !nam1_1.is_rc;  // false < true
        if (nam1_1.query_end != nam2_1.query_end) return nam1_1.query_end < nam2_1.query_end;
        if (nam1_1.query_start != nam2_1.query_start) return nam1_1.query_start < nam2_1.query_start;
        if (nam1_1.ref_end != nam2_1.ref_end) return nam1_1.ref_end < nam2_1.ref_end;
        if (nam1_1.ref_start != nam2_1.ref_start) return nam1_1.ref_start < nam2_1.ref_start;
        if (nam1_2.score != nam2_2.score) return nam1_2.score > nam2_2.score;
        if (nam1_2.is_rc != nam2_2.is_rc) return !nam1_2.is_rc;
        if (nam1_2.query_end != nam2_2.query_end) return nam1_2.query_end < nam2_2.query_end;
        if (nam1_2.query_start != nam2_2.query_start) return nam1_2.query_start < nam2_2.query_start;
        if (nam1_2.ref_end != nam2_2.ref_end) return nam1_2.ref_end < nam2_2.ref_end;
        return nam1_2.ref_start < nam2_2.ref_start;

    });

    return;
}

__device__ void gpu_get_best_scoring_nam_pairs(
    my_vector<gpu_NamPair>& joint_nam_scores,
    my_vector<Nam>& nams1,
    my_vector<Nam>& nams2,
    float mu,
    float sigma,
    int max_tries,
    int tid
) {
    int nams1_len = nams1.size();
    int nams2_len = nams2.size();
    my_vector<bool> added_n1(nams1_len);
    my_vector<bool> added_n2(nams2_len);
    for(int i = 0; i < nams1_len; i++) added_n1.push_back(false);
    for(int i = 0; i < nams2_len; i++) added_n2.push_back(false);

    int best_joint_hits = 0;
    for (int i = 0; i < nams1_len; i++) {
        const Nam &nam1 = nams1[i];
        for (int j = 0; j < nams2_len; j++) {
            const Nam &nam2 = nams2[j];
            int joint_hits = nam1.n_hits + nam2.n_hits;
            if (joint_hits < 0.5 * best_joint_hits || joint_nam_scores.size() > max_tries * 2) {
                break;
            }
            if (gpu_is_proper_nam_pair(nam1, nam2, mu, sigma)) {
                joint_nam_scores.push_back(gpu_NamPair{joint_hits, &nams1, &nams2, i, j});
                added_n1[i] = 1;
                added_n2[j] = 1;
                best_joint_hits = my_max(joint_hits, best_joint_hits);
            }
        }
        if (joint_nam_scores.size() > max_tries * 2) break;
    }

    // Find high-scoring R1 NAMs that are not part of a proper pair
    Nam dummy_nam;
    dummy_nam.ref_start = -1;
    int best_joint_hits1 = best_joint_hits > 0 ? best_joint_hits : nams1[0].n_hits;
    for(int i = 0; i < my_min(nams1.size(), max_tries); i++) {
//    for(int i = 0; i < nams1.size(); i++) {
        Nam nam1 = nams1[i];
        if (nam1.n_hits < best_joint_hits1 / 2) {
            break;
        }
        if (added_n1[i]) {
            continue;
        }
        joint_nam_scores.push_back(gpu_NamPair{nam1.n_hits, &nams1, &nams2, i, -1});
    }

    // Find high-scoring R2 NAMs that are not part of a proper pair
    int best_joint_hits2 = best_joint_hits > 0 ? best_joint_hits : nams2[0].n_hits;
    for(int i = 0; i < my_min(nams2.size(), max_tries); i++) {
//    for(int i = 0; i < nams2.size(); i++) {
        Nam nam2 = nams2[i];
        if (nam2.n_hits < best_joint_hits2 / 2) {
            break;
        }
        if (added_n2[i]) {
            continue;
        }
        joint_nam_scores.push_back(gpu_NamPair{nam2.n_hits, &nams1, &nams2, -1, i});
    }

    quick_sort_iterative(&(joint_nam_scores[0]), 0, joint_nam_scores.size() - 1, [](const gpu_NamPair &n1, const gpu_NamPair &n2) {
        if (n1.score != n2.score) return n1.score > n2.score;
    });

    return;
}

__device__ static unsigned char gpu_revcomp_table[256] = {
    'N', 'N', 'N', 'N',  'N', 'N', 'N', 'N',  'N', 'N', 'N', 'N',  'N', 'N', 'N', 'N',
    'N', 'N', 'N', 'N',  'N', 'N', 'N', 'N',  'N', 'N', 'N', 'N',  'N', 'N', 'N', 'N',
    'N', 'N', 'N', 'N',  'N', 'N', 'N', 'N',  'N', 'N', 'N', 'N',  'N', 'N', 'N', 'N',
    'N', 'N', 'N', 'N',  'N', 'N', 'N', 'N',  'N', 'N', 'N', 'N',  'N', 'N', 'N', 'N',
    'N', 'T', 'N', 'G',  'N', 'N', 'N', 'C',  'N', 'N', 'N', 'N',  'N', 'N', 'N', 'N',
    'N', 'N', 'N', 'N',  'A', 'A', 'N', 'N',  'N', 'N', 'N', 'N',  'N', 'N', 'N', 'N',
    'N', 'T', 'N', 'G',  'N', 'N', 'N', 'C',  'N', 'N', 'N', 'N',  'N', 'N', 'N', 'N',
    'N', 'N', 'N', 'N',  'A', 'A', 'N', 'N',  'N', 'N', 'N', 'N',  'N', 'N', 'N', 'N',
    'N', 'N', 'N', 'N',  'N', 'N', 'N', 'N',  'N', 'N', 'N', 'N',  'N', 'N', 'N', 'N',
    'N', 'N', 'N', 'N',  'N', 'N', 'N', 'N',  'N', 'N', 'N', 'N',  'N', 'N', 'N', 'N',
    'N', 'N', 'N', 'N',  'N', 'N', 'N', 'N',  'N', 'N', 'N', 'N',  'N', 'N', 'N', 'N',
    'N', 'N', 'N', 'N',  'N', 'N', 'N', 'N',  'N', 'N', 'N', 'N',  'N', 'N', 'N', 'N',
    'N', 'N', 'N', 'N',  'N', 'N', 'N', 'N',  'N', 'N', 'N', 'N',  'N', 'N', 'N', 'N',
    'N', 'N', 'N', 'N',  'N', 'N', 'N', 'N',  'N', 'N', 'N', 'N',  'N', 'N', 'N', 'N',
    'N', 'N', 'N', 'N',  'N', 'N', 'N', 'N',  'N', 'N', 'N', 'N',  'N', 'N', 'N', 'N',
    'N', 'N', 'N', 'N',  'N', 'N', 'N', 'N',  'N', 'N', 'N', 'N',  'N', 'N', 'N', 'N'
};

__device__ void align_PE_part0(
    GPUAlignTmpRes& align_tmp_res,
    const AlignmentParameters& aligner_parameters,
    my_vector<Nam>& nams1,
    my_vector<Nam>& nams2,
    char* seq1, int seq_len1,
    char* seq2, int seq_len2,
    int k,
    const GPUReferences& references,
    float dropoff,
    GPUInsertSizeDistribution& isize_est,
    unsigned max_tries,
    size_t max_secondary
) {
    assert(nams1.empty() && nams2.empty());
    align_tmp_res.type = 0;
    return;
}

__device__ void align_PE_part12(
    GPUAlignTmpRes& align_tmp_res,
    const AlignmentParameters& aligner_parameters,
    my_vector<Nam>& nams1,
    my_vector<Nam>& nams2,
    char* seq1, char* rc1, int seq_len1,
    char* seq2, char* rc2, int seq_len2,
    int k,
    const GPUReferences& references,
    float dropoff,
    GPUInsertSizeDistribution& isize_est,
    unsigned max_tries,
    size_t max_secondary,
    int type,
    int read_id
) {
    //assert(!nams1.empty() && nams2.empty());
    const auto mu = isize_est.mu;
    const auto sigma = isize_est.sigma;
    GPURead read1{seq1, rc1, seq_len1};
    GPURead read2{seq2, rc2, seq_len2};
    double secondary_dropoff = 2 * aligner_parameters.mismatch + aligner_parameters.gap_open;
//    align_tmp_res.type = 1;
    gpu_rescue_read_part(
			type, align_tmp_res, type == 1 ? read2 : read1, type == 1 ? read1 : read2, aligner_parameters, references, type == 1 ? nams1 : nams2, max_tries, dropoff, k, mu,
        sigma, max_secondary, secondary_dropoff, type == 1 ? false : true
    );
    for (size_t j = 0; j < align_tmp_res.todo_nams.size(); j += 2) {
        assert(align_tmp_res.is_extend_seed[j]);
        if (align_tmp_res.type == 1)
            assert(align_tmp_res.is_read1[j]);
        else
            assert(!align_tmp_res.is_read1[j]);
        if (!align_tmp_res.done_align[j]) {
            gpu_part2_extend_seed_get_str(
                align_tmp_res, j, read1, read2, references, read_id
            );
        }
        assert(!align_tmp_res.is_extend_seed[j + 1]);
        if (align_tmp_res.type == 1)
            assert(!align_tmp_res.is_read1[j + 1]);
        else
            assert(align_tmp_res.is_read1[j + 1]);
        if (!align_tmp_res.done_align[j + 1]) {
            gpu_part2_rescue_mate_get_str(
                align_tmp_res, j + 1, read1, read2, references, mu, sigma, read_id
            );
        }
    }
    return;
}


__device__ void align_PE_part3(
    GPUAlignTmpRes& align_tmp_res,
    const AlignmentParameters& aligner_parameters,
    my_vector<Nam>& nams1,
    my_vector<Nam>& nams2,
    char* seq1, char* rc1, int seq_len1,
    char* seq2, char* rc2, int seq_len2,
    int k,
    const GPUReferences& references,
    float dropoff,
    GPUInsertSizeDistribution& isize_est,
    unsigned max_tries,
    size_t max_secondary,
    int read_id
) {
    assert(!nams1.empty() && !nams2.empty());
    const auto mu = isize_est.mu;
    const auto sigma = isize_est.sigma;
    GPURead read1{seq1, rc1, seq_len1};
    GPURead read2{seq2, rc2, seq_len2};
    double secondary_dropoff = 2 * aligner_parameters.mismatch + aligner_parameters.gap_open;

    // Deal with the typical case that both reads map uniquely and form a proper pair
    assert(gpu_top_dropoff(nams1) < dropoff && gpu_top_dropoff(nams2) < dropoff && gpu_is_proper_nam_pair(nams1[0], nams2[0], mu, sigma));
//    align_tmp_res.type = 3;
    Nam n_max1 = nams1[0];
    Nam n_max2 = nams2[0];

    bool consistent_nam1 = gpu_reverse_nam_if_needed(n_max1, read1, references, k);
    bool consistent_nam2 = gpu_reverse_nam_if_needed(n_max2, read2, references, k);

    align_tmp_res.is_read1.push_back(true);
    bool gapped1 = gpu_extend_seed_part(align_tmp_res, aligner_parameters, n_max1, references, read1, consistent_nam1);


    align_tmp_res.is_read1.push_back(false);
    bool gapped2 = gpu_extend_seed_part(align_tmp_res, aligner_parameters, n_max2, references, read2, consistent_nam2);

    int mapq1 = gpu_get_mapq(nams1, n_max1);
    int mapq2 = gpu_get_mapq(nams2, n_max2);
    align_tmp_res.mapq1 = mapq1;
    align_tmp_res.mapq2 = mapq2;

    assert(align_tmp_res.is_extend_seed[0]);
    assert(align_tmp_res.is_read1[0]);
    if (!align_tmp_res.done_align[0]) {
        gpu_part2_extend_seed_get_str(
            align_tmp_res, 0, read1, read2, references, read_id
        );
    }
    assert(align_tmp_res.is_extend_seed[1]);
    assert(!align_tmp_res.is_read1[1]);
    if (!align_tmp_res.done_align[1]) {
        gpu_part2_extend_seed_get_str(
            align_tmp_res, 1, read1, read2, references, read_id
        );
    }
    return;

}

__device__ void align_PE_part4(
    GPUAlignTmpRes& align_tmp_res,
    const AlignmentParameters& aligner_parameters,
    my_vector<Nam>& nams1,
    my_vector<Nam>& nams2,
    char* seq1, char* rc1, int seq_len1,
    char* seq2, char* rc2, int seq_len2,
    int k,
    const GPUReferences& references,
    float dropoff,
    GPUInsertSizeDistribution& isize_est,
    int max_tries,
    size_t max_secondary,
    int tid,
    int read_id
) {
    assert(!nams1.empty() && !nams2.empty());

    const auto mu = isize_est.mu;
    const auto sigma = isize_est.sigma;
    GPURead read1{seq1, rc1, seq_len1};
    GPURead read2{seq2, rc2, seq_len2};
    double secondary_dropoff = 2 * aligner_parameters.mismatch + aligner_parameters.gap_open;

    my_vector<gpu_NamPair> joint_nam_scores(nams1.size() + nams2.size());
    gpu_get_best_scoring_nam_pairs(joint_nam_scores, nams1, nams2, mu, sigma, max_tries, tid);

    if (joint_nam_scores.size() > max_tries) joint_nam_scores.length = max_tries;
    int nams1_len = nams1.size();
    int nams2_len = nams2.size();
    my_vector<bool> is_aligned1(nams1_len + 1);
    my_vector<bool> is_aligned2(nams2_len + 1);
    for (int i = 0; i <= nams1_len; i++) is_aligned1.push_back(false);
    for (int i = 0; i <= nams2_len; i++) is_aligned2.push_back(false);

    {
        Nam n1_max = nams1[0];
        bool consistent_nam1 = gpu_reverse_nam_if_needed(n1_max, read1, references, k);
        align_tmp_res.is_read1.push_back(true);
        bool gapped1 = gpu_extend_seed_part(align_tmp_res, aligner_parameters, n1_max, references, read1, consistent_nam1);
        is_aligned1[0] = 1;

        Nam n2_max = nams2[0];
        bool consistent_nam2 = gpu_reverse_nam_if_needed(n2_max, read2, references, k);
        align_tmp_res.is_read1.push_back(false);
        bool gapped2 = gpu_extend_seed_part(align_tmp_res, aligner_parameters, n2_max, references, read2, consistent_nam2);
        is_aligned2[0] = 1;
    }

    Nam dummy_nam;
    dummy_nam.ref_start = -1;

    // Turn pairs of high-scoring NAMs into pairs of alignments
    int high_scores_size = 0;
    double max_score = joint_nam_scores[0].score;
    align_tmp_res.type4_loop_size = 0;
    for(int i = 0; i < joint_nam_scores.size(); i++) {
        double score_ = joint_nam_scores[i].score;
        int id1 = joint_nam_scores[i].i1 == -1 ? nams1_len : joint_nam_scores[i].i1;
        int id2 = joint_nam_scores[i].i2 == -1 ? nams2_len : joint_nam_scores[i].i2;
        Nam n1 = joint_nam_scores[i].i1 == -1 ? dummy_nam : nams1[joint_nam_scores[i].i1];
        Nam n2 = joint_nam_scores[i].i2 == -1 ? dummy_nam : nams2[joint_nam_scores[i].i2];

        float score_dropoff = (float) score_ / max_score;
        if (high_scores_size >= max_tries || score_dropoff < dropoff) {
            break;
        }

        align_tmp_res.type4_nams.push_back(n1);
        align_tmp_res.type4_nams.push_back(n2);
        align_tmp_res.type4_loop_size++;

        if (n1.ref_start >= 0) {
            if (is_aligned1[id1] == 1) {

            } else {
                bool consistent_nam = gpu_reverse_nam_if_needed(n1, read1, references, k);
                align_tmp_res.is_read1.push_back(true);
                bool gapped = gpu_extend_seed_part(align_tmp_res, aligner_parameters, n1, references, read1, consistent_nam);
                is_aligned1[id1] = 1;
            }
        } else {
            gpu_reverse_nam_if_needed(n2, read2, references, k);
            align_tmp_res.is_read1.push_back(true);
            bool is_unaligned = gpu_rescue_mate_part(align_tmp_res, aligner_parameters, n2, references, read1, mu, sigma, k);
        }

        if (n2.ref_start >= 0) {
            if (is_aligned2[id2] == 1) {

            } else {
                bool consistent_nam = gpu_reverse_nam_if_needed(n2, read2, references, k);
                align_tmp_res.is_read1.push_back(false);
                bool gapped = gpu_extend_seed_part(align_tmp_res, aligner_parameters, n2, references, read2, consistent_nam);
                is_aligned2[id2] = 1;
            }
        } else {
            gpu_reverse_nam_if_needed(n1, read1, references, k);
            align_tmp_res.is_read1.push_back(false);
            bool is_unaligned = gpu_rescue_mate_part(align_tmp_res, aligner_parameters, n1, references, read2, mu, sigma, k);
        }
        high_scores_size++;
    }

    for (size_t j = 0; j < align_tmp_res.todo_nams.size(); j++) {
        if (!align_tmp_res.done_align[j]) {
            if (align_tmp_res.is_extend_seed[j]) {
                gpu_part2_extend_seed_get_str(
                    align_tmp_res, j, read1, read2, references, read_id
                );
            } else {
                gpu_part2_rescue_mate_get_str(
                    align_tmp_res, j, read1, read2, references, mu, sigma, read_id
                );
            }
        }
    }
    return;
}

#define BLOCK_SIZE 32


__device__ void check_hits(my_vector<my_pair<int, Hit>> &hits_per_ref) {
    // check if sort is correct
    if (hits_per_ref.size() < 2) return;
    for(int i = 0; i < hits_per_ref.size() - 1; i++) {
        //        if(hits_per_ref[i].first > hits_per_ref[i + 1].first) {
        //            printf("sort error [%d,%d] [%d,%d]\n", hits_per_ref[i].first, hits_per_ref[i].second.query_start, hits_per_ref[i + 1].first, hits_per_ref[i + 1].second.query_start);
        //            assert(false);
        //        }
        if(hits_per_ref[i].first == hits_per_ref[i + 1].first && hits_per_ref[i].second.query_start > hits_per_ref[i + 1].second.query_start) {
            printf("sort error [%d,%d] [%d,%d]\n", hits_per_ref[i].first, hits_per_ref[i].second.query_start, hits_per_ref[i + 1].first, hits_per_ref[i + 1].second.query_start);
            assert(false);
        }
    }
}

__device__ void check_nams(my_vector<Nam> &nams) {
    // check if sort is correct
    if (nams.size() < 2) return;
    for(int i = 0; i < nams.size() - 1; i++) {
        if(nams[i].score < nams[i + 1].score) {
            printf("sort error [%lf,%d] [%lf,%d]\n", nams[i].score, nams[i].query_end, nams[i + 1].score, nams[i + 1].query_end);
            assert(false);
        }
        //        if(nams[i].score == nams[i + 1].score && nams[i].query_end > nams[i + 1].query_end) {
        //            printf("sort error [%lf,%d] [%lf,%d]\n", nams[i].score, nams[i].query_end, nams[i + 1].score, nams[i + 1].query_end);
        //            assert(false);
        //        }
    }
}

__device__ void sort_hits_single(
    my_vector<my_pair<int, Hit>>& hits_per_ref
) {
    //bubble_sort(&(hits_per_ref[0]), hits_per_ref.size());
    quick_sort(&(hits_per_ref[0]), hits_per_ref.size());
}

__device__ void topk_quick_sort(my_vector<Nam>& nams, int mx_num) {
    if (nams.size() == 0) return;
    //const int MAX_STACK = 64;
    //int left_stack[MAX_STACK];
    //int right_stack[MAX_STACK];
    if (nams.size() < mx_num) mx_num = nams.size();
    my_vector<int>ll(nams.size() * 2);
    my_vector<int>rr(nams.size() * 2);
    int *left_stack = ll.data;
    int *right_stack = rr.data;
    int top = -1;

    int left = 0, right = nams.size() - 1;
    left_stack[++top] = left;
    right_stack[top] = right;

    while (top >= 0) {
        left = left_stack[top];
        right = right_stack[top--];

        if (left >= right) continue;

        Nam pivot = nams[right];
        int i = left - 1;

        for (int j = left; j < right; ++j) {
            if (nams[j].score > pivot.score) {
                ++i;
                Nam tmp = nams[i];
                nams[i] = nams[j];
                nams[j] = tmp;
            }
        }

        Nam tmp = nams[i + 1];
        nams[i + 1] = nams[right];
        nams[right] = tmp;

        int pivot_index = i + 1;

        if (pivot_index > mx_num) {
            left_stack[++top] = left;
            right_stack[top] = pivot_index - 1;
        } else if (pivot_index < mx_num - 1) {
            left_stack[++top] = pivot_index + 1;
            right_stack[top] = right;
        } else {
        }
    }
}


__device__ void sort_nams_single(
    my_vector<Nam>& nams
) {
    //bubble_sort(&(hits_per_ref[0]), hits_per_ref.size());
    quick_sort_iterative(&(nams[0]), 0, nams.size() - 1, [](const Nam &n1, const Nam &n2) {
        //if(n1.score != n2.score) return n1.score > n2.score;
        if(n1.n_hits != n2.n_hits) return n1.n_hits > n2.n_hits;
        if(n1.query_end != n2.query_end) return n1.query_end < n2.query_end;
        if(n1.query_start != n2.query_start) return n1.query_start < n2.query_start;
        if(n1.ref_end != n2.ref_end) return n1.ref_end < n2.ref_end;
        if(n1.ref_start != n2.ref_start) return n1.ref_start < n2.ref_start;
        if(n1.ref_id != n2.ref_id) return n1.ref_id < n2.ref_id;
        return n1.is_rc < n2.is_rc;
    });
}

__device__ void sort_nams_single2(
    my_vector<Nam>& nams
) {
    //quick_sort(&(nams[0]), nams.size());
    quick_sort_iterative(&(nams[0]), 0, nams.size() - 1, [](const Nam &n1, const Nam &n2) {
        if(n1.is_rc != n2.is_rc) return n1.is_rc < n2.is_rc;
        if(n1.ref_id != n2.ref_id) return n1.ref_id < n2.ref_id;
        //if(n1.score != n2.score) return n1.score > n2.score;
        if(n1.n_hits != n2.n_hits) return n1.n_hits > n2.n_hits;
        if(n1.query_end != n2.query_end) return n1.query_end < n2.query_end;
        if(n1.query_start != n2.query_start) return n1.query_start < n2.query_start;
        if(n1.ref_end != n2.ref_end) return n1.ref_end < n2.ref_end;
        if(n1.ref_start != n2.ref_start) return n1.ref_start < n2.ref_start;
        //return n1.is_rc < n2.is_rc;
        return true;
    });


}

__device__ int find_ref_ids(int ref_id, int* head, ref_ids_edge* edges) {
    int key = ref_id % key_mod_val;
    for (int i = head[key]; i != -1; i = edges[i].pre) {
        if (edges[i].ref_id == ref_id) return i;
    }
    return -1;
}

__device__ void sort_nams_get_k(my_vector<Nam>& nams, int mx_num) {
    int limit = mx_num;
    if (limit > nams.size()) limit = nams.size();
    for (int i = 0; i < limit; ++i) {
        int max_idx = i;
        for (int j = i + 1; j < nams.size(); ++j) {
            if (nams[j].score > nams[max_idx].score) {
                max_idx = j;
            }
        }
        if (max_idx != i) {
            Nam tmp = nams[i];
            nams[i] = nams[max_idx];
            nams[max_idx] = tmp;
        }
    }
}


__device__ void sort_nams_by_score(my_vector<Nam>& nams, int mx_num) {
    my_vector<my_pair<int, my_vector<Nam>*>> all_nams;
    int* head = (int*)my_malloc(key_mod_val * sizeof(int));
    my_vector<ref_ids_edge> edges;
    for (int i = 0; i < key_mod_val; i++) head[i] = -1;
    int score_group_num = 0;
    for (int i = 0; i < nams.size(); i++) {
        int score_key = (int)(nams[i].score);
        int score_rank = find_ref_ids(score_key, head, edges.data);
        if (score_rank == -1) {
            score_rank = score_group_num;
            int key = score_key % key_mod_val;
            edges.push_back({head[key], score_key});
            head[key] = score_group_num++;
            my_vector<Nam>* bucket = (my_vector<Nam>*)my_malloc(sizeof(my_vector<Nam>));
            bucket->init();
            all_nams.push_back({score_key, bucket});
        }
        all_nams[score_rank].second->push_back(nams[i]);
    }
    nams.clear();
    quick_sort_iterative(&(all_nams[0]), 0, all_nams.size() - 1,
                         [] (const my_pair<int, my_vector<Nam>*>& a, const my_pair<int, my_vector<Nam>*>& b) {
                             return a.first > b.first;
                         });
    for (int i = 0; i < all_nams.size(); i++) {
        for (int j = 0; j < all_nams[i].second->size(); j++) {
            if (nams.size() == mx_num) break;
            nams.push_back((*all_nams[i].second)[j]);
        }
        all_nams[i].second->release();
        my_free(all_nams[i].second);
    }
    my_free(head);
}


__device__ void sort_hits_by_refid(
    my_vector<my_pair<int, Hit>>& hits_per_ref
) {
    my_vector<my_pair<int, my_vector<Hit>*>> all_hits;
    int *head = (int*)my_malloc(key_mod_val * sizeof(int));
    my_vector<ref_ids_edge> edges;
    for(int i = 0; i < key_mod_val; i++) head[i] = -1;
    int ref_ids_num = 0;
    for(int i = 0; i < hits_per_ref.size(); i++) {
        int ref_id = hits_per_ref[i].first;
        int find_ref_id_rank = find_ref_ids(ref_id, head, edges.data);
        if (find_ref_id_rank == -1) {
            find_ref_id_rank = ref_ids_num;
            int key = ref_id % key_mod_val;
            edges.push_back({head[key], ref_id});
            head[key] = ref_ids_num++;
            my_vector<Hit>* hits = (my_vector<Hit>*)my_malloc(sizeof(my_vector<Hit>));
            hits->init();
            all_hits.push_back({ref_id, hits});
        }
        all_hits[find_ref_id_rank].second->push_back(hits_per_ref[i].second);
    }
    hits_per_ref.clear();
    for(int i = 0; i < all_hits.size(); i++) {
        for(int j = 0; j < all_hits[i].second->size(); j++) {
            hits_per_ref.push_back({all_hits[i].first, (*all_hits[i].second)[j]});
        }
        all_hits[i].second->release();
        my_free(all_hits[i].second);
    }
    my_free(head);
}

__device__ void sort_hits_parallel(
    my_vector<my_pair<int, Hit>>& hits_per_ref,
    int k,
    bool is_revcomp,
    int tid
) {
    if(hits_per_ref.size() == 0) return;
    //int num_hits = hits_per_ref.size();

    //const int items_per_thread = 160;
    //int real_num_hits = items_per_thread * BLOCK_SIZE;
    //if(real_num_hits < num_hits) {
    //    printf("real_num_hits %d num_hits %d\n", real_num_hits, num_hits);
    //}
    //assert(real_num_hits >= num_hits);

    //typedef hipcub::BlockRadixSort<unsigned long long, BLOCK_SIZE, items_per_thread, int> BlockRadixSort;
    //__shared__ typename BlockRadixSort::TempStorage temp_storage;

    //unsigned long long thread_keys[items_per_thread];
    //int thread_indices[items_per_thread];

    //__shared__ int* old_ref_end;
    //__shared__ int* old_query_end;
    //if(tid == 0) {
    //    old_ref_end = (int*)my_malloc(real_num_hits * sizeof(int));
    //    old_query_end = (int*)my_malloc(real_num_hits * sizeof(int));
    //}
    //__syncthreads();

    //for (int i = 0; i < items_per_thread; ++i) {
    //    int idx = tid * items_per_thread + i;
    //    if (idx < num_hits) {
    //        thread_keys[i] = (static_cast<unsigned long long>(hits_per_ref[idx].first) << 48) |
    //                         (static_cast<unsigned long long>(hits_per_ref[idx].second.query_start & 0xFFFF) << 32) |
    //                         (static_cast<unsigned long long>(hits_per_ref[idx].second.ref_start) & 0xFFFFFFFF);
    //        thread_indices[i] = idx;
    //        old_ref_end[idx] = hits_per_ref[idx].second.ref_end;
    //        old_query_end[idx] = hits_per_ref[idx].second.query_end;
    //    } else {
    //        thread_keys[i] = ULLONG_MAX;
    //        thread_indices[i] = -1;
    //        old_ref_end[idx] = 0;
    //        old_query_end[idx] = 0;
    //    }
    //}
    //__syncthreads();

    //BlockRadixSort(temp_storage).Sort(thread_keys, thread_indices);
    //__syncthreads();

    //for (int i = 0; i < items_per_thread; ++i) {
    //    int idx = tid * items_per_thread + i;
    //    if (idx < num_hits) {
    //        hits_per_ref[idx].first = thread_keys[i] >> 48;
    //        hits_per_ref[idx].second.query_start = (thread_keys[i] >> 32) & 0xFFFF;
    //        hits_per_ref[idx].second.ref_start = thread_keys[i] & 0xFFFFFFFF;
    //        hits_per_ref[idx].second.ref_end = old_ref_end[thread_indices[i]];
    //        hits_per_ref[idx].second.query_end = old_query_end[thread_indices[i]];
    //    }
    //}
    //__syncthreads();
    //if(tid == 0) {
    //    my_free(old_ref_end);
    //    my_free(old_query_end);
    //}


}

__device__ size_t my_lower_bound(my_pair<int, Hit>* hits, size_t i_start, size_t i_end, int target) {
    size_t left = i_start, right = i_end;
    while (left < right) {
        size_t mid = left + (right - left) / 2;
        if (hits[mid].second.ref_start < target) {
            left = mid + 1;
        } else {
            right = mid;
        }
    }
    return left;
}

__device__ void salign_merge_hits(
    my_vector<my_pair<int, Hit>>& hits_per_ref,
    int k,
    bool is_revcomp,
    my_vector<Nam>& nams
) {
    if(hits_per_ref.size() == 0) return;
    int ref_num = 0;
    my_vector<int> each_ref_size;
    int pre_ref_id = hits_per_ref[0].first;
    int now_ref_num = 1;
    for(int i = 1; i < hits_per_ref.size(); i++) {
        int ref_id = hits_per_ref[i].first;
        Hit hit = hits_per_ref[i].second;
        if(ref_id != pre_ref_id) {
            ref_num++;
            pre_ref_id = ref_id;
            each_ref_size.push_back(now_ref_num);
            now_ref_num = 1;
        } else {
            now_ref_num++;
        }
    }
    ref_num++;
    each_ref_size.push_back(now_ref_num);
    //int mx_hits_per_ref = 0;
    //for (int i = 0; i < each_ref_size.size(); i++) {
    //    mx_hits_per_ref = my_max(mx_hits_per_ref, each_ref_size[i]);
    //}

    my_vector<Nam> open_nams;
    //(mx_hits_per_ref);

    int now_vec_pos = 0;
    for (int rid = 0; rid < ref_num; rid++) {
        if(rid != 0) now_vec_pos += each_ref_size[rid - 1];
        int ref_id = hits_per_ref[now_vec_pos].first;
        open_nams.clear();
        unsigned int prev_q_start = 0;
        size_t hits_size = each_ref_size[rid];
        my_pair<int, Hit>* hits = &(hits_per_ref[now_vec_pos]);
        for (size_t i = 0; i < hits_size; ) {
            size_t i_start = i;
            size_t i_end = i + 1;
            size_t i_size;
            while(i_end < hits_size && hits[i_end].second.query_start == hits[i].second.query_start) i_end++;
            i = i_end;
            i_size = i_end - i_start;
            //for(int j = 0; j < i_size - 1; j++) {
            //    assert(hits[i_start + j].second.ref_start <= hits[i_start + j + 1].second.ref_start);
            //}
            //quick_sort(&(hits[i_start]), i_size);
            my_vector<bool> is_added(i_size);
            for(size_t j = 0; j < i_size; j++) is_added.push_back(false);
            int query_start = hits[i_start].second.query_start;
            int cnt_done = 0;
            for (int k = 0; k < open_nams.size(); k++) {
                Nam& o = open_nams[k];
                if ( query_start > o.query_end ) continue;
                size_t lower = my_lower_bound(hits, i_start, i_end, o.ref_prev_hit_startpos + 1);
                size_t upper = my_lower_bound(hits, i_start, i_end, o.ref_end + 1);
                for (size_t j = lower; j < upper; j++) {
                    if(is_added[j - i_start]) continue;
                    Hit& h = hits[j].second;
                    {
                        if (o.ref_prev_hit_startpos < h.ref_start && h.ref_start <= o.ref_end) {
                            if ((h.query_end > o.query_end) && (h.ref_end > o.ref_end)) {
                                o.query_end = h.query_end;
                                o.ref_end = h.ref_end;
                                //                        o.previous_query_start = h.query_s;
                                //                        o.previous_ref_start = h.ref_s; // keeping track so that we don't . Can be caused by interleaved repeats.
                                o.query_prev_hit_startpos = h.query_start;
                                o.ref_prev_hit_startpos = h.ref_start;
                                o.n_hits++;
                                //                        o.score += (float)1/ (float)h.count;
                                is_added[j - i_start] = true;
                                cnt_done++;
                                break;
                            } else if ((h.query_end <= o.query_end) && (h.ref_end <= o.ref_end)) {
                                //                        o.previous_query_start = h.query_s;
                                //                        o.previous_ref_start = h.ref_s; // keeping track so that we don't . Can be caused by interleaved repeats.
                                o.query_prev_hit_startpos = h.query_start;
                                o.ref_prev_hit_startpos = h.ref_start;
                                o.n_hits++;
                                //                        o.score += (float)1/ (float)h.count;
                                is_added[j - i_start] = true;
                                cnt_done++;
                                break;
                            }
                        }
                    }
                }
                if(cnt_done == i_size) break;
            }

            // Add the hit to open matches
            for(size_t j = 0; j < i_size; j++) {
                if (!is_added[j]){
                    Nam n;
                    n.query_start = hits[i_start + j].second.query_start;
                    n.query_end = hits[i_start + j].second.query_end;
                    n.ref_start = hits[i_start + j].second.ref_start;
                    n.ref_end = hits[i_start + j].second.ref_end;
                    n.ref_id = ref_id;
                    //                n.previous_query_start = h.query_s;
                    //                n.previous_ref_start = h.ref_s;
                    n.query_prev_hit_startpos = hits[i_start + j].second.query_start;
                    n.ref_prev_hit_startpos = hits[i_start + j].second.ref_start;
                    n.n_hits = 1;
                    n.is_rc = is_revcomp;
                    //                n.score += (float)1 / (float)h.count;
                    open_nams.push_back(n);
                }
            }

            // Only filter if we have advanced at least k nucleotides
            if (query_start > prev_q_start + k) {

                // Output all NAMs from open_matches to final_nams that the current hit have passed
                for (int k = 0; k < open_nams.size(); k++) {
                    Nam& n = open_nams[k];
                    if (n.query_end < query_start) {
                        int n_max_span = my_max(n.query_span(), n.ref_span());
                        int n_min_span = my_min(n.query_span(), n.ref_span());
                        float n_score;
                        n_score = ( 2*n_min_span -  n_max_span) > 0 ? (float) (n.n_hits * ( 2*n_min_span -  n_max_span) ) : 1;   // this is really just n_hits * ( min_span - (offset_in_span) ) );
                        //                        n_score = n.n_hits * n.query_span();
                        n.score = n_score;
                        n.nam_id = nams.size();
                        nams.push_back(n);
                    }
                }

                // Remove all NAMs from open_matches that the current hit have passed
                auto c = query_start;
                int old_open_size = open_nams.size();
                open_nams.clear();
                for (int in = 0; in < old_open_size; ++in) {
                    if (!(open_nams[in].query_end < c)) {
                        open_nams.push_back(open_nams[in]);
                    }
                }
                prev_q_start = query_start;
            }
        }
        // Add all current open_matches to final NAMs
        for (int k = 0; k < open_nams.size(); k++) {
            Nam& n = open_nams[k];
            int n_max_span = my_max(n.query_span(), n.ref_span());
            int n_min_span = my_min(n.query_span(), n.ref_span());
            float n_score;
            n_score = ( 2*n_min_span -  n_max_span) > 0 ? (float) (n.n_hits * ( 2*n_min_span -  n_max_span) ) : 1;   // this is really just n_hits * ( min_span - (offset_in_span) ) );
            //            n_score = n.n_hits * n.query_span();
            n.score = n_score;
            n.nam_id = nams.size();
            nams.push_back(n);
        }
    }
}


__device__ void merge_hits(
    my_vector<my_pair<int, Hit>>& hits_per_ref,
    int k,
    bool is_revcomp,
    my_vector<Nam>& nams
) {
    if(hits_per_ref.size() == 0) return;
    int num_hits = hits_per_ref.size();

    int ref_num = 0;
    my_vector<int> each_ref_size;
    int pre_ref_id = hits_per_ref[0].first;
    int now_ref_num = 1;
    for(int i = 1; i < hits_per_ref.size(); i++) {
        int ref_id = hits_per_ref[i].first;
        Hit hit = hits_per_ref[i].second;
        if(ref_id != pre_ref_id) {
            ref_num++;
            pre_ref_id = ref_id;
            each_ref_size.push_back(now_ref_num);
            now_ref_num = 1;
        } else {
            now_ref_num++;
        }
    }
    ref_num++;
    each_ref_size.push_back(now_ref_num);

    my_vector<Nam> open_nams;

    int now_vec_pos = 0;
    for (int i = 0; i < ref_num; i++) {

        if(i != 0) now_vec_pos += each_ref_size[i - 1];
        int ref_id = hits_per_ref[now_vec_pos].first;
        open_nams.clear();
        unsigned int prev_q_start = 0;

        for (int j = 0; j < each_ref_size[i]; j++) {
            Hit& h = hits_per_ref[now_vec_pos + j].second;
            bool is_added = false;
            for (int k = 0; k < open_nams.size(); k++) {
                Nam& o = open_nams[k];

                // Extend NAM
                if ((o.query_prev_hit_startpos < h.query_start) && (h.query_start <= o.query_end ) && (o.ref_prev_hit_startpos < h.ref_start) && (h.ref_start <= o.ref_end) ){
                    if ( (h.query_end > o.query_end) && (h.ref_end > o.ref_end) ) {
                        o.query_end = h.query_end;
                        o.ref_end = h.ref_end;
                        //                        o.previous_query_start = h.query_s;
                        //                        o.previous_ref_start = h.ref_s; // keeping track so that we don't . Can be caused by interleaved repeats.
                        o.query_prev_hit_startpos = h.query_start;
                        o.ref_prev_hit_startpos = h.ref_start;
                        o.n_hits ++;
                        //                        o.score += (float)1/ (float)h.count;
                        is_added = true;
                        break;
                    }
                    else if ((h.query_end <= o.query_end) && (h.ref_end <= o.ref_end)) {
                        //                        o.previous_query_start = h.query_s;
                        //                        o.previous_ref_start = h.ref_s; // keeping track so that we don't . Can be caused by interleaved repeats.
                        o.query_prev_hit_startpos = h.query_start;
                        o.ref_prev_hit_startpos = h.ref_start;
                        o.n_hits ++;
                        //                        o.score += (float)1/ (float)h.count;
                        is_added = true;
                        break;
                    }
                }

            }

            // Add the hit to open matches
            if (!is_added){
                Nam n;
                n.query_start = h.query_start;
                n.query_end = h.query_end;
                n.ref_start = h.ref_start;
                n.ref_end = h.ref_end;
                n.ref_id = ref_id;
                //                n.previous_query_start = h.query_s;
                //                n.previous_ref_start = h.ref_s;
                n.query_prev_hit_startpos = h.query_start;
                n.ref_prev_hit_startpos = h.ref_start;
                n.n_hits = 1;
                n.is_rc = is_revcomp;
                //                n.score += (float)1 / (float)h.count;
                open_nams.push_back(n);
            }

            // Only filter if we have advanced at least k nucleotides
            if (h.query_start > prev_q_start + k) {
                // Output all NAMs from open_matches to final_nams that the current hit have passed
                for (int k = 0; k < open_nams.size(); k++) {
                    Nam& n = open_nams[k];
                    if (n.query_end < h.query_start) {
                        int n_max_span = my_max(n.query_span(), n.ref_span());
                        int n_min_span = my_min(n.query_span(), n.ref_span());
                        float n_score;
                        n_score = ( 2*n_min_span -  n_max_span) > 0 ? (float) (n.n_hits * ( 2*n_min_span -  n_max_span) ) : 1;   // this is really just n_hits * ( min_span - (offset_in_span) ) );
                        //                        n_score = n.n_hits * n.query_span();
                        n.score = n_score;
                        n.nam_id = nams.size();
                        nams.push_back(n);
                    }
                }

                // Remove all NAMs from open_matches that the current hit have passed
                auto c = h.query_start;
                int old_open_size = open_nams.size();
                open_nams.clear();
                for (int in = 0; in < old_open_size; ++in) {
                    if (!(open_nams[in].query_end < c)) {
                        open_nams.push_back(open_nams[in]);
                    }
                }
                prev_q_start = h.query_start;
            }
        }

        // Add all current open_matches to final NAMs
        for (int k = 0; k < open_nams.size(); k++) {
            Nam& n = open_nams[k];
            int n_max_span = my_max(n.query_span(), n.ref_span());
            int n_min_span = my_min(n.query_span(), n.ref_span());
            float n_score;
            n_score = ( 2*n_min_span -  n_max_span) > 0 ? (float) (n.n_hits * ( 2*n_min_span -  n_max_span) ) : 1;   // this is really just n_hits * ( min_span - (offset_in_span) ) );
            //            n_score = n.n_hits * n.query_span();
            n.score = n_score;
            n.nam_id = nams.size();
            nams.push_back(n);
        }
    }
}

__device__ void add_to_hits_per_ref(
    my_vector<my_pair<int, Hit>>& hits_per_ref,
    int query_start,
    int query_end,
    size_t position,
    const RefRandstrobe *d_randstrobes,
    size_t d_randstrobes_size,
    int k
) {
    int min_diff = 1 << 30;
    for (const auto hash = gpu_get_hash(d_randstrobes, d_randstrobes_size, position); gpu_get_hash(d_randstrobes, d_randstrobes_size, position) == hash; ++position) {
        int ref_start = d_randstrobes[position].position;
        int ref_end = ref_start + d_randstrobes[position].strobe2_offset() + k;
        int diff = std::abs((query_end - query_start) - (ref_end - ref_start));
        if (diff <= min_diff) {
            hits_per_ref.push_back({d_randstrobes[position].reference_index(), Hit{query_start, query_end, ref_start, ref_end}});
            min_diff = diff;
        }
    }
}


#define GPU_thread_task_size 1

__global__ void gpu_rescue_get_hits(
    int bits,
    unsigned int filter_cutoff,
    int rescue_cutoff,
    const RefRandstrobe *d_randstrobes,
    size_t d_randstrobes_size,
    const my_bucket_index_t *d_randstrobe_start_indices,
    int num_tasks,
    IndexParameters *index_para,
    uint64_t *global_hits_num,
    my_vector<QueryRandstrobe>* global_randstrobes,
    my_vector<my_pair<int, Hit>>* hits_per_ref0s,
    my_vector<my_pair<int, Hit>>* hits_per_ref1s,
    int* global_todo_ids
)
{
    int global_id = blockIdx.x * blockDim.x + threadIdx.x;
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    int l_range = global_id * GPU_thread_task_size;
    int r_range = l_range + GPU_thread_task_size;
    if (r_range > num_tasks) r_range = num_tasks;
    for (int id = l_range; id < r_range; id++) {
        int real_id = global_todo_ids[id];
        my_vector<my_pair<int, Hit>>* hits_per_ref0;
        my_vector<my_pair<int, Hit>>* hits_per_ref1;
        hits_per_ref0 = (my_vector<my_pair<int, Hit>>*)my_malloc(sizeof(my_vector<my_pair<int, Hit>>));
        hits_per_ref1 = (my_vector<my_pair<int, Hit>>*)my_malloc(sizeof(my_vector<my_pair<int, Hit>>));
        hits_per_ref0->init();
        hits_per_ref1->init();

        my_vector<RescueHit> hits_t0;
        my_vector<RescueHit> hits_t1;
        for (int i = 0; i < global_randstrobes[real_id].size(); i++) {
            QueryRandstrobe q = global_randstrobes[real_id][i];
            //size_t position = gpu_find(d_randstrobes, d_randstrobe_start_indices, q.hash, bits);
            size_t position = q.hash;
            if (position != static_cast<size_t>(-1)) {
                if(position >= d_randstrobes_size) {
                    printf("position > d_randstrobes_size : %llu %llu\n", position, d_randstrobes_size);
                    assert(false);
                }
                unsigned int count = gpu_get_count(d_randstrobes, d_randstrobe_start_indices, position, bits);
                RescueHit rh{position, count, q.start, q.end};
                if(q.is_reverse) hits_t1.push_back(rh);
                else hits_t0.push_back(rh);
            }
        }
        global_randstrobes[real_id].release();
        quick_sort(&(hits_t0[0]), hits_t0.size());
        quick_sort(&(hits_t1[0]), hits_t1.size());

#define pre_sort

#ifdef pre_sort
        int cnt0 = 0, cnt1 = 0;
        for (int i = 0; i < hits_t0.size(); i++) {
            RescueHit &rh = hits_t0[i];
            if ((rh.count > rescue_cutoff && cnt0 >= 5) || rh.count > rescue_threshold) {
                break;
            }
            cnt0++;
        }
        for (int i = 0; i < hits_t1.size(); i++) {
            RescueHit &rh = hits_t1[i];
            if ((rh.count > rescue_cutoff && cnt1 >= 5) || rh.count > rescue_threshold) {
                break;
            }
            cnt1++;
        }
        quick_sort_iterative(&(hits_t0[0]), 0, cnt0 - 1, [](const RescueHit &r1, const RescueHit &r2) {
            return r1.query_start < r2.query_start;
        });
        quick_sort_iterative(&(hits_t1[0]), 0, cnt1 - 1, [](const RescueHit &r1, const RescueHit &r2) {
            return r1.query_start < r2.query_start;
        });
        for (int i = 0; i < cnt0; i++) {
            RescueHit &rh = hits_t0[i];
            add_to_hits_per_ref(*hits_per_ref0, rh.query_start, rh.query_end, rh.position, d_randstrobes, d_randstrobes_size, index_para->syncmer.k);
        }
        for (int i = 0; i < cnt1; i++) {
            RescueHit &rh = hits_t1[i];
            add_to_hits_per_ref(*hits_per_ref1, rh.query_start, rh.query_end, rh.position, d_randstrobes, d_randstrobes_size, index_para->syncmer.k);
        }
#else
        int cnt = 0;
        for (int i = 0; i < hits_t0.size(); i++) {
            RescueHit &rh = hits_t0[i];
            if ((rh.count > rescue_cutoff && cnt >= 5) || rh.count > rescue_threshold) {
                break;
            }
            add_to_hits_per_ref(*hits_per_ref0, rh.query_start, rh.query_end, rh.position, d_randstrobes, d_randstrobes_size, index_para->syncmer.k);
            cnt++;
        }
        cnt = 0;
        for (int i = 0; i < hits_t1.size(); i++) {
            RescueHit &rh = hits_t1[i];
            if ((rh.count > rescue_cutoff && cnt >= 5) || rh.count > rescue_threshold) {
                break;
            }
            add_to_hits_per_ref(*hits_per_ref1, rh.query_start, rh.query_end, rh.position, d_randstrobes, d_randstrobes_size, index_para->syncmer.k);
            cnt++;
        }
#endif
        global_hits_num[real_id] = hits_per_ref0->size() + hits_per_ref1->size();
        hits_per_ref0s[real_id] = *hits_per_ref0;
        hits_per_ref1s[real_id] = *hits_per_ref1;
        my_free(hits_per_ref0);
        my_free(hits_per_ref1);
    }
}

__global__ void gpu_rescue_sort_hits_parallel(
    int num_tasks,
    IndexParameters *index_para,
    my_vector<my_pair<int, Hit>>* hits_per_ref0s,
    my_vector<my_pair<int, Hit>>* hits_per_ref1s,
    int* global_todo_ids
)
{
    int global_id = blockIdx.x * blockDim.x + threadIdx.x;
    int bid = blockIdx.x;
    int tid = threadIdx.x;

    int l_range = bid * GPU_thread_task_size;
    int r_range = l_range + GPU_thread_task_size;
    if (r_range > num_tasks) r_range = num_tasks;

    for (int id = l_range; id < r_range; id++) {
        int real_id = global_todo_ids[id];
        sort_hits_parallel(hits_per_ref0s[real_id], index_para->syncmer.k, 0, tid);
        sort_hits_parallel(hits_per_ref1s[real_id], index_para->syncmer.k, 1, tid);
    }
}

__global__ void gpu_rescue_merge_hits_get_nams(
    int num_tasks,
    IndexParameters *index_para,
    uint64_t *global_nams_info,
    my_vector<my_pair<int, Hit>>* hits_per_ref0s,
    my_vector<my_pair<int, Hit>>* hits_per_ref1s,
    my_vector<Nam> *global_nams,
    int* global_todo_ids
)
{
    int global_id = blockIdx.x * blockDim.x + threadIdx.x;
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    int l_range = global_id * GPU_thread_task_size;
    int r_range = l_range + GPU_thread_task_size;
    if (r_range > num_tasks) r_range = num_tasks;
    for (int id = l_range; id < r_range; id++) {
        int real_id = global_todo_ids[id];
        my_vector<Nam> *nams = (my_vector<Nam>*)my_malloc(sizeof(my_vector<Nam>));
        nams->init(128);
        salign_merge_hits(hits_per_ref0s[real_id], index_para->syncmer.k, 0, *nams);
        salign_merge_hits(hits_per_ref1s[real_id], index_para->syncmer.k, 1, *nams);


        //quick_sort(nams->data, nams->size());
        //quick_sort_iterative(nams->data, 0, nams->size() - 1, [](const Nam &a, const Nam &b) {
        //        if(a.score != b.score) return a.score > b.score;
        //        if(a.query_end != b.query_end) return a.query_end < b.query_end;
        //        if(a.query_start != b.query_start) return a.query_start < b.query_start;
        //        if(a.ref_end != b.ref_end) return a.ref_end < b.ref_end;
        //        if(a.ref_start != b.ref_start) return a.ref_start < b.ref_start;
        //});

        //check_nams(*nams);

        uint64_t local_nams_info = 0;
        for (int i = 0; i < nams->size(); i++) {
            local_nams_info += (*nams)[i].ref_id + int((*nams)[i].score) + (*nams)[i].query_start + (*nams)[i].query_end;
        }
        global_nams_info[real_id] += local_nams_info;
        global_nams[real_id] = *nams;
        my_free(nams);
        hits_per_ref0s[real_id].release();
        hits_per_ref1s[real_id].release();
    }
}


__global__ void gpu_get_randstrobes(
    int num_tasks,
    int *pre_sum,
    int *lens,
    char *all_seqs,
    IndexParameters *index_para,
    int *randstrobe_sizes,
    uint64_t *hashes,
    my_vector<QueryRandstrobe>* global_randstrobes
) {
    int global_id = blockIdx.x * blockDim.x + threadIdx.x;
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    int l_range = global_id * GPU_thread_task_size;
    int r_range = l_range + GPU_thread_task_size;
    if (r_range > num_tasks) r_range = num_tasks;
    int read_num = num_tasks / 2;
    for (int id = l_range; id < r_range; id++) {
        int read_id = id % read_num;
        int is_read2 = id / read_num;
        size_t len;
        char *seq, *rc;
        if (is_read2 == 0) {
            len = lens[read_id];
            seq = all_seqs + pre_sum[read_id];
            rc = all_seqs + pre_sum[read_id + read_num];
        } else {
            len = lens[read_id + read_num * 2];
            seq = all_seqs + pre_sum[read_id + read_num * 2];
            rc = all_seqs + pre_sum[read_id + read_num * 3];
        }

        my_vector<Syncmer> syncmers(len);

        const int k = index_para->syncmer.k;
        const int s = index_para->syncmer.s;
        const int t = index_para->syncmer.t_syncmer;

        const uint64_t kmask = (1ULL << 2 * k) - 1;
        const uint64_t smask = (1ULL << 2 * s) - 1;
        const uint64_t kshift = (k - 1) * 2;
        const uint64_t sshift = (s - 1) * 2;
        uint64_t gpu_qs[200];
        int l_pos = 0;
        int r_pos = 0;
        uint64_t qs_min_val = UINT64_MAX;
        int qs_min_pos = -1;
        int l = 0;
        uint64_t xk[2] = {0, 0};
        uint64_t xs[2] = {0, 0};
        for (size_t i = 0; i < len; i++) {
            int c = gpu_seq_nt4_table[(uint8_t) seq[i]];
            if (c < 4) { // not an "N" base
                xk[0] = (xk[0] << 2 | c) & kmask;                  // forward strand
                xk[1] = xk[1] >> 2 | (uint64_t)(3 - c) << kshift;  // reverse strand
                xs[0] = (xs[0] << 2 | c) & smask;                  // forward strand
                xs[1] = xs[1] >> 2 | (uint64_t)(3 - c) << sshift;  // reverse strand
                if (++l < s) {
                    continue;
                }
                // we find an s-mer
                uint64_t ys = xs[0] < xs[1] ? xs[0] : xs[1];
                uint64_t hash_s = gpu_syncmer_smer_hash(ys);
                gpu_qs[r_pos++] = hash_s;
                // not enough hashes in the queue, yet
                if (r_pos - l_pos < k - s + 1) {
                    continue;
                }
                if (r_pos - l_pos == k - s + 1) { // We are at the last s-mer within the first k-mer, need to decide if we add it
                    for (int j = l_pos; j < r_pos; j++) {
                        if (gpu_qs[j] < qs_min_val) {
                            qs_min_val = gpu_qs[j];
                            qs_min_pos = i - k + j - l_pos + 1;
                        }
                    }
                } else {
                    // update queue and current minimum and position
                    l_pos++;
                    if (qs_min_pos == i - k) { // we popped the previous minimizer, find new brute force
                        qs_min_val = UINT64_MAX;
                        qs_min_pos = i - s + 1;
                        for (int j = r_pos - 1; j >= l_pos; j--) { //Iterate in reverse to choose the rightmost minimizer in a window
                            if (gpu_qs[j] < qs_min_val) {
                                qs_min_val = gpu_qs[j];
                                qs_min_pos = i - k + j - l_pos + 1;
                            }
                        }
                    } else if (hash_s < qs_min_val) { // the new value added to queue is the new minimum
                        qs_min_val = hash_s;
                        qs_min_pos = i - s + 1;
                    }
                }
                if (qs_min_pos == i - k + t) { // occurs at t:th position in k-mer
                    uint64_t yk = xk[0] < xk[1] ? xk[0] : xk[1];
                    syncmers.push_back(Syncmer{gpu_syncmer_kmer_hash(yk), i - k + 1});
                }
            } else {
                // if there is an "N", restart
                qs_min_val = UINT64_MAX;
                qs_min_pos = -1;
                l = xs[0] = xs[1] = xk[0] = xk[1] = 0;
                r_pos = 0;
                l_pos = 0;
            }
        }


        const int w_min = index_para->randstrobe.w_min;
        const int w_max = index_para->randstrobe.w_max;
        const uint64_t q = index_para->randstrobe.q;
        const int max_dist = index_para->randstrobe.max_dist;

        my_vector<QueryRandstrobe> *randstrobes;
        randstrobes = (my_vector<QueryRandstrobe>*)my_malloc(sizeof(my_vector<QueryRandstrobe>));
        randstrobes->init((my_max(syncmers.size() - w_min, 0)) * 2);


        for (int strobe1_index = 0; strobe1_index + w_min < syncmers.size(); strobe1_index++) {
            unsigned int w_end = (strobe1_index + w_max < syncmers.size() - 1) ? (strobe1_index + w_max) : syncmers.size() - 1;
            auto strobe1 = syncmers[strobe1_index];
            auto max_position = strobe1.position + max_dist;
            unsigned int w_start = strobe1_index + w_min;
            uint64_t min_val = 0xFFFFFFFFFFFFFFFF;
            Syncmer strobe2 = strobe1;
            for (auto i = w_start; i <= w_end && syncmers[i].position <= max_position; i++) {
                uint64_t hash_diff = (strobe1.hash ^ syncmers[i].hash) & q;
                uint64_t res = __popcll(hash_diff);
                if (res < min_val) {
                    min_val = res;
                    strobe2 = syncmers[i];
                }
            }
            randstrobes->push_back(
                QueryRandstrobe{
                    gpu_randstrobe_hash(strobe1.hash, strobe2.hash), static_cast<uint32_t>(strobe1.position),
                    static_cast<uint32_t>(strobe2.position) + index_para->syncmer.k, false
                }
            );
        }


        for (int i = 0; i < syncmers.size() / 2; i++) {
            my_swap(syncmers[i], syncmers[syncmers.size() - i - 1]);
        }
        for (size_t i = 0; i < syncmers.size(); i++) {
            syncmers[i].position = len - syncmers[i].position - (*index_para).syncmer.k;
        }

        for (int strobe1_index = 0; strobe1_index + w_min < syncmers.size(); strobe1_index++) {
            unsigned int w_end = (strobe1_index + w_max < syncmers.size() - 1) ? (strobe1_index + w_max) : syncmers.size() - 1;
            auto strobe1 = syncmers[strobe1_index];
            auto max_position = strobe1.position + max_dist;
            unsigned int w_start = strobe1_index + w_min;
            uint64_t min_val = 0xFFFFFFFFFFFFFFFF;
            Syncmer strobe2 = strobe1;
            for (auto i = w_start; i <= w_end && syncmers[i].position <= max_position; i++) {
                uint64_t hash_diff = (strobe1.hash ^ syncmers[i].hash) & q;
                uint64_t res = __popcll(hash_diff);
                if (res < min_val) {
                    min_val = res;
                    strobe2 = syncmers[i];
                }
            }
            randstrobes->push_back(
                QueryRandstrobe{
                    gpu_randstrobe_hash(strobe1.hash, strobe2.hash), static_cast<uint32_t>(strobe1.position),
                    static_cast<uint32_t>(strobe2.position) + index_para->syncmer.k, true
                }
            );
        }


        randstrobe_sizes[id] += randstrobes->size();
        for (int i = 0; i < randstrobes->size(); i++) hashes[id] += (*randstrobes)[i].hash;
        global_randstrobes[id] = *randstrobes;
        my_free(randstrobes);
        //        randstrobe_sizes[id] += syncmers.size();
    }
}

__global__ void gpu_get_hits_after(
    int bits,
    unsigned int filter_cutoff,
    int rescue_cutoff,
    const RefRandstrobe *d_randstrobes,
    size_t d_randstrobes_size,
    const my_bucket_index_t *d_randstrobe_start_indices,
    int num_tasks,
    IndexParameters *index_para,
    uint64_t *global_hits_num,
    my_vector<QueryRandstrobe>* global_randstrobes,
    my_vector<my_pair<int, Hit>>* hits_per_ref0s,
    my_vector<my_pair<int, Hit>>* hits_per_ref1s
) {
    int global_id = blockIdx.x * blockDim.x + threadIdx.x;
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    int l_range = global_id * GPU_thread_task_size;
    int r_range = l_range + GPU_thread_task_size;
    if (r_range > num_tasks) r_range = num_tasks;
    for (int id = l_range; id < r_range; id++) {
        int sum_seeds0 = 0;
        int sum_seeds1 = 0;
        for (int i = 0; i < global_randstrobes[id].size(); i++) {
            if (global_randstrobes[id][i].is_reverse) {
                sum_seeds1++;
            } else {
                sum_seeds0++;
            }
        }
        my_vector<my_pair<int, Hit>>* hits_per_ref0;
        my_vector<my_pair<int, Hit>>* hits_per_ref1;
        hits_per_ref0 = (my_vector<my_pair<int, Hit>>*)my_malloc(sizeof(my_vector<my_pair<int, Hit>>));
        hits_per_ref1 = (my_vector<my_pair<int, Hit>>*)my_malloc(sizeof(my_vector<my_pair<int, Hit>>));
        hits_per_ref0->init(sum_seeds0 * 2);
        hits_per_ref1->init(sum_seeds1 * 2);

        uint64_t local_total_hits = 0;
        uint64_t local_nr_good_hits = 0;
        for (int i = 0; i < global_randstrobes[id].size(); i++) {
            QueryRandstrobe q = global_randstrobes[id][i];
            size_t position = q.hash;
            if (position != static_cast<size_t>(-1)) {
                local_total_hits++;
                bool res = gpu_is_filtered(d_randstrobes, d_randstrobes_size, position, filter_cutoff);
                if (res) continue;
                local_nr_good_hits++;
                if(q.is_reverse) {
                    add_to_hits_per_ref(*hits_per_ref1, q.start, q.end, position, d_randstrobes, d_randstrobes_size, index_para->syncmer.k);
                } else {
                    add_to_hits_per_ref(*hits_per_ref0, q.start, q.end, position, d_randstrobes, d_randstrobes_size, index_para->syncmer.k);
                }
            }
        }
        float nonrepetitive_fraction = local_total_hits > 0 ? ((float) local_nr_good_hits) / ((float) local_total_hits) : 1.0;

        if (nonrepetitive_fraction < 0.7 || hits_per_ref0->size() + hits_per_ref1->size() == 0) {
            hits_per_ref0->release();
            hits_per_ref1->release();
        } else {
            global_randstrobes[id].release();
        }
        global_hits_num[id] = hits_per_ref0->size() + hits_per_ref1->size();
        hits_per_ref0s[id] = *hits_per_ref0;
        hits_per_ref1s[id] = *hits_per_ref1;
        my_free(hits_per_ref0);
        my_free(hits_per_ref1);
    }
}

__global__ void gpu_get_hits_pre(
    int bits,
    unsigned int filter_cutoff,
    int rescue_cutoff,
    const RefRandstrobe *d_randstrobes,
    size_t d_randstrobes_size,
    const my_bucket_index_t *d_randstrobe_start_indices,
    int num_tasks,
    IndexParameters *index_para,
    uint64_t *global_hits_num,
    my_vector<QueryRandstrobe>* global_randstrobes,
    my_vector<my_pair<int, Hit>>* hits_per_ref0s,
    my_vector<my_pair<int, Hit>>* hits_per_ref1s
) {
    int global_id = blockIdx.x * blockDim.x + threadIdx.x;
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    int l_range = global_id * GPU_thread_task_size;
    int r_range = l_range + GPU_thread_task_size;
    if (r_range > num_tasks) r_range = num_tasks;
    for (int id = l_range; id < r_range; id++) {
        for (int i = 0; i < global_randstrobes[id].size(); i++) {
            QueryRandstrobe q = global_randstrobes[id][i];
            size_t position = gpu_find(d_randstrobes, d_randstrobe_start_indices, q.hash, bits);
            //if(position != static_cast<size_t>(-1) && position >= d_randstrobes_size) {
            //    printf("position GG %zu %zu\n", position, d_randstrobes_size);
            //    assert(false);
            //}
            global_randstrobes[id][i].hash = position;
        }
    }
}

__global__ void gpu_get_hits(
    int bits,
    unsigned int filter_cutoff,
    int rescue_cutoff,
    const RefRandstrobe *d_randstrobes,
    size_t d_randstrobes_size,
    const my_bucket_index_t *d_randstrobe_start_indices,
    int num_tasks,
    IndexParameters *index_para,
    uint64_t *global_hits_num,
    my_vector<QueryRandstrobe>* global_randstrobes,
    my_vector<my_pair<int, Hit>>* hits_per_ref0s,
    my_vector<my_pair<int, Hit>>* hits_per_ref1s
) {
    int global_id = blockIdx.x * blockDim.x + threadIdx.x;
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    int l_range = global_id * GPU_thread_task_size;
    int r_range = l_range + GPU_thread_task_size;
    if (r_range > num_tasks) r_range = num_tasks;
    for (int id = l_range; id < r_range; id++) {
        int read_id = id / 2;
        int rev = id % 2;

        my_vector<my_pair<int, Hit>>* hits_per_ref0;
        my_vector<my_pair<int, Hit>>* hits_per_ref1;
        hits_per_ref0 = (my_vector<my_pair<int, Hit>>*)my_malloc(sizeof(my_vector<my_pair<int, Hit>>));
        hits_per_ref1 = (my_vector<my_pair<int, Hit>>*)my_malloc(sizeof(my_vector<my_pair<int, Hit>>));
        hits_per_ref0->init();
        hits_per_ref1->init();

        uint64_t local_total_hits = 0;
        uint64_t local_nr_good_hits = 0;
        for (int i = 0; i < global_randstrobes[id].size(); i++) {
            QueryRandstrobe q = global_randstrobes[id][i];
            size_t position = gpu_find(d_randstrobes, d_randstrobe_start_indices, q.hash, bits);
            global_randstrobes[id][i].hash = position;
            if (position != static_cast<size_t>(-1)) {
                local_total_hits++;
                bool res = gpu_is_filtered(d_randstrobes, d_randstrobes_size, position, filter_cutoff);
                if (res) continue;
                local_nr_good_hits++;
                if(q.is_reverse) {
                    add_to_hits_per_ref(*hits_per_ref1, q.start, q.end, position, d_randstrobes, d_randstrobes_size, index_para->syncmer.k);
                } else {
                    add_to_hits_per_ref(*hits_per_ref0, q.start, q.end, position, d_randstrobes, d_randstrobes_size, index_para->syncmer.k);
                }
            }
        }
        float nonrepetitive_fraction = local_total_hits > 0 ? ((float) local_nr_good_hits) / ((float) local_total_hits) : 1.0;

        if (nonrepetitive_fraction < 0.7 || hits_per_ref0->size() + hits_per_ref1->size() == 0) {
            hits_per_ref0->release();
            hits_per_ref1->release();
        } else {
            global_randstrobes[id].release();
        }
        global_hits_num[id] = hits_per_ref0->size() + hits_per_ref1->size();
        hits_per_ref0s[id] = *hits_per_ref0;
        hits_per_ref1s[id] = *hits_per_ref1;
        my_free(hits_per_ref0);
        my_free(hits_per_ref1);
    }
}


__global__ void gpu_sort_nams(
    int num_tasks,
    my_vector<Nam> *global_nams,
    MappingParameters *mapping_parameters
) {
    int global_id = blockIdx.x * blockDim.x + threadIdx.x;
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    int l_range = global_id * GPU_thread_task_size;
    int r_range = l_range + GPU_thread_task_size;
    if (r_range > num_tasks) r_range = num_tasks;
    int read_num = num_tasks / 2;
    for (int id = l_range; id < r_range; id++) {
        int max_tries = mapping_parameters->max_tries;
        sort_nams_by_score(global_nams[id], max_tries * 2);
//        sort_nams_by_score(global_nams[id], 1e9);
        global_nams[id].length = my_min(global_nams[id].length, max_tries * 2);
    }
}

__global__ void gpu_rescue_sort_hits(
    int num_tasks,
    my_vector<my_pair<int, Hit>>* hits_per_ref0s,
    my_vector<my_pair<int, Hit>>* hits_per_ref1s,
    int* global_todo_ids
) {
    int global_id = blockIdx.x * blockDim.x + threadIdx.x;
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    int l_range = global_id * GPU_thread_task_size;
    int r_range = l_range + GPU_thread_task_size;
    if (r_range > num_tasks) r_range = num_tasks;
    for (int id = l_range; id < r_range; id++) {
        int real_id = global_todo_ids[id];
        //        sort_hits_single(hits_per_ref0s[real_id]);
        //        sort_hits_single(hits_per_ref1s[real_id]);
        sort_hits_by_refid(hits_per_ref0s[real_id]);
        sort_hits_by_refid(hits_per_ref1s[real_id]);
        //        check_hits(hits_per_ref0s[real_id]);
        //        check_hits(hits_per_ref1s[real_id]);
    }
}

__global__ void gpu_sort_hits(
    int num_tasks,
    my_vector<my_pair<int, Hit>>* hits_per_ref0s,
    my_vector<my_pair<int, Hit>>* hits_per_ref1s,
    int* global_todo_ids
) {
    int global_id = blockIdx.x * blockDim.x + threadIdx.x;
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    int l_range = global_id * GPU_thread_task_size;
    int r_range = l_range + GPU_thread_task_size;
    if (r_range > num_tasks) r_range = num_tasks;
    for (int id = l_range; id < r_range; id++) {
        int real_id = global_todo_ids[id];
        //        sort_hits_single(hits_per_ref0s[real_id]);
        //        sort_hits_single(hits_per_ref1s[real_id]);
        sort_hits_by_refid(hits_per_ref0s[real_id]);
        sort_hits_by_refid(hits_per_ref1s[real_id]);
    }
}


__global__ void gpu_merge_hits_get_nams(
    int num_tasks,
    IndexParameters *index_para,
    uint64_t *global_nams_info,
    my_vector<my_pair<int, Hit>>* hits_per_ref0s,
    my_vector<my_pair<int, Hit>>* hits_per_ref1s,
    my_vector<Nam> *global_nams,
    int* global_todo_ids
) {
    int global_id = blockIdx.x * blockDim.x + threadIdx.x;
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    int l_range = global_id * GPU_thread_task_size;
    int r_range = l_range + GPU_thread_task_size;
    if (r_range > num_tasks) r_range = num_tasks;
    for (int id = l_range; id < r_range; id++) {
        int real_id = global_todo_ids[id];
        my_vector<Nam> *nams = (my_vector<Nam>*)my_malloc(sizeof(my_vector<Nam>));
        nams->init();
        salign_merge_hits(hits_per_ref0s[real_id], index_para->syncmer.k, 0, *nams);
        salign_merge_hits(hits_per_ref1s[real_id], index_para->syncmer.k, 1, *nams);
        hits_per_ref0s[real_id].release();
        hits_per_ref1s[real_id].release();
        uint64_t local_nams_info = 0;
        for (int i = 0; i < nams->size(); i++) {
            local_nams_info += (*nams)[i].ref_id + int((*nams)[i].score) + (*nams)[i].query_start + (*nams)[i].query_end;
        }
        global_nams_info[real_id] += local_nams_info;
        global_nams[real_id] = *nams;
        my_free(nams);
    }
}


__global__ void gpu_pre_cal_type(
    int num_tasks,
    float dropoff_threshold,
    my_vector<Nam> *global_nams,
    int *global_todo_ids) {
    int global_id = blockIdx.x * blockDim.x + threadIdx.x;
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    int l_range = global_id * GPU_thread_task_size;
    int r_range = l_range + GPU_thread_task_size;
    if (r_range > num_tasks) r_range = num_tasks;
    for (int id = l_range; id < r_range; id++) {
        GPUInsertSizeDistribution isize_est;
        my_vector<Nam> &nams1 = global_nams[id];
        my_vector<Nam> &nams2 = global_nams[id + num_tasks];
        if (nams1.empty() && nams2.empty()) {
            global_todo_ids[id] = 0;
        } else if (!nams1.empty() && nams2.empty()) {
            global_todo_ids[id] = 1;
        } else if (nams1.empty() && !nams2.empty()) {
            global_todo_ids[id] = 2;
        } else if (gpu_top_dropoff(nams1) < dropoff_threshold && gpu_top_dropoff(nams2) < dropoff_threshold && gpu_is_proper_nam_pair(nams1[0], nams2[0], isize_est.mu, isize_est.sigma)) {
            global_todo_ids[id] = 3;
        } else {
            global_todo_ids[id] = 4;
        }
        //        global_nams[id].release();
        //        global_nams[id + num_tasks].release();
    }
}

__global__ void gpu_align_PE0(
    int num_tasks,
    int s_len,
    IndexParameters *index_para,
    uint64_t *global_align_info,
    AlignmentParameters* aligner_parameters,
    int *pre_sum,
    int *lens,
    char *all_seqs,
    GPUReferences *global_references,
    MappingParameters *mapping_parameters,
    my_vector<Nam> *global_nams,
    int *global_todo_ids,
    GPUAlignTmpRes *global_align_res
) {
    int global_id = blockIdx.x * blockDim.x + threadIdx.x;
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    int l_range = global_id * GPU_thread_task_size;
    int r_range = l_range + GPU_thread_task_size;
    if (r_range > num_tasks) r_range = num_tasks;
    for (int id = l_range; id < r_range; id++) {
        int real_id = global_todo_ids[id];
        global_nams[real_id].release();
        global_nams[real_id + s_len].release();
    }
}

__global__ void gpu_align_PE12(
    int num_tasks,
    int s_len,
    IndexParameters *index_para,
    uint64_t *global_align_info,
    AlignmentParameters* aligner_parameters,
    int *pre_sum,
    int *lens,
    char *all_seqs,
    GPUReferences *global_references,
    MappingParameters *mapping_parameters,
    my_vector<Nam> *global_nams,
    int *global_todo_ids,
    GPUAlignTmpRes *global_align_res
) {
    int global_id = blockIdx.x * blockDim.x + threadIdx.x;
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    int l_range = global_id * GPU_thread_task_size;
    int r_range = l_range + GPU_thread_task_size;
    if (r_range > num_tasks) r_range = num_tasks;
    for (int id = l_range; id < r_range; id++) {
        int real_id = global_todo_ids[id] >> 1;
        int type = global_todo_ids[id] % 2 == 0 ? 1 : 2;
        size_t seq_len1, seq_len2;
        seq_len1 = lens[real_id];
        seq_len2 = lens[real_id + s_len * 2];
        char *seq1, *seq2, *rc1, *rc2;
        seq1 = all_seqs + pre_sum[real_id];
        rc1 = all_seqs + pre_sum[real_id + s_len];
        seq2 = all_seqs + pre_sum[real_id + s_len * 2];
        rc2 = all_seqs + pre_sum[real_id + s_len * 3];

        GPUAlignTmpRes* align_tmp_res = &global_align_res[real_id];
        GPUInsertSizeDistribution isize_est;
        align_PE_part12(*align_tmp_res, *aligner_parameters, global_nams[real_id], global_nams[real_id + s_len],
                       seq1, rc1, seq_len1, seq2, rc2, seq_len2, index_para->syncmer.k, *global_references,
                       mapping_parameters->dropoff_threshold, isize_est, mapping_parameters->max_tries, mapping_parameters->max_secondary, type, real_id);
        global_align_info[real_id] += align_tmp_res->type + align_tmp_res->mapq1 + align_tmp_res->mapq2 + align_tmp_res->type4_loop_size;
        global_align_info[real_id] += align_tmp_res->is_extend_seed.size() + align_tmp_res->consistent_nam.size() + align_tmp_res->is_read1.size() +
                                      align_tmp_res->type4_nams.size() + align_tmp_res->todo_nams.size() + align_tmp_res->done_align.size() + align_tmp_res->align_res.size();
        uint64_t local_sum = 0;
        for (int i = 0; i < align_tmp_res->todo_nams.size(); i++) {
            local_sum += align_tmp_res->todo_nams[i].ref_id;
        }
        global_align_info[real_id] += local_sum;
        global_nams[real_id].release();
        global_nams[real_id + s_len].release();
    }
}

__global__ void gpu_align_PE3(
    int num_tasks,
    int s_len,
    IndexParameters *index_para,
    uint64_t *global_align_info,
    AlignmentParameters* aligner_parameters,
    int *pre_sum,
    int *lens,
    char *all_seqs,
    GPUReferences *global_references,
    MappingParameters *mapping_parameters,
    my_vector<Nam> *global_nams,
    int *global_todo_ids,
    GPUAlignTmpRes *global_align_res
) {
    int global_id = blockIdx.x * blockDim.x + threadIdx.x;
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    int l_range = global_id * GPU_thread_task_size;
    int r_range = l_range + GPU_thread_task_size;
    if (r_range > num_tasks) r_range = num_tasks;
    for (int id = l_range; id < r_range; id++) {
        int real_id = global_todo_ids[id];
        size_t seq_len1, seq_len2;
        seq_len1 = lens[real_id];
        seq_len2 = lens[real_id + s_len * 2];
        char *seq1, *seq2, *rc1, *rc2;
        seq1 = all_seqs + pre_sum[real_id];
        rc1 = all_seqs + pre_sum[real_id + s_len];
        seq2 = all_seqs + pre_sum[real_id + s_len * 2];
        rc2 = all_seqs + pre_sum[real_id + s_len * 3];

        GPUAlignTmpRes* align_tmp_res = &global_align_res[real_id];
        GPUInsertSizeDistribution isize_est;
        align_PE_part3(*align_tmp_res, *aligner_parameters, global_nams[real_id], global_nams[real_id + s_len],
                       seq1, rc1, seq_len1, seq2, rc2, seq_len2, index_para->syncmer.k, *global_references,
                       mapping_parameters->dropoff_threshold, isize_est, mapping_parameters->max_tries, mapping_parameters->max_secondary, real_id);
        global_align_info[real_id] += align_tmp_res->type + align_tmp_res->mapq1 + align_tmp_res->mapq2 + align_tmp_res->type4_loop_size;
        global_align_info[real_id] += align_tmp_res->is_extend_seed.size() + align_tmp_res->consistent_nam.size() + align_tmp_res->is_read1.size() +
                                      align_tmp_res->type4_nams.size() + align_tmp_res->todo_nams.size() + align_tmp_res->done_align.size() + align_tmp_res->align_res.size();
        uint64_t local_sum = 0;
        for (int i = 0; i < align_tmp_res->todo_nams.size(); i++) {
            local_sum += align_tmp_res->todo_nams[i].ref_id;
        }
        global_align_info[real_id] += local_sum;
        global_nams[real_id].release();
        global_nams[real_id + s_len].release();
    }
}


__global__ void gpu_align_PE4(
    int num_tasks,
    int s_len,
    IndexParameters *index_para,
    uint64_t *global_align_info,
    AlignmentParameters* aligner_parameters,
    int *pre_sum,
    int *lens,
    char *all_seqs,
    GPUReferences *global_references,
    MappingParameters *mapping_parameters,
    my_vector<Nam> *global_nams,
    int *global_todo_ids,
    GPUAlignTmpRes *global_align_res
) {
    int global_id = blockIdx.x * blockDim.x + threadIdx.x;
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    int l_range = global_id * GPU_thread_task_size;
    int r_range = l_range + GPU_thread_task_size;
    if (r_range > num_tasks) r_range = num_tasks;
    for (int id = l_range; id < r_range; id++) {
        int real_id = global_todo_ids[id];
        size_t seq_len1, seq_len2;
        seq_len1 = lens[real_id];
        seq_len2 = lens[real_id + s_len * 2];
        char *seq1, *seq2, *rc1, *rc2;
        seq1 = all_seqs + pre_sum[real_id];
        rc1 = all_seqs + pre_sum[real_id + s_len];
        seq2 = all_seqs + pre_sum[real_id + s_len * 2];
        rc2 = all_seqs + pre_sum[real_id + s_len * 3];

        GPUAlignTmpRes* align_tmp_res = &global_align_res[real_id];
        GPUInsertSizeDistribution isize_est;
        align_PE_part4(*align_tmp_res, *aligner_parameters, global_nams[real_id], global_nams[real_id + s_len],
                       seq1, rc1, seq_len1, seq2, rc2, seq_len2, index_para->syncmer.k, *global_references,
                       mapping_parameters->dropoff_threshold, isize_est, mapping_parameters->max_tries, mapping_parameters->max_secondary, tid, real_id);
        global_align_info[real_id] += align_tmp_res->type + align_tmp_res->mapq1 + align_tmp_res->mapq2 + align_tmp_res->type4_loop_size;
        global_align_info[real_id] += align_tmp_res->is_extend_seed.size() + align_tmp_res->consistent_nam.size() + align_tmp_res->is_read1.size() +
                                      align_tmp_res->type4_nams.size() + align_tmp_res->todo_nams.size() + align_tmp_res->done_align.size() + align_tmp_res->align_res.size();
        uint64_t local_sum = 0;
        for (int i = 0; i < align_tmp_res->todo_nams.size(); i++) {
            local_sum += align_tmp_res->todo_nams[i].ref_id;
        }
        global_align_info[real_id] += local_sum;
        global_nams[real_id].release();
        global_nams[real_id + s_len].release();
    }
}

__global__ void gpu_free_align_res(int num_tasks, GPUAlignTmpRes *global_align_res) {
    int global_id = blockIdx.x * blockDim.x + threadIdx.x;
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    int l_range = global_id * GPU_thread_task_size;
    int r_range = l_range + GPU_thread_task_size;
    if (r_range > num_tasks) r_range = num_tasks;
    for (int id = l_range; id < r_range; id++) {
        global_align_res[id].is_extend_seed.release();
        global_align_res[id].consistent_nam.release();
        global_align_res[id].is_read1.release();
        global_align_res[id].type4_nams.release();
        global_align_res[id].todo_nams.release();
        global_align_res[id].done_align.release();
        global_align_res[id].align_res.release();
    }
}

klibpp::KSeq gpu_ConvertNeo2KSeq(neoReference ref) {
    klibpp::KSeq res;
    res.name = std::string((char *) ref.base + ref.pname, ref.lname);
    if (!res.name.empty()) {
        size_t space_pos = res.name.find(' ');
        int l_pos = 0;
        if (res.name[0] == '@') l_pos = 1;
        if (space_pos != std::string::npos) {
            res.name = res.name.substr(l_pos, space_pos - l_pos);
        } else {
            res.name = res.name.substr(l_pos);
        }
    }
    res.seq = std::string((char *) ref.base + ref.pseq, ref.lseq);
    res.comment = std::string((char *) ref.base + ref.pstrand, ref.lstrand);
    res.qual = std::string((char *) ref.base + ref.pqual, ref.lqual);
    return res;
}

thread_local uint64_t check_sum = 0;
thread_local uint64_t size_tot = 0;

thread_local uint64_t global_hits_num12 = 0;
thread_local uint64_t global_hits_num3 = 0;

thread_local uint64_t global_nams_info12 = 0;
thread_local uint64_t global_nams_info3 = 0;

thread_local uint64_t global_align_info123 = 0;


thread_local double gpu_copy1 = 0;
thread_local double gpu_copy2 = 0;
thread_local double gpu_cost1 = 0;
thread_local double gpu_cost2 = 0;
thread_local double gpu_cost2_1 = 0;
thread_local double gpu_cost2_2 = 0;
thread_local double gpu_cost3 = 0;
thread_local double gpu_cost4 = 0;
thread_local double gpu_cost5 = 0;
thread_local double gpu_cost6 = 0;
thread_local double gpu_cost7 = 0;
thread_local double gpu_cost8 = 0;
thread_local double gpu_cost9 = 0;
thread_local double gpu_cost10 = 0;
thread_local double gpu_cost10_0 = 0;
thread_local double gpu_cost10_1 = 0;
thread_local double gpu_cost10_2 = 0;
thread_local double gpu_cost10_3 = 0;
thread_local double gpu_cost10_4 = 0;
thread_local double gpu_cost11 = 0;
thread_local double gpu_cost11_copy1 = 0;
thread_local double gpu_cost11_copy2 = 0;
thread_local double tot_cost = 0;

template <typename T>
std::vector<T> copy_vector_to_host(const my_vector<T>& device_vec) {
    std::vector<T> host_vec(device_vec.length);
//    hipMemcpy(host_vec.data(), (const void*)device_vec.data, sizeof(T) * device_vec.length, hipMemcpyDeviceToHost);
    memcpy(host_vec.data(), (const void*)device_vec.data, sizeof(T) * device_vec.length);
    return host_vec;
}

template <typename T>
void fast_copy_vector_to_host(const my_vector<T>& device_vec, std::vector<T>& host_vec) {
    host_vec.resize(device_vec.length);
    memcpy(host_vec.data(), (const void*)device_vec.data, sizeof(T) * device_vec.length);
}



void print_global_align_res(GPUAlignTmpRes* global_align_res, int batch_size) {
    for (int i = 0; i < batch_size; ++i) {
        const GPUAlignTmpRes& tmp = global_align_res[i];

        printf("=== global_align_res[%d] ===\n", i);
        printf("type = %d, mapq1 = %d, mapq2 = %d, type4_loop_size = %d, type4_nams_size %d, is_read1_size %d\n",
               tmp.type, tmp.mapq1, tmp.mapq2, tmp.type4_loop_size, tmp.type4_nams.length, tmp.is_read1.length);

        std::vector<int> is_extend_seeds = copy_vector_to_host(tmp.is_extend_seed);
        std::vector<int> consistent_nams = copy_vector_to_host(tmp.consistent_nam);
        std::vector<int> is_read1s       = copy_vector_to_host(tmp.is_read1);
        std::vector<Nam> type4_nams      = copy_vector_to_host(tmp.type4_nams);
        std::vector<Nam> todo_nams       = copy_vector_to_host(tmp.todo_nams);
        std::vector<int> done_flags      = copy_vector_to_host(tmp.done_align);
        std::vector<GPUAlignment> aligns = copy_vector_to_host(tmp.align_res);

        printf("is_extend_seeds (size = %lu):\n", is_extend_seeds.size());
        for (size_t j = 0; j < is_extend_seeds.size(); ++j) {
            printf("%d ", is_extend_seeds[j]);
        }
        printf("\n");

        printf("consistent_nams (size = %lu):\n", consistent_nams.size());
        for (size_t j = 0; j < consistent_nams.size(); ++j) {
            printf("%d ", consistent_nams[j]);
        }
        printf("\n");

        printf("is_read1s (size = %lu):\n", is_read1s.size());
        for (size_t j = 0; j < is_read1s.size(); ++j) {
            printf("%d ", is_read1s[j]);
        }
        printf("\n");

        printf("type4_nams (size = %lu):\n", type4_nams.size());
        for (size_t j = 0; j < type4_nams.size(); ++j) {
            const Nam& n = type4_nams[j];
            printf("  [%zu] ref_id=%d ref_start=%d ref_end=%d query_start=%d query_end=%d score=%.2f rc=%d\n",
                   j, n.ref_id, n.ref_start, n.ref_end, n.query_start, n.query_end, n.score, n.is_rc);
        }

        printf("todo_nams (size = %lu):\n", todo_nams.size());
        for (size_t j = 0; j < todo_nams.size(); ++j) {
            const Nam& n = todo_nams[j];
            printf("  [%zu] ref_id=%d ref_start=%d ref_end=%d query_start=%d query_end=%d score=%.2f rc=%d\n",
                   j, n.ref_id, n.ref_start, n.ref_end, n.query_start, n.query_end, n.score, n.is_rc);
        }

        printf("done_flags (size = %lu):\n", done_flags.size());
        for (size_t j = 0; j < done_flags.size(); ++j) {
            printf("%d ", done_flags[j]);
        }
        printf("\n");

        printf("align_res (size = %lu):\n", aligns.size());
        assert(aligns.size() == done_flags.size());
        for (size_t j = 0; j < aligns.size(); ++j) {
            const GPUAlignment& aln = aligns[j];
            //if (done_flags[j] == 1 && aln.is_unaligned == 0) {
            if (done_flags[j] == 1) {
                printf("  [%zu] ref_id=%d ref_start=%d ed=%d global_ed=%d score=%d len=%d is_rc=%d unaligned=%d gapped=%d\n",
                       j, aln.ref_id, aln.ref_start, aln.edit_distance, aln.global_ed, aln.score,
                       aln.length, aln.is_rc, aln.is_unaligned, aln.gapped);
            } else {
                printf("  [%zu] unaligned\n", j);
            }
        }
    }
}

void fast_copy_align_res(GPUAlignTmpRes* global_align_res, int batch_size, std::vector<AlignTmpRes>& align_tmp_results) {
//    assert(align_tmp_results.size() == 0);
    uint64_t cigar_size = 0;
    uint64_t mx_cigar_size = 0;
//    printf("align size %d\n", align_tmp_results.size());
    for (int i = 0; i < batch_size; ++i) {
        const GPUAlignTmpRes& tmp = global_align_res[i];
        AlignTmpRes align_tmp_res;

        align_tmp_res.type = tmp.type;
        align_tmp_res.mapq1 = tmp.mapq1;
        align_tmp_res.mapq2 = tmp.mapq2;
        align_tmp_res.type4_loop_size = tmp.type4_loop_size;

        double t0 = GetTime();
        fast_copy_vector_to_host(tmp.is_extend_seed, align_tmp_res.is_extend_seed);
        fast_copy_vector_to_host(tmp.consistent_nam, align_tmp_res.consistent_nam);
        fast_copy_vector_to_host(tmp.is_read1, align_tmp_res.is_read1);
        fast_copy_vector_to_host(tmp.type4_nams, align_tmp_res.type4_nams);
        fast_copy_vector_to_host(tmp.todo_nams, align_tmp_res.todo_nams);
        fast_copy_vector_to_host(tmp.done_align, align_tmp_res.done_align);
        gpu_cost11_copy1 += GetTime() - t0;

        t0 = GetTime();
        assert(tmp.align_res.length == tmp.cigar_info.length);
//        for (int j = 0; j < tmp.cigar_info.length; j++) {
//            cigar_size += tmp.cigar_info[j].cigar[0];
//            mx_cigar_size = std::max(mx_cigar_size, (uint64_t)tmp.cigar_info[j].cigar[0]);
//        }
        for (int j = 0; j < tmp.align_res.length; j++) {
            Cigar host_cigar;
//            for (int k = 0; k < tmp.cigar_info[j].cigar[0]; k++) {
//                host_cigar.m_ops.push_back(tmp.cigar_info[j].cigar[k + 1]);
//            }
            host_cigar.m_ops.resize(tmp.cigar_info[j].cigar[0]);
            memcpy(host_cigar.m_ops.data(), tmp.cigar_info[j].cigar + 1, sizeof(int) * tmp.cigar_info[j].cigar[0]);

            align_tmp_res.align_res.push_back({
                tmp.align_res[j].ref_id,
                tmp.align_res[j].ref_start,
                host_cigar,
                tmp.align_res[j].edit_distance,
                tmp.align_res[j].global_ed,
                tmp.align_res[j].score,
                tmp.align_res[j].length,
                tmp.align_res[j].is_rc,
                tmp.align_res[j].is_unaligned,
                tmp.align_res[j].gapped
            });
        }
        align_tmp_results.push_back(align_tmp_res);
        gpu_cost11_copy2 += GetTime() - t0;
    }
//    printf("cigar_size = %lu, mx = %lu\n", cigar_size, mx_cigar_size);
}

void copy_align_res(GPUAlignTmpRes* global_align_res, int batch_size, std::vector<AlignTmpRes>& align_tmp_results) {
    assert(align_tmp_results.size() == 0);
    for (int i = 0; i < batch_size; ++i) {
        const GPUAlignTmpRes& tmp = global_align_res[i];

        AlignTmpRes align_tmp_res;
        align_tmp_res.type = tmp.type;
        align_tmp_res.mapq1 = tmp.mapq1;
        align_tmp_res.mapq2 = tmp.mapq2;
        align_tmp_res.type4_loop_size = tmp.type4_loop_size;

        double t0 = GetTime();
        std::vector<int> is_extend_seeds = copy_vector_to_host(tmp.is_extend_seed);
        std::vector<int> consistent_nams = copy_vector_to_host(tmp.consistent_nam);
        std::vector<int> is_read1s       = copy_vector_to_host(tmp.is_read1);
        std::vector<Nam> type4_nams      = copy_vector_to_host(tmp.type4_nams);
        std::vector<Nam> todo_nams       = copy_vector_to_host(tmp.todo_nams);
        std::vector<int> done_flags      = copy_vector_to_host(tmp.done_align);
        std::vector<GPUAlignment> aligns = copy_vector_to_host(tmp.align_res);
        gpu_cost11_copy1 += GetTime() - t0;

        t0 = GetTime();
        align_tmp_res.is_extend_seed.assign(is_extend_seeds.begin(), is_extend_seeds.end());
        align_tmp_res.consistent_nam.assign(consistent_nams.begin(), consistent_nams.end());
        align_tmp_res.is_read1.assign(is_read1s.begin(), is_read1s.end());
        align_tmp_res.type4_nams.assign(type4_nams.begin(), type4_nams.end());
        align_tmp_res.todo_nams.assign(todo_nams.begin(), todo_nams.end());
        align_tmp_res.done_align.assign(done_flags.begin(), done_flags.end());
        for (int j = 0; j < aligns.size(); j++) {
            align_tmp_res.align_res.push_back({
                aligns[j].ref_id,
                aligns[j].ref_start,
                Cigar(),
                aligns[j].edit_distance,
                aligns[j].global_ed,
                aligns[j].score,
                aligns[j].length,
                aligns[j].is_rc,
                aligns[j].is_unaligned,
                aligns[j].gapped
            });
        }
        gpu_cost11_copy2 += GetTime() - t0;

        align_tmp_results.push_back(align_tmp_res);
    }
}


struct ThreadContext {
    int device_id;
    hipStream_t stream;

    ThreadContext(int tid, int gpuid) {
        device_id = gpuid;
        hipSetDevice(device_id);
        hipStreamCreate(&stream);
    }

    ~ThreadContext() {
        hipSetDevice(device_id);
        hipStreamDestroy(stream);
    }
};

#define batch_size 200000ll
#define batch_seq_szie batch_size * 160ll

void GPU_part2_rescue_mate_get_str(
    std::vector<std::string>& todo_querys,
    std::vector<std::string>& todo_refs,
    GPUAlignTmpRes& align_tmp_res,
    int j,
    Read &read1,
    Read &read2,
    const References& references,
    const Aligner& aligner,
    float mu,
    float sigma
) {
    Nam nam = align_tmp_res.todo_nams[j];
    Read read = align_tmp_res.is_read1[j] ? read1 : read2;
    int a, b;
    std::string r_tmp;
    auto read_len = read.size();

    if (nam.is_rc) {
        r_tmp = read.seq;
        a = nam.ref_start - nam.query_start - (mu + 5 * sigma);
        b = nam.ref_start - nam.query_start + read_len / 2;  // at most half read overlap
    } else {
        r_tmp = read.rc;                                              // mate is rc since fr orientation
        a = nam.ref_end + (read_len - nam.query_end) - read_len / 2;  // at most half read overlap
        b = nam.ref_end + (read_len - nam.query_end) + (mu + 5 * sigma);
    }

    auto ref_len = static_cast<int>(references.lengths[nam.ref_id]);
    auto ref_start = std::max(0, std::min(a, ref_len));
    auto ref_end = std::min(ref_len, std::max(0, b));

    std::string ref_segm = references.sequences[nam.ref_id].substr(ref_start, ref_end - ref_start);
    todo_querys.push_back(r_tmp);
    todo_refs.push_back(ref_segm);
}

void GPU_part2_extend_seed_get_str(
    std::vector<std::string>& todo_querys,
    std::vector<std::string>& todo_refs,
    GPUAlignTmpRes& align_tmp_res,
    int j,
    Read &read1,
    Read &read2,
    const References& references,
    const Aligner& aligner
) {
    Nam nam = align_tmp_res.todo_nams[j];
    Read read = align_tmp_res.is_read1[j] ? read1 : read2;
    AlignmentInfo info;
    int result_ref_start;
    const std::string query = nam.is_rc ? read.rc : read.seq;
    const std::string& ref = references.sequences[nam.ref_id];

    const auto projected_ref_start = std::max(0, nam.ref_start - nam.query_start);
    const auto projected_ref_end = std::min(nam.ref_end + query.size() - nam.query_end, ref.size());

    const int diff = std::abs(nam.ref_span() - nam.query_span());
    const int ext_left = std::min(50, projected_ref_start);
    const int ref_start = projected_ref_start - ext_left;
    const int ext_right = std::min(std::size_t(50), ref.size() - nam.ref_end);
    const auto ref_segm_size = read.size() + diff + ext_left + ext_right;
    const auto ref_segm = ref.substr(ref_start, ref_segm_size);
    todo_querys.push_back(query);
    todo_refs.push_back(ref_segm);
}

void GPU_part2_extend_seed_store_res(
    GPUAlignTmpRes& align_tmp_res,
    int j,
    const neoRcRef &read1,
    const neoRcRef &read2,
    const References& references,
    const AlignmentInfo info
) {
    Nam nam = align_tmp_res.todo_nams[j];
    const neoRcRef &read = align_tmp_res.is_read1[j] ? read1 : read2;
    int result_ref_start;
    size_t query_size = read.read.lseq;
    const std::string& ref = references.sequences[nam.ref_id];

    const auto projected_ref_start = std::max(0, nam.ref_start - nam.query_start);
    const auto projected_ref_end = std::min(nam.ref_end + query_size - nam.query_end, ref.size());

    const int diff = std::abs(nam.ref_span() - nam.query_span());
    const int ext_left = std::min(50, projected_ref_start);
    const int ref_start = projected_ref_start - ext_left;
    const int ext_right = std::min(std::size_t(50), ref.size() - nam.ref_end);
    const auto ref_segm_size = query_size + diff + ext_left + ext_right;
    result_ref_start = ref_start + info.ref_start;
    int softclipped = info.query_start + (query_size - info.query_end);
    GPUAlignment& alignment = align_tmp_res.align_res[j];
//    alignment.cigar = std::move(info.cigar);
    alignment.edit_distance = info.edit_distance;
    alignment.global_ed = info.edit_distance + softclipped;
    alignment.score = info.sw_score;
    alignment.ref_start = result_ref_start;
    alignment.length = info.ref_span();
    alignment.is_rc = nam.is_rc;
    alignment.is_unaligned = false;
    alignment.ref_id = nam.ref_id;
    alignment.gapped = true;

    if (info.cigar.m_ops.size() > MAX_CIGAR_ITEM) {
        printf("host cigar too big %d\n", info.cigar.m_ops.size());
    }
    align_tmp_res.cigar_info[j].cigar[0] = info.cigar.m_ops.size();
    for (int k = 0; k < info.cigar.m_ops.size(); k++) {
        align_tmp_res.cigar_info[j].cigar[k + 1] = info.cigar.m_ops[k];
    }
}

void GPU_part2_rescue_mate_store_res(
    GPUAlignTmpRes& align_tmp_res,
    int j,
    const neoRcRef &read1,
    const neoRcRef &read2,
    const References& references,
    const AlignmentInfo& info,
    float mu,
    float sigma
) {
    Nam nam = align_tmp_res.todo_nams[j];
    const neoRcRef &read = align_tmp_res.is_read1[j] ? read1 : read2;
    int a, b;
    auto read_len = read.read.lseq;

    if (nam.is_rc) {
        a = nam.ref_start - nam.query_start - (mu + 5 * sigma);
        b = nam.ref_start - nam.query_start + read_len / 2;  // at most half read overlap
    } else {
        a = nam.ref_end + (read_len - nam.query_end) - read_len / 2;  // at most half read overlap
        b = nam.ref_end + (read_len - nam.query_end) + (mu + 5 * sigma);
    }

    auto ref_len = static_cast<int>(references.lengths[nam.ref_id]);
    auto ref_start = std::max(0, std::min(a, ref_len));
    auto ref_end = std::min(ref_len, std::max(0, b));

    GPUAlignment& alignment = align_tmp_res.align_res[j];
//    alignment.cigar = info.cigar;
    alignment.edit_distance = info.edit_distance;
    alignment.score = info.sw_score;
    alignment.ref_start = ref_start + info.ref_start;
    alignment.is_rc = !nam.is_rc;
    alignment.ref_id = nam.ref_id;
    alignment.is_unaligned = info.cigar.empty();
    alignment.length = info.ref_span();
    if (info.cigar.m_ops.size() > MAX_CIGAR_ITEM) {
        printf("host cigar too big %d\n", info.cigar.m_ops.size());
    }
    align_tmp_res.cigar_info[j].cigar[0] = info.cigar.m_ops.size();
    for (int k = 0; k < info.cigar.m_ops.size(); k++) {
        align_tmp_res.cigar_info[j].cigar[k + 1] = info.cigar.m_ops[k];
    }

}

struct GPUScoredAlignmentPair {
    double score;
    std::pair<GPUAlignment, CigarData> alignment1;
    std::pair<GPUAlignment, CigarData> alignment2;
};

static inline float GPU_normal_pdf(float x, float mu, float sigma) {
    static const float inv_sqrt_2pi = 0.3989422804014327;
    const float a = (x - mu) / sigma;
    
    return inv_sqrt_2pi / sigma * std::exp(-0.5f * a * a);
}

static inline std::vector<GPUScoredAlignmentPair> GPU_get_best_scoring_pairs(
    const std::vector<std::pair<GPUAlignment, CigarData>>& alignments1,
    const std::vector<std::pair<GPUAlignment, CigarData>>& alignments2,
    float mu,
    float sigma
) {
    std::vector<GPUScoredAlignmentPair> pairs;
    for (auto& aa1 : alignments1) {
        for (auto& aa2 : alignments2) {
            GPUAlignment a1 = aa1.first;
            GPUAlignment a2 = aa2.first;
            float dist = std::abs(a1.ref_start - a2.ref_start);
            double score = a1.score + a2.score;
            if ((a1.is_rc ^ a2.is_rc) && (dist < mu + 4 * sigma)) {
                score += log(GPU_normal_pdf(dist, mu, sigma));
            } else {  // individual score
                // 10 corresponds to a value of log(GPU_normal_pdf(dist, mu, sigma)) of more than 4 stddevs away
                score -= 10;
            }
            pairs.push_back(GPUScoredAlignmentPair{score, aa1, aa2});
        }
    }

    return pairs;
}

void GPU_deduplicate_scored_pairs(std::vector<GPUScoredAlignmentPair>& pairs) {
    int prev_ref_start1 = pairs[0].alignment1.first.ref_start;
    int prev_ref_start2 = pairs[0].alignment2.first.ref_start;
    int prev_ref_id1 = pairs[0].alignment1.first.ref_id;
    int prev_ref_id2 = pairs[0].alignment2.first.ref_id;
    size_t j = 1;
    for (size_t i = 1; i < pairs.size(); i++) {
        int ref_start1 = pairs[i].alignment1.first.ref_start;
        int ref_start2 = pairs[i].alignment2.first.ref_start;
        int ref_id1 = pairs[i].alignment1.first.ref_id;
        int ref_id2 = pairs[i].alignment2.first.ref_id;
        if (ref_start1 != prev_ref_start1 || ref_start2 != prev_ref_start2 || ref_id1 != prev_ref_id1 ||
            ref_id2 != prev_ref_id2) {
            prev_ref_start1 = ref_start1;
            prev_ref_start2 = ref_start2;
            prev_ref_id1 = ref_id1;
            prev_ref_id2 = ref_id2;
            pairs[j] = pairs[i];
            j++;
        }
    }
    pairs.resize(j);
}

static std::pair<int, int> GPU_joint_mapq_from_high_scores(const std::vector<GPUScoredAlignmentPair>& pairs) {
    if (pairs.size() <= 1) {
        return std::make_pair(60, 60);
    }
    auto score1 = pairs[0].score;
    auto score2 = pairs[1].score;
    if (score1 == score2) {
        return std::make_pair(0, 0);
    }
    int mapq;
    const int diff = score1 - score2;  // (1.0 - (S1 - S2) / S1);
    //  float log10_p = diff > 6 ? -6.0 : -diff; // Corresponds to: p_error= 0.1^diff // change in sw score times rough illumina error rate. This is highly heauristic, but so seem most computations of mapq scores
    if (score1 > 0 && score2 > 0) {
        mapq = std::min(60, diff);
        //            mapq1 = -10 * log10_p < 60 ? -10 * log10_p : 60;
    } else if (score1 > 0 && score2 <= 0) {
        mapq = 60;
    } else {  // both negative SW one is better
        mapq = 1;
    }
    return std::make_pair(mapq, mapq);
}

bool GPU_is_proper_pair(const std::pair<GPUAlignment, CigarData>& alignment1, const std::pair<GPUAlignment, CigarData>& alignment2, float mu, float sigma) {
    const int dist = alignment2.first.ref_start - alignment1.first.ref_start;
    const bool same_reference = alignment1.first.ref_id == alignment2.first.ref_id;
    const bool both_aligned = same_reference && !alignment1.first.is_unaligned && !alignment2.first.is_unaligned;
    const bool r1_r2 = !alignment1.first.is_rc && alignment2.first.is_rc && dist >= 0; // r1 ---> <---- r2
    const bool r2_r1 = !alignment2.first.is_rc && alignment1.first.is_rc && dist <= 0; // r2 ---> <---- r1
    const bool rel_orientation_good = r1_r2 || r2_r1;
    const bool insert_good = std::abs(dist) <= mu + 6 * sigma;

    return both_aligned && insert_good && rel_orientation_good;
}

void GPU_rescue_read_last(
    int flag,
    GPUAlignTmpRes& align_tmp_res,
    const Read& read2,  // read to be rescued
    const Read& read1,  // read that has NAMs
    const Aligner& aligner,
    const References& references,
    std::array<Details, 2>& details,
    float mu,
    float sigma,
    size_t max_secondary,
    double secondary_dropoff,
    Sam& sam,
    const klibpp::KSeq& record1,
    const klibpp::KSeq& record2,
    bool swap_r1r2,  // TODO get rid of this
    std::minstd_rand& random_engine
) {
    std::vector<std::pair<GPUAlignment, CigarData>> alignments1;
    std::vector<std::pair<GPUAlignment, CigarData>> alignments2;
    int res_num = align_tmp_res.todo_nams.size();
    assert(res_num % 2 == 0);
    for (int i = 0; i < res_num; i += 2) {
        alignments1.push_back(std::make_pair(align_tmp_res.align_res[i], align_tmp_res.cigar_info[i]));
        alignments2.push_back(std::make_pair(align_tmp_res.align_res[i + 1], align_tmp_res.cigar_info[i + 1]));
        details[1].mate_rescue += !align_tmp_res.align_res[i + 1].is_unaligned;
        //        fprintf(stderr, "3 a1 score %d\n", align_tmp_res.align_res[i].score);
        //        fprintf(stderr, "3 a2 score %d\n", align_tmp_res.align_res[i + 1].score);
    }
    std::sort(alignments1.begin(), alignments1.end(),
              [](const std::pair<GPUAlignment, CigarData>& a,
                 const std::pair<GPUAlignment, CigarData>& b) {
                  return a.first.score > b.first.score;
              });
    std::sort(alignments2.begin(), alignments2.end(),
              [](const std::pair<GPUAlignment, CigarData>& a,
                 const std::pair<GPUAlignment, CigarData>& b) {
                  return a.first.score > b.first.score;
              });

    // Calculate best combined score here
    auto high_scores = GPU_get_best_scoring_pairs(alignments1, alignments2, mu, sigma);

    std::sort(high_scores.begin(), high_scores.end(),
              [](const GPUScoredAlignmentPair& a,
                 const GPUScoredAlignmentPair& b) {
                  return a.score > b.score;
              });
    GPU_deduplicate_scored_pairs(high_scores);
//    pick_random_top_pair(high_scores, random_engine);

    auto [mapq1, mapq2] = GPU_joint_mapq_from_high_scores(high_scores);

    // append both alignments to string here
    if (max_secondary == 0) {
        auto best_aln_pair = high_scores[0];
        std::pair<GPUAlignment, CigarData> alignment1 = best_aln_pair.alignment1;
        std::pair<GPUAlignment, CigarData> alignment2 = best_aln_pair.alignment2;
        if (swap_r1r2) {
            sam.add_pair(
                alignment2, alignment1, record2, record1, read2.rc, read1.rc, mapq2, mapq1,
                GPU_is_proper_pair(alignment2, alignment1, mu, sigma), true, details
            );
        } else {
            sam.add_pair(
                alignment1, alignment2, record1, record2, read1.rc, read2.rc, mapq1, mapq2,
                GPU_is_proper_pair(alignment1, alignment2, mu, sigma), true, details
            );
        }
    } else {
        auto max_out = std::min(high_scores.size(), (size_t)max_secondary);
        bool is_primary = true;
        auto best_aln_pair = high_scores[0];
        auto s_max = best_aln_pair.score;
        for (size_t i = 0; i < max_out; ++i) {
            if (i > 0) {
                is_primary = false;
                mapq1 = 0;
                mapq2 = 0;
            }
            auto aln_pair = high_scores[i];
            auto s_score = aln_pair.score;
            auto alignment1 = aln_pair.alignment1;
            auto alignment2 = aln_pair.alignment2;
            if (s_max - s_score < secondary_dropoff) {
                if (swap_r1r2) {
                    bool is_proper = GPU_is_proper_pair(alignment2, alignment1, mu, sigma);
                    std::array<Details, 2> swapped_details{details[1], details[0]};
                    sam.add_pair(
                        alignment2, alignment1, record2, record1, read2.rc, read1.rc, mapq2, mapq1,
                        is_proper, is_primary, swapped_details
                    );
                } else {
                    bool is_proper = GPU_is_proper_pair(alignment1, alignment2, mu, sigma);
                    sam.add_pair(
                        alignment1, alignment2, record1, record2, read1.rc, read2.rc, mapq1, mapq2,
                        is_proper, is_primary, details
                    );
                }
            } else {
                break;
            }
        }
    }
}

void GPU_align_PE_read_last(
    GPUAlignTmpRes& align_tmp_res,
    const neoRcRef &data1,
    const neoRcRef &data2,
    Sam& sam,
    std::string& outstring,
    InsertSizeDistribution& isize_est,
    const Aligner& aligner,
    const MappingParameters& map_param,
    const IndexParameters& index_parameters,
    const References& references,
    const StrobemerIndex& index,
    std::minstd_rand& random_engine
) {
    std::array<Details, 2> details;
    const auto mu = isize_est.mu;
    const auto sigma = isize_est.sigma;
    double secondary_dropoff = 2 * aligner.parameters.mismatch + aligner.parameters.gap_open;

    //    fprintf(stderr, "type %d\n", align_tmp_res.type);
    if (align_tmp_res.type == 0) {
        // None of the reads have any NAMs
        sam.add_unmapped_pair(data1.read, data2.read);
    } else if (align_tmp_res.type == 1) {
        auto record1 = gpu_ConvertNeo2KSeq(data1.read);
        auto record2 = gpu_ConvertNeo2KSeq(data2.read);
        Read read1(record1.seq);
        Read read2(record2.seq);
        GPU_rescue_read_last(
            1, align_tmp_res, read2, read1, aligner, references, details, mu,
            sigma, map_param.max_secondary, secondary_dropoff, sam, record1, record2, false, random_engine
        );
    } else if (align_tmp_res.type == 2) {
        auto record1 = gpu_ConvertNeo2KSeq(data1.read);
        auto record2 = gpu_ConvertNeo2KSeq(data2.read);
        Read read1(record1.seq);
        Read read2(record2.seq);
        GPU_rescue_read_last(
            2, align_tmp_res, read1, read2, aligner, references, details, mu,
            sigma, map_param.max_secondary, secondary_dropoff, sam, record2, record1, true, random_engine
        );
    } else if (align_tmp_res.type == 3) {
        assert(align_tmp_res.todo_nams.size() == 2);
        int mapq1 = align_tmp_res.mapq1;
        int mapq2 = align_tmp_res.mapq2;
        std::pair<GPUAlignment, CigarData> alignment1 = std::make_pair(align_tmp_res.align_res[0], align_tmp_res.cigar_info[0]);
        std::pair<GPUAlignment, CigarData> alignment2 = std::make_pair(align_tmp_res.align_res[1], align_tmp_res.cigar_info[1]);
        bool is_proper = GPU_is_proper_pair(alignment1, alignment2, mu, sigma);
        bool is_primary = true;
        sam.add_pair(
            alignment1, alignment2, data1.read, data2.read, data1.rc, data2.rc, mapq1, mapq2, is_proper, is_primary,
            details
        );
    } else if (align_tmp_res.type == 4) {
        int pos = 0;
        robin_hood::unordered_map<int, std::pair<GPUAlignment, CigarData>> is_aligned1;
        robin_hood::unordered_map<int, std::pair<GPUAlignment, CigarData>> is_aligned2;

        std::pair<GPUAlignment, CigarData> a1_indv_max, a2_indv_max;
        {

            auto n1_max = align_tmp_res.todo_nams[pos];
            //            fprintf(stderr, "get n1 %d from %d\n", n1_max.nam_id, pos);
            a1_indv_max = std::make_pair(align_tmp_res.align_res[pos], align_tmp_res.cigar_info[pos]);
            is_aligned1[n1_max.nam_id] = a1_indv_max;

            pos++;

            auto n2_max = align_tmp_res.todo_nams[pos];
            //            fprintf(stderr, "get n2 %d from %d\n", n2_max.nam_id, pos);
            a2_indv_max = std::make_pair(align_tmp_res.align_res[pos], align_tmp_res.cigar_info[pos]);
            is_aligned2[n2_max.nam_id] = a2_indv_max;

            pos++;
        }

        std::vector<GPUScoredAlignmentPair> high_scores;
        assert(align_tmp_res.type4_loop_size * 2 == align_tmp_res.type4_nams.size());

        for(int i = 0; i < align_tmp_res.type4_loop_size; i++) {
            Nam n1 = align_tmp_res.type4_nams[i * 2];
            Nam n2 = align_tmp_res.type4_nams[i * 2 + 1];

            std::pair<GPUAlignment, CigarData> a1;
            // ref_start == -1 is a marker for a dummy NAM
            if (n1.ref_start >= 0) {
                if (is_aligned1.find(n1.nam_id) != is_aligned1.end()) {
                    a1 = is_aligned1[n1.nam_id];
                } else {
                    a1 = std::make_pair(align_tmp_res.align_res[pos], align_tmp_res.cigar_info[pos]);
                    assert(n1.nam_id == align_tmp_res.todo_nams[pos].nam_id);
                    pos++;
                    is_aligned1[n1.nam_id] = a1;
                }
            } else {
                a1 = std::make_pair(align_tmp_res.align_res[pos], align_tmp_res.cigar_info[pos]);
                assert(n2.nam_id == align_tmp_res.todo_nams[pos].nam_id);
                pos++;
                details[0].mate_rescue += !a1.first.is_unaligned;
            }
            if (a1.first.score > a1_indv_max.first.score) {
                a1_indv_max = a1;
            }

            std::pair<GPUAlignment, CigarData> a2;
            // ref_start == -1 is a marker for a dummy NAM
            if (n2.ref_start >= 0) {
                if (is_aligned2.find(n2.nam_id) != is_aligned2.end()) {
                    //                    fprintf(stderr, "find n2 %d\n", n2.nam_id);
                    a2 = is_aligned2[n2.nam_id];
                } else {
                    //                    fprintf(stderr, "get n2 %d from %d\n", n2.nam_id, pos);
                    a2 = std::make_pair(align_tmp_res.align_res[pos], align_tmp_res.cigar_info[pos]);
                    assert(n2.nam_id == align_tmp_res.todo_nams[pos].nam_id);
                    pos++;
                    is_aligned2[n2.nam_id] = a2;
                }
            } else {
                a2 = std::make_pair(align_tmp_res.align_res[pos], align_tmp_res.cigar_info[pos]);
                assert(n1.nam_id == align_tmp_res.todo_nams[pos].nam_id);
                pos++;
                details[1].mate_rescue += !a2.first.is_unaligned;
            }
            if (a2.first.score > a2_indv_max.first.score) {
                a2_indv_max = a2;
            }

            bool r1_r2 = a2.first.is_rc && (a1.first.ref_start <= a2.first.ref_start) &&
                         ((a2.first.ref_start - a1.first.ref_start) < mu + 10 * sigma);  // r1 ---> <---- r2
            bool r2_r1 = a1.first.is_rc && (a2.first.ref_start <= a1.first.ref_start) &&
                         ((a1.first.ref_start - a2.first.ref_start) < mu + 10 * sigma);  // r2 ---> <---- r1

            double combined_score;
            if (r1_r2 || r2_r1) {
                // Treat a1/a2 as a pair
                float x = std::abs(a1.first.ref_start - a2.first.ref_start);
                combined_score = (double) a1.first.score + (double) a2.first.score +
                                 std::max(-20.0f + 0.001f, log(GPU_normal_pdf(x, mu, sigma)));
                //* (1 - s2 / s1) * min_matches * log(s1);
            } else {
                // Treat a1/a2 as two single-end reads
                // 20 corresponds to a value of log(GPU_normal_pdf(x, mu, sigma)) of more than 5 stddevs away (for most reasonable values of stddev)
                combined_score = (double) a1.first.score + (double) a2.first.score - 20;
            }

            GPUScoredAlignmentPair aln_pair{combined_score, a1, a2};
            high_scores.push_back(aln_pair);

        }
        assert(pos == align_tmp_res.todo_nams.size());

        // Finally, add highest scores of both mates as individually mapped
        double combined_score =
            (double) a1_indv_max.first.score + (double) a2_indv_max.first.score -
            20;  // 20 corresponds to  a value of log( GPU_normal_pdf(x, mu, sigma ) ) of more than 5 stddevs away (for most reasonable values of stddev)
        GPUScoredAlignmentPair aln_tuple{combined_score, a1_indv_max, a2_indv_max};
        high_scores.push_back(aln_tuple);

        std::sort(high_scores.begin(), high_scores.end(),
                  [](const GPUScoredAlignmentPair& a,
                     const GPUScoredAlignmentPair& b) {
                      return a.score > b.score;
                  });
        GPU_deduplicate_scored_pairs(high_scores);

        auto [mapq1, mapq2] = GPU_joint_mapq_from_high_scores(high_scores);
        auto best_aln_pair = high_scores[0];
        auto alignment1 = best_aln_pair.alignment1;
        auto alignment2 = best_aln_pair.alignment2;
        if (map_param.max_secondary == 0) {
            bool is_proper = GPU_is_proper_pair(alignment1, alignment2, mu, sigma);
            sam.add_pair(
                alignment1, alignment2, data1.read, data2.read, data1.rc, data2.rc, mapq1, mapq2, is_proper, true,
                details
            );

        } else {
            auto max_out = std::min(high_scores.size(), (size_t)map_param.max_secondary);
            // remove eventual duplicates - comes from, e.g., adding individual best alignments above (if identical to joint best alignment)
            float s_max = best_aln_pair.score;
            bool is_primary = true;
            for (size_t i = 0; i < max_out; ++i) {
                auto aln_pair = high_scores[i];
                alignment1 = aln_pair.alignment1;
                alignment2 = aln_pair.alignment2;
                float s_score = aln_pair.score;
                if (i > 0) {
                    is_primary = false;
                    mapq1 = 255;
                    mapq2 = 255;
                }

                if (s_max - s_score < secondary_dropoff) {
                    bool is_proper = GPU_is_proper_pair(alignment1, alignment2, mu, sigma);
                    sam.add_pair(
                        alignment1, alignment2, data1.read, data2.read, data1.rc, data2.rc, mapq1, mapq2, is_proper,
                        is_primary, details
                    );
                } else {
                    break;
                }
            }
        }
    }
}

#include <immintrin.h>

const __m256i map_table = _mm256_setr_epi8(
    gpu_nt2int_mod8[0], gpu_nt2int_mod8[1], gpu_nt2int_mod8[2], gpu_nt2int_mod8[3],
    gpu_nt2int_mod8[4], gpu_nt2int_mod8[5], gpu_nt2int_mod8[6], gpu_nt2int_mod8[7],
    gpu_nt2int_mod8[0], gpu_nt2int_mod8[1], gpu_nt2int_mod8[2], gpu_nt2int_mod8[3],
    gpu_nt2int_mod8[4], gpu_nt2int_mod8[5], gpu_nt2int_mod8[6], gpu_nt2int_mod8[7],
    gpu_nt2int_mod8[0], gpu_nt2int_mod8[1], gpu_nt2int_mod8[2], gpu_nt2int_mod8[3],
    gpu_nt2int_mod8[4], gpu_nt2int_mod8[5], gpu_nt2int_mod8[6], gpu_nt2int_mod8[7],
    gpu_nt2int_mod8[0], gpu_nt2int_mod8[1], gpu_nt2int_mod8[2], gpu_nt2int_mod8[3],
    gpu_nt2int_mod8[4], gpu_nt2int_mod8[5], gpu_nt2int_mod8[6], gpu_nt2int_mod8[7]
);

void print_m128i_bits(__m128i value) {
    alignas(32) uint8_t bytes[16];
    _mm_store_si128((__m128i*)bytes, value);

    for (int i = 0; i < 16; ++i) {
        for (int bit = 0; bit <= 7; bit++) {
            std::cout << ((bytes[i] >> bit) & 1);
        }
        std::cout << ' ';
    }
    std::cout << std::endl;
}

void print_m256i_bits(__m256i value) {
    alignas(32) uint8_t bytes[32];
    _mm256_store_si256((__m256i*)bytes, value);

    for (int i = 0; i < 32; ++i) {
        for (int bit = 0; bit <= 7; bit++) {
            std::cout << ((bytes[i] >> bit) & 1);
        }
        std::cout << ' ';
    }
    std::cout << std::endl;
}

void pack_sequence(const char* seq_ptr, int len, uint8_t * out_ptr, const int* pre_sum, int i) {
    int j = 0;
    const __m256i mask8 = _mm256_set1_epi32(0xFF);
    const __m256i mask2 = _mm256_set1_epi8(0x07);
    while (j + 32 <= len) {
//        for (int ii = 0; ii < 32; ii++) printf("%c", seq_ptr[j + ii]);
//        printf("\n");
        __m256i raw = _mm256_loadu_si256((const __m256i*)(seq_ptr + j));
//        printf("raw : ");
//        print_m256i_bits(raw);

        raw = _mm256_and_si256(raw, mask2);
        __m256i mapped = _mm256_shuffle_epi8(map_table, raw);
//        printf("mapped : ");
//        print_m256i_bits(mapped);

        __m256i shifted2 = _mm256_srli_epi32(mapped, 6);
        __m256i shifted4 = _mm256_srli_epi32(mapped, 12);
        __m256i shifted6 = _mm256_srli_epi32(mapped, 18);

        __m256i pack1 = _mm256_or_si256(mapped, shifted2);
        __m256i pack2 = _mm256_or_si256(shifted4, shifted6);
        __m256i packed = _mm256_or_si256(pack1, pack2);
//        printf("packed : ");
//        print_m256i_bits(packed);

        __m256i low8 = _mm256_and_si256(packed, mask8);
//        printf("low8 : ");
//        print_m256i_bits(low8);

        __m128i low = _mm256_castsi256_si128(low8);
        __m128i high = _mm256_extracti128_si256(low8, 1);

        __m128i low_bytes = _mm_shuffle_epi8(low, _mm_setr_epi8(
                                                      0, 4, 8, 12,
                                                      -1, -1, -1, -1,
                                                      -1, -1, -1, -1,
                                                      -1, -1, -1, -1
                                                  ));
//        print_m128i_bits(low_bytes);

        __m128i high_bytes = _mm_shuffle_epi8(high, _mm_setr_epi8(
                                                        0, 4, 8, 12,
                                                        -1, -1, -1, -1,
                                                        -1, -1, -1, -1,
                                                        -1, -1, -1, -1
                                                    ));
//        print_m128i_bits(high_bytes);


        __m128i merged = _mm_unpacklo_epi32(low_bytes, high_bytes);
//        printf("merged : ");
//        print_m128i_bits(merged);

        uint64_t result = _mm_cvtsi128_si64(merged);
        *(uint64_t*)(out_ptr + pre_sum[i] + (j / 4)) = result;

        j += 32;
    }

    for (; j < len; j += 4) {
        uint8_t packed = 0;
        for (int k = 0; k < 4; ++k) {
            if (j + k < len) {
                char c = seq_ptr[j + k];
                uint8_t code = gpu_nt2int_mod8[c & 7];
                packed |= (code << (k * 2));
            }
        }
        out_ptr[pre_sum[i] + j / 4] = packed;
    }
}


void GPU_align_PE(std::vector<neoRcRef> &data1s, std::vector<neoRcRef> &data2s,
                  ThreadContext& ctx, std::vector<AlignTmpRes> &align_tmp_results,
                  uint64_t* global_hits_num, uint64_t* global_nams_info, uint64_t* global_align_info,
                  const StrobemerIndex& index, AlignmentParameters *d_aligner, MappingParameters* d_map_param, IndexParameters *d_index_para,
                  GPUReferences *global_references, RefRandstrobe *d_randstrobes, my_bucket_index_t *d_randstrobe_start_indices,
                  my_vector<QueryRandstrobe> *global_randstrobes, int *global_todo_ids, int *global_randstrobe_sizes, uint64_t * global_hashes_value,
                  my_vector<my_pair<int, Hit>> *global_hits_per_ref0s, my_vector<my_pair<int, Hit>> *global_hits_per_ref1s, my_vector<Nam> *global_nams, GPUAlignTmpRes *global_align_res,
                  char *d_seq, int *d_len, int *d_pre_sum, char *h_seq, int *h_len, int *h_pre_sum) {

    assert(data1s.size() == data2s.size());
    assert(data1s.size() <= batch_size);

    double t0, t1;
    t0 = GetTime();
    int l_id, r_id, s_len;

    t1 = GetTime();
    uint64_t tot_len = 0;
    h_pre_sum[0] = 0;
    s_len = data1s.size();
    for (int i = 0; i < s_len * 4; i++) {
        int read_id = i % s_len;
        if (i < s_len) { // read1 seq
            h_len[i] = data1s[read_id].read.lseq;
            h_pre_sum[i + 1] = h_pre_sum[i] + h_len[i];
            char* seq_ptr = (char*)data1s[read_id].read.base + data1s[read_id].read.pseq;
            memcpy(h_seq + h_pre_sum[i], seq_ptr, h_len[i]);
        } else if (i < s_len * 2) { // read1 rc
            h_len[i] = data1s[read_id].read.lseq;
            h_pre_sum[i + 1] = h_pre_sum[i] + h_len[i];
            char* seq_ptr = data1s[read_id].rc;
            memcpy(h_seq + h_pre_sum[i], seq_ptr, h_len[i]);
        } else if (i < s_len * 3) { // read2 seq
            h_len[i] = data2s[read_id].read.lseq;
            h_pre_sum[i + 1] = h_pre_sum[i] + h_len[i];
            char* seq_ptr = (char*)data2s[read_id].read.base + data2s[read_id].read.pseq;
            memcpy(h_seq + h_pre_sum[i], seq_ptr, h_len[i]);
        } else { // read2 rc
            h_len[i] = data2s[read_id].read.lseq;
            h_pre_sum[i + 1] = h_pre_sum[i] + h_len[i];
            char* seq_ptr = data2s[read_id].rc;
            memcpy(h_seq + h_pre_sum[i], seq_ptr, h_len[i]);
        }
    }

    tot_len = h_pre_sum[s_len * 4];
    printf("cal tot len %llu\n", tot_len);

    gpu_copy1 += GetTime() - t1;

    t1 = GetTime();
    hipMemcpy(d_seq, h_seq, tot_len, hipMemcpyHostToDevice);
    hipMemcpy(d_len, h_len, s_len * sizeof(int) * 4 + 1, hipMemcpyHostToDevice);
    hipMemcpy(d_pre_sum, h_pre_sum, s_len * sizeof(int) * 4 + 1, hipMemcpyHostToDevice);
    gpu_copy2 += GetTime() - t1;

    for (l_id = 0; l_id < data1s.size(); l_id += batch_size) {
        r_id = l_id + batch_size;
        if (r_id > data1s.size()) r_id = data1s.size();
        s_len = r_id - l_id;

        char* local_d_seq = d_seq;
        int* local_d_len = d_len + l_id;
        int* local_d_pre_sum = d_pre_sum + l_id;


        for (int i = 0; i < s_len * 2; i++) {
            // check infos
            global_randstrobe_sizes[i] = 0;
            global_hashes_value[i] = 0;
            global_hits_num[i] = 0;
            global_nams_info[i] = 0;

            global_hits_per_ref0s[i].data = nullptr;
            global_hits_per_ref0s[i].length = 0;
            global_hits_per_ref1s[i].data = nullptr;
            global_hits_per_ref1s[i].length = 0;

            global_randstrobes[i].data = nullptr;
            global_randstrobes[i].length = 0;

            global_nams[i].data = nullptr;
            global_nams[i].length = 0;
        }

        t1 = GetTime();
        int threads_per_block;
        int reads_per_block;
        int blocks_per_grid;

        threads_per_block = 8;
        reads_per_block = threads_per_block * GPU_thread_task_size;
        blocks_per_grid = (s_len * 2 + reads_per_block - 1) / reads_per_block;
        gpu_get_randstrobes<<<blocks_per_grid, threads_per_block, 0, ctx.stream>>>(s_len * 2, local_d_pre_sum, local_d_len, local_d_seq, d_index_para,
                                                                    global_randstrobe_sizes, global_hashes_value, global_randstrobes);
        hipDeviceSynchronize();
        gpu_cost1 += GetTime() - t1;
        //printf("get randstrobe done\n");

        t1 = GetTime();

        double t11 = GetTime();
        threads_per_block = 8;
        reads_per_block = threads_per_block * GPU_thread_task_size;
        blocks_per_grid = (s_len * 2 + reads_per_block - 1) / reads_per_block;
        gpu_get_hits_pre<<<blocks_per_grid, threads_per_block, 0, ctx.stream>>>(index.bits, index.filter_cutoff, d_map_param->rescue_cutoff, d_randstrobes, index.randstrobes.size(), d_randstrobe_start_indices,
                                                                 s_len * 2, d_index_para, global_hits_num, global_randstrobes,
                                                                 global_hits_per_ref0s, global_hits_per_ref1s);
        hipDeviceSynchronize();
        gpu_cost2_1 += GetTime() - t11;
        //printf("get hits pre done\n");


        t11 = GetTime();
        threads_per_block = 8;
        reads_per_block = threads_per_block * GPU_thread_task_size;
        blocks_per_grid = (s_len * 2 + reads_per_block - 1) / reads_per_block;
        gpu_get_hits_after<<<blocks_per_grid, threads_per_block, 0, ctx.stream>>>(index.bits, index.filter_cutoff, d_map_param->rescue_cutoff, d_randstrobes, index.randstrobes.size(), d_randstrobe_start_indices,
                                                                   s_len * 2, d_index_para, global_hits_num, global_randstrobes,
                                                                   global_hits_per_ref0s, global_hits_per_ref1s);
        hipDeviceSynchronize();
        gpu_cost2_2 += GetTime() - t11;
        //printf("get hits after done\n");

        gpu_cost2 += GetTime() - t1;

        int todo_cnt = 0;
        for (int i = 0; i < s_len * 2; i++) {
            if (global_randstrobes[i].data == nullptr) { // pass filter
                global_todo_ids[todo_cnt] = i;
                todo_cnt++;
                assert(global_hits_per_ref0s[i].data != nullptr);
                assert(global_hits_per_ref1s[i].data != nullptr);
            } else {
                assert(global_hits_per_ref0s[i].data == nullptr);
                assert(global_hits_per_ref1s[i].data == nullptr);
            }
        }
    //    printf("normal read num %d\n", todo_cnt);

        t1 = GetTime();
        threads_per_block = 8;
        reads_per_block = threads_per_block * GPU_thread_task_size;
        blocks_per_grid = (todo_cnt + reads_per_block - 1) / reads_per_block;
        gpu_sort_hits<<<blocks_per_grid, threads_per_block, 0, ctx.stream>>>(todo_cnt, global_hits_per_ref0s, global_hits_per_ref1s, global_todo_ids);
        hipDeviceSynchronize();
        gpu_cost3 += GetTime() - t1;
    //    printf("sort hits done\n");

        t1 = GetTime();
        threads_per_block = 8;
        reads_per_block = threads_per_block * GPU_thread_task_size;
        blocks_per_grid = (todo_cnt + reads_per_block - 1) / reads_per_block;
        gpu_merge_hits_get_nams<<<blocks_per_grid, threads_per_block, 0, ctx.stream>>>(todo_cnt, d_index_para, global_nams_info,
                                                                        global_hits_per_ref0s, global_hits_per_ref1s, global_nams, global_todo_ids);
        hipDeviceSynchronize();
        gpu_cost4 += GetTime() - t1;
    //    printf("merge hits done\n");


        for (size_t i = 0; i < s_len * 2; ++i) {
            size_tot += global_randstrobe_sizes[i];
            check_sum += global_hashes_value[i];
            global_hits_num12 += global_hits_num[i];
            global_nams_info12 += global_nams_info[i];
        }

        todo_cnt = 0;
        for(int i = 0; i < s_len * 2; i++) {
            if (global_randstrobes[i].data != nullptr) {
                global_todo_ids[todo_cnt] = i;
                todo_cnt++;
            }
        }

        //printf("rescue read num %d\n", todo_cnt);

        for (int i = 0; i < s_len * 2; i++) {
            global_hits_num[i] = 0;
            global_nams_info[i] = 0;
            assert(global_hits_per_ref0s[i].data == nullptr);
            assert(global_hits_per_ref1s[i].data == nullptr);

            global_hits_per_ref0s[i].data = nullptr;
            global_hits_per_ref0s[i].length = 0;
            global_hits_per_ref1s[i].data = nullptr;
            global_hits_per_ref1s[i].length = 0;
        }

        t1 = GetTime();
        threads_per_block = 8;
        reads_per_block = threads_per_block * GPU_thread_task_size;
        blocks_per_grid = (todo_cnt + reads_per_block - 1) / reads_per_block;
        gpu_rescue_get_hits<<<blocks_per_grid, threads_per_block, 0, ctx.stream>>>(index.bits, index.filter_cutoff, d_map_param->rescue_cutoff, d_randstrobes, index.randstrobes.size(), d_randstrobe_start_indices,
                                                                    todo_cnt, d_index_para, global_hits_num, global_randstrobes,
                                                                    global_hits_per_ref0s, global_hits_per_ref1s, global_todo_ids);
        hipDeviceSynchronize();
        gpu_cost5 += GetTime() - t1;
        //printf("rescue get hits done\n");

        t1 = GetTime();
        threads_per_block = 8;
        reads_per_block = threads_per_block * GPU_thread_task_size;
        blocks_per_grid = (todo_cnt + reads_per_block - 1) / reads_per_block;
        gpu_rescue_sort_hits<<<blocks_per_grid, threads_per_block, 0, ctx.stream>>>(todo_cnt, global_hits_per_ref0s, global_hits_per_ref1s, global_todo_ids);
        hipDeviceSynchronize();
        gpu_cost6 += GetTime() - t1;
        //printf("rescue sort hits done\n");


        t1 = GetTime();
        threads_per_block = 8;
        reads_per_block = threads_per_block * GPU_thread_task_size;
        blocks_per_grid = (todo_cnt + reads_per_block - 1) / reads_per_block;
        gpu_rescue_merge_hits_get_nams<<<blocks_per_grid, threads_per_block, 0, ctx.stream>>>(todo_cnt, d_index_para, global_nams_info,
                                                                               global_hits_per_ref0s, global_hits_per_ref1s, global_nams, global_todo_ids);
        hipDeviceSynchronize();
        gpu_cost7 += GetTime() - t1;
        //printf("rescue merge hits done\n");

        for (int i = 0; i < s_len; i++) {
            global_align_info[i] = 0;
        }

        t1 = GetTime();
        threads_per_block = 8;
        reads_per_block = threads_per_block * GPU_thread_task_size;
        blocks_per_grid = (s_len * 2 + reads_per_block - 1) / reads_per_block;
        gpu_sort_nams<<<blocks_per_grid, threads_per_block, 0, ctx.stream>>>(s_len * 2, global_nams, d_map_param);
        hipDeviceSynchronize();
        gpu_cost8 += GetTime() - t1;
        //printf("sort nams done\n");

        t1 = GetTime();
        threads_per_block = 8;
        reads_per_block = threads_per_block * GPU_thread_task_size;
        blocks_per_grid = (s_len + reads_per_block - 1) / reads_per_block;
        gpu_pre_cal_type<<<blocks_per_grid, threads_per_block, 0, ctx.stream>>>(s_len, d_map_param->dropoff_threshold, global_nams, global_todo_ids);
        hipDeviceSynchronize();
        gpu_cost9 += GetTime() - t1;

        std::vector<int> types[5];
        for (int i = 0; i < s_len; i++) {
            assert(global_todo_ids[i] <= 4);
            types[global_todo_ids[i]].push_back(i);
            global_align_res[i].type = global_todo_ids[i];
            global_align_res[i].mapq1 = 0;
            global_align_res[i].mapq2 = 0;
            global_align_res[i].type4_loop_size = 0;
            global_align_res[i].is_extend_seed.length = 0;
            global_align_res[i].consistent_nam.length = 0;
            global_align_res[i].is_read1.length = 0;
            global_align_res[i].type4_nams.length = 0;
            global_align_res[i].todo_nams.length = 0;
            global_align_res[i].done_align.length = 0;
            global_align_res[i].align_res.length = 0;
            global_align_res[i].cigar_info.length = 0;
            global_align_res[i].todo_infos.length = 0;
        }
        //printf("types: %d %d %d %d %d\n", types[0].size(), types[1].size(), types[2].size(), types[3].size(), types[4].size());

        t1 = GetTime();

        t11 = GetTime();
        for (int i = 0; i < types[0].size(); i++) {
            global_todo_ids[i] = types[0][i];
        }
        threads_per_block = 8;
        reads_per_block = threads_per_block * GPU_thread_task_size;
        blocks_per_grid = (types[0].size() + reads_per_block - 1) / reads_per_block;
        gpu_align_PE0<<<blocks_per_grid, threads_per_block, 0, ctx.stream>>>(types[0].size(), s_len, d_index_para, global_align_info, d_aligner, local_d_pre_sum, local_d_len, local_d_seq,
                                                              global_references, d_map_param, global_nams, global_todo_ids, global_align_res);
        hipDeviceSynchronize();
        gpu_cost10_0 += GetTime() - t11;
	t11 = GetTime();
        for (int i = 0; i < types[1].size(); i++) {
            global_todo_ids[i] = types[1][i] * 2;
        }
        for (int i = 0; i < types[2].size(); i++) {
            global_todo_ids[i + types[1].size()] = types[2][i] * 2 + 1;
        }
        threads_per_block = 8;
        reads_per_block = threads_per_block * GPU_thread_task_size;
        blocks_per_grid = (types[1].size() + types[2].size() + reads_per_block - 1) / reads_per_block;
        gpu_align_PE12<<<blocks_per_grid, threads_per_block, 0, ctx.stream>>>(types[1].size() + types[2].size(), s_len, d_index_para, global_align_info, d_aligner, local_d_pre_sum, local_d_len, local_d_seq,
                                                              global_references, d_map_param, global_nams, global_todo_ids, global_align_res);
	hipDeviceSynchronize();
	gpu_cost10_1 += GetTime() - t11;

        t11 = GetTime();
        for (int i = 0; i < types[3].size(); i++) {
            global_todo_ids[i] = types[3][i];
        }
        threads_per_block = 8;
        reads_per_block = threads_per_block * GPU_thread_task_size;
        blocks_per_grid = (types[3].size() + reads_per_block - 1) / reads_per_block;
        gpu_align_PE3<<<blocks_per_grid, threads_per_block, 0, ctx.stream>>>(types[3].size(), s_len, d_index_para, global_align_info, d_aligner, local_d_pre_sum, local_d_len, local_d_seq,
                                                              global_references, d_map_param, global_nams, global_todo_ids, global_align_res);
        hipDeviceSynchronize();
        gpu_cost10_3 += GetTime() - t11;

        t11 = GetTime();
        std::vector<std::pair<int, int>> nams_id;
        for (int i = 0; i < types[4].size(); i++) {
            int id1 = types[4][i];
            int id2 = types[4][i] + s_len;
            nams_id.push_back(std::make_pair(global_nams[id1].length + global_nams[id2].length, types[4][i]));
        }
        std::sort(nams_id.begin(), nams_id.end());
        for (int i = 0; i < types[4].size(); i++) {
            global_todo_ids[i] = nams_id[i].second;
        }
        threads_per_block = 4;
        reads_per_block = threads_per_block * GPU_thread_task_size;
        blocks_per_grid = (types[4].size() + reads_per_block - 1) / reads_per_block;
        gpu_align_PE4<<<blocks_per_grid, threads_per_block, 0, ctx.stream>>>(types[4].size(), s_len, d_index_para, global_align_info, d_aligner, d_pre_sum, local_d_len, local_d_seq,
                                                              global_references, d_map_param, global_nams, global_todo_ids, global_align_res);
        hipDeviceSynchronize();
        gpu_cost10_4 += GetTime() - t11;

        gpu_cost10 += GetTime() - t1;
        //printf("align done\n");


        for (int i = 0; i < s_len * 2; ++i) {
            global_hits_num3 += global_hits_num[i];
            global_nams_info3 += global_nams_info[i];
        }
        for (int i = 0; i < s_len; i++) {
            global_align_info123 += global_align_info[i];
        }
    }

    tot_cost += GetTime() - t0;

}

std::once_flag init_flag_ref[4];
std::once_flag init_flag_pool[4];

GPUReferences *global_references[4];
RefRandstrobe *d_randstrobes[4];
my_bucket_index_t *d_randstrobe_start_indices[4];

void init_shared_data(const References& references, const StrobemerIndex& index, const int gpu_id, int thread_id) {
    hipSetDevice(gpu_id);
    printf("init_shared_data thread_id = %d, gpu_id = %d\n", thread_id, gpu_id);
    hipMallocManaged(&global_references[gpu_id], sizeof(GPUReferences));
    global_references[gpu_id]->num_refs = references.size();
    hipMalloc(&global_references[gpu_id]->sequences.data, references.size() * sizeof(my_string));
    global_references[gpu_id]->sequences.length = references.size();
    global_references[gpu_id]->sequences.capacity = references.size();
    for (int i = 0; i < references.size(); i++) {
        my_string ref;
        ref.slen = references.lengths[i];
        hipMalloc(&ref.data, references.lengths[i]);
        hipMemcpy(ref.data, references.sequences[i].data(), references.lengths[i], hipMemcpyHostToDevice);
        hipMemcpy(global_references[gpu_id]->sequences.data + i, &ref, sizeof(my_string), hipMemcpyHostToDevice);
    }
    hipMalloc(&global_references[gpu_id]->lengths.data, references.size() * sizeof(int));
    hipMemcpy(global_references[gpu_id]->lengths.data, references.lengths.data(), references.size() * sizeof(int), hipMemcpyHostToDevice);
    global_references[gpu_id]->lengths.length = references.size();
    global_references[gpu_id]->lengths.capacity = references.size();

    hipMalloc(&d_randstrobes[gpu_id], index.randstrobes.size() * sizeof(RefRandstrobe));
    hipMalloc(&d_randstrobe_start_indices[gpu_id], index.randstrobe_start_indices.size() * sizeof(my_bucket_index_t));
    hipMemset(d_randstrobes[gpu_id], 0, index.randstrobes.size() * sizeof(RefRandstrobe));
    hipMemset(d_randstrobe_start_indices[gpu_id], 0, index.randstrobe_start_indices.size() * sizeof(my_bucket_index_t));
    hipMemcpy(d_randstrobes[gpu_id], index.randstrobes.data(), index.randstrobes.size() * sizeof(RefRandstrobe), hipMemcpyHostToDevice);
    hipMemcpy(d_randstrobe_start_indices[gpu_id], index.randstrobe_start_indices.data(), index.randstrobe_start_indices.size() * sizeof(my_bucket_index_t), hipMemcpyHostToDevice);
}

void init_mm_safe(uint64_t num_bytes, uint64_t seed, int gpu_id, int thread_id) {
    printf("init_mm_safe thread_id = %d, gpu_id = %d\n", thread_id, gpu_id);
    init_mm(num_bytes, seed);
}

void copy_GPUAlignTmpRes_to_AlignTmpRes(const GPUAlignTmpRes& src, AlignTmpRes& dst) {
    dst.type = src.type;
    dst.mapq1 = src.mapq1;
    dst.mapq2 = src.mapq2;
    dst.type4_loop_size = src.type4_loop_size;

    dst.is_extend_seed.assign(src.is_extend_seed.data, src.is_extend_seed.data + src.is_extend_seed.size());
    dst.consistent_nam.assign(src.consistent_nam.data, src.consistent_nam.data + src.consistent_nam.size());
    dst.is_read1.assign(src.is_read1.data, src.is_read1.data + src.is_read1.size());
    dst.type4_nams.assign(src.type4_nams.data, src.type4_nams.data + src.type4_nams.size());
    dst.todo_nams.assign(src.todo_nams.data, src.todo_nams.data + src.todo_nams.size());
    dst.done_align.assign(src.done_align.data, src.done_align.data + src.done_align.size());


    dst.align_res.resize(src.align_res.size());
    for (size_t i = 0; i < src.align_res.size(); ++i) {
        const GPUAlignment& ga = src.align_res[i];
        Alignment& a = dst.align_res[i];

        a.ref_id = ga.ref_id;
        a.ref_start = ga.ref_start;
        // a.cigar 不赋值，忽略
        a.edit_distance = ga.edit_distance;
        a.global_ed = ga.global_ed;
        a.score = ga.score;
        a.length = ga.length;
        a.is_rc = ga.is_rc;
        a.is_unaligned = ga.is_unaligned;
        a.gapped = ga.gapped;
    }
}

void PrintStr(const char* str, int len) {
    for(int i = 0; i < len; i++) printf("%c", str[i]);
    printf("\n");
}


void perform_task_async_pe_fx_GPU(
    InputBuffer& input_buffer,
    OutputBuffer& output_buffer,
    AlignmentStatistics& statistics,
    int& done,
    const AlignmentParameters& aln_params,
    MappingParameters map_param,
    const IndexParameters& index_parameters,
    const References& references,
    const StrobemerIndex& index,
    const std::string& read_group_id,
    const int thread_id,
    rabbit::fq::FastqDataPool& fastqPool,
    rabbit::core::TDataQueue<rabbit::fq::FastqDataPairChunk> &dq,
    bool use_good_numa,
    int gpu_id
) {

    if(use_good_numa) {
        cpu_set_t cpuset;
        CPU_ZERO(&cpuset);
        CPU_SET(thread_id, &cpuset);
        pthread_t current_thread = pthread_self();
        if (pthread_setaffinity_np(current_thread, sizeof(cpu_set_t), &cpuset) != 0) {
            std::cerr << "Error setting thread affinity" << std::endl;
        }
    }

    //map_param.max_tries = 2;
    //printf("thread %d--%d\n", thread_id, gpu_id);
    ThreadContext ctx(thread_id, gpu_id);

    bool eof = false;
    Aligner aligner{aln_params};
    std::minstd_rand random_engine;
    std::minstd_rand pre_random_engine;
    size_t chunk_index = 0;
    std::vector<AlignTmpRes> align_tmp_results;
    thread_local double time_tot = 0;
    thread_local double time0 = 0;
    thread_local double time0_1 = 0;
    thread_local double time0_2 = 0;
    thread_local double time0_3 = 0;
    thread_local double time0_4 = 0;
    thread_local double time1 = 0;    //time except extend and output
    thread_local double time1_1 = 0;    //time except extend and output
    thread_local double time1_1_1 = 0;
    thread_local double time1_2 = 0;    //time except extend and output
    thread_local double time1_3 = 0;
    thread_local double time2_1 = 0;  //time to filter nams and get todo_strings
    thread_local double time2_1_1 = 0;
    thread_local double time2_2 = 0;  //time to do ssw on gpu
    thread_local double time2_3 = 0;  //time to post-process the gpu results
    thread_local double time2_4 = 0;  //time to store ssw results
    thread_local double time3_1 = 0;  //time to construct sam
    thread_local double time3_2 = 0;  //time to output
    thread_local double time4 = 0;

    double t_0, t_1, t_2;


    t_0 = GetTime();

    t_1 = GetTime();

    t_2 = GetTime();
    uint64_t num_bytes = 24 * 1024ll * 1024ll * 1024ll;
    uint64_t seed = 13;
    std::call_once(init_flag_pool[gpu_id], init_mm_safe, num_bytes, seed, gpu_id, thread_id);
    time0_1 += GetTime() - t_2;

    t_2 = GetTime();
    AlignmentParameters *d_aligner;
    hipMallocManaged(&d_aligner, sizeof(AlignmentParameters));
    hipMemcpy(d_aligner, &aln_params, sizeof(AlignmentParameters), hipMemcpyHostToDevice);
    MappingParameters* d_map_param;
    hipMallocManaged(&d_map_param, sizeof(MappingParameters));
    hipMemcpy(d_map_param, &map_param, sizeof(MappingParameters), hipMemcpyHostToDevice);
    IndexParameters *d_index_para;
    hipMallocManaged(&d_index_para, sizeof(IndexParameters));
    hipMemcpy(d_index_para, &index_parameters, sizeof(IndexParameters), hipMemcpyHostToDevice);
//    std::call_once(init_flag_ref[gpu_id], init_shared_data, references, index, gpu_id, thread_id);
    time0_2 += GetTime() - t_2;


    t_2 = GetTime();
    my_vector<QueryRandstrobe> *global_randstrobes;
    hipMallocManaged(&global_randstrobes, batch_size * 2 * sizeof(my_vector<QueryRandstrobe>));
    int *global_todo_ids;
    hipMallocManaged(&global_todo_ids, batch_size * 2 * sizeof(int));
    int *global_randstrobe_sizes;
    hipMallocManaged(&global_randstrobe_sizes, batch_size * 2 * sizeof(int));
    uint64_t * global_hashes_value;
    hipMallocManaged(&global_hashes_value, batch_size * 2 * sizeof(uint64_t));
    my_vector<my_pair<int, Hit>> *global_hits_per_ref0s;
    hipMallocManaged(&global_hits_per_ref0s, batch_size * 2 * sizeof(my_vector<my_pair<int, Hit>>));
    my_vector<my_pair<int, Hit>> *global_hits_per_ref1s;
    hipMallocManaged(&global_hits_per_ref1s, batch_size * 2 * sizeof(my_vector<my_pair<int, Hit>>));
    my_vector<Nam> *global_nams;
    hipMallocManaged(&global_nams, batch_size * 2 * sizeof(my_vector<Nam>));
    GPUAlignTmpRes *global_align_res;
    hipMallocManaged(&global_align_res, batch_size * 2 * sizeof(GPUAlignTmpRes));
    uint64_t pre_vec_size = 4 * sizeof(int) + 2 * sizeof(Nam) + sizeof(GPUAlignment) + sizeof(CigarData) + sizeof(TODOInfos);
    uint64_t global_align_res_data_size = batch_size * MAX_TRIES_LIMIT2 * pre_vec_size;
    printf("global_align_res_data_size -- %llu\n", global_align_res_data_size);
    char *global_align_res_data;
    hipMallocManaged(&global_align_res_data, global_align_res_data_size);
    for (int i = 0; i < batch_size; i++) {
        GPUAlignTmpRes *tmp = global_align_res + i;
        tmp->type = 0, tmp->mapq1 = 0, tmp->mapq2 = 0, tmp->type4_loop_size = 0;
        char* base_ptr = global_align_res_data + i * MAX_TRIES_LIMIT2 * pre_vec_size;

        tmp->is_extend_seed.data = (int*)base_ptr;
        tmp->is_extend_seed.length = 0;
        tmp->is_extend_seed.capacity = MAX_TRIES_LIMIT2;
        base_ptr += MAX_TRIES_LIMIT2 * sizeof(int);

        tmp->consistent_nam.data = (int*)base_ptr;
        tmp->consistent_nam.length = 0;
        tmp->consistent_nam.capacity = MAX_TRIES_LIMIT2;
        base_ptr += MAX_TRIES_LIMIT2 * sizeof(int);

        tmp->is_read1.data = (int*)base_ptr;
        tmp->is_read1.length = 0;
        tmp->is_read1.capacity = MAX_TRIES_LIMIT2;
        base_ptr += MAX_TRIES_LIMIT2 * sizeof(int);

        tmp->type4_nams.data = (Nam*)base_ptr;
        tmp->type4_nams.length = 0;
        tmp->type4_nams.capacity = MAX_TRIES_LIMIT2;
        base_ptr += MAX_TRIES_LIMIT2 * sizeof(Nam);

        tmp->todo_nams.data = (Nam*)base_ptr;
        tmp->todo_nams.length = 0;
        tmp->todo_nams.capacity = MAX_TRIES_LIMIT2;
        base_ptr += MAX_TRIES_LIMIT2 * sizeof(Nam);

        tmp->done_align.data = (int*)base_ptr;
        tmp->done_align.length = 0;
        tmp->done_align.capacity = MAX_TRIES_LIMIT2;
        base_ptr += MAX_TRIES_LIMIT2 * sizeof(int);

        tmp->align_res.data = (GPUAlignment*)base_ptr;
        tmp->align_res.length = 0;
        tmp->align_res.capacity = MAX_TRIES_LIMIT2;
        base_ptr += MAX_TRIES_LIMIT2 * sizeof(GPUAlignment);

        tmp->cigar_info.data = (CigarData*)base_ptr;
        tmp->cigar_info.length = 0;
        tmp->cigar_info.capacity = MAX_TRIES_LIMIT2;
        base_ptr += MAX_TRIES_LIMIT2 * sizeof(CigarData);

        tmp->todo_infos.data = (TODOInfos*)base_ptr;
        tmp->todo_infos.length = 0;
        tmp->todo_infos.capacity = MAX_TRIES_LIMIT2;
        base_ptr += MAX_TRIES_LIMIT2 * sizeof(TODOInfos);

        assert(base_ptr - global_align_res_data == (i + 1) * MAX_TRIES_LIMIT2 * pre_vec_size);
    }
    time0_3 += GetTime() - t_2;


    t_2 = GetTime();
    const int seq_size_alloc = batch_seq_szie;
    char *d_seq;
    int *d_len;
    int *d_pre_sum;
    hipHostAlloc(&d_seq, seq_size_alloc * 4, hipHostMallocDefault);
    hipMemset(d_seq, 0, seq_size_alloc * 4);
    hipHostAlloc(&d_len, (batch_size + 1) * sizeof(int) * 4, hipHostMallocDefault);
    hipMemset(d_len, 0, (batch_size + 1) * sizeof(int) * 4);
    hipHostAlloc(&d_pre_sum, (batch_size + 1) * sizeof(int) * 4, hipHostMallocDefault);
    hipMemset(d_pre_sum, 0, (batch_size + 1) * sizeof(int) * 4);

    int *h_len = new int[(batch_size + 1) * 4];
    int *h_pre_sum = new int[(batch_size + 1) * 4];
    char *h_seq = new char[seq_size_alloc * 4];

    uint64_t * global_hits_num;
    hipMallocManaged(&global_hits_num, batch_size * 2 * sizeof(uint64_t));

    uint64_t * global_nams_info;
    hipMallocManaged(&global_nams_info, batch_size * 2 * sizeof(uint64_t));

    uint64_t * global_align_info;
    hipMallocManaged(&global_align_info, batch_size * sizeof(uint64_t));
    time0_4 += GetTime() - t_2;


    time0 += GetTime() - t_1;


    std::vector<std::string_view> todo_querys;
    std::vector<std::string_view> todo_refs;
    std::vector<std::string> h_todo_querys;
    std::vector<std::string> h_todo_refs;
    std::vector<AlignmentInfo> info_results;
    std::vector<gasal_tmp_res> gasal_results_tmp;
    std::vector<gasal_tmp_res> gasal_results;
    std::vector<neoRcRef> data1s;
    std::vector<neoRcRef> data2s;
    std::vector<neoReference> neo_data1s;
    std::vector<neoReference> neo_data2s;

    char* rc_data1 = new char[batch_seq_szie];
    char* rc_data2 = new char[batch_seq_szie];

    while (!eof) {
        todo_querys.clear();
        todo_refs.clear();
        info_results.clear();
        gasal_results_tmp.clear();
        gasal_results.clear();
        data1s.clear();
        data2s.clear();

        rabbit::fq::FastqDataPairChunk *fqdatachunks[128];

        InsertSizeDistribution isize_est;
        int real_chunk_num = 0;
        int chunk_num = rand() % 8 + 8 + 1;
//        int chunk_num = 1;
        //find nams
        {
            t_1 = GetTime();
            bool res;
            rabbit::int64 id;
            t_2 = GetTime();
            int rc_pos1 = 0, rc_pos2 = 0;
            for (int chunk_id = 0; chunk_id < chunk_num; chunk_id++) {
                res = dq.Pop(id, fqdatachunks[chunk_id]);
                if(res) {
                    neo_data1s.clear();
                    neo_data2s.clear();
                    rabbit::fq::chunkFormat((rabbit::fq::FastqDataChunk*)(fqdatachunks[chunk_id]->left_part), neo_data1s);
                    rabbit::fq::chunkFormat((rabbit::fq::FastqDataChunk*)(fqdatachunks[chunk_id]->right_part), neo_data2s);
                    assert(neo_data1s.size() == neo_data2s.size());
                    double t_3 = GetTime();
                    for(int i = 0; i < neo_data1s.size(); i++) {
                        char* name1 = (char *) neo_data1s[i].base + neo_data1s[i].pname;
                        if(neo_data1s[i].lname > 0 && name1[0] == '@') {
                            neo_data1s[i].pname++;
                            neo_data1s[i].lname--;
                            name1++;
                        }
                        for(int j = 0; j < neo_data1s[i].lname; j++) {
                            if (name1[j] == ' ') {
                                neo_data1s[i].lname = j;
                                break;
                            }
                        }
                        char* name2 = (char *) neo_data2s[i].base + neo_data2s[i].pname;
                        if(neo_data2s[i].lname > 0 && name2[0] == '@') {
                            neo_data2s[i].pname++;
                            neo_data2s[i].lname--;
                            name2++;
                        }
                        for(int j = 0; j < neo_data2s[i].lname; j++) {
                            if (name2[j] == ' ') {
                                neo_data2s[i].lname = j;
                                break;
                            }
                        }
                        char* seq1 = (char *) neo_data1s[i].base + neo_data1s[i].pseq;
                        data1s.push_back({neo_data1s[i], rc_data1 + rc_pos1});
                        for (int j = 0; j < neo_data1s[i].lseq; j++) {
                            rc_data1[rc_pos1++] = rc_gpu_nt2int_mod8[seq1[neo_data1s[i].lseq - 1 - j] & 7];
                        }
                        char* seq2 = (char *) neo_data2s[i].base + neo_data2s[i].pseq;
                        data2s.push_back({neo_data2s[i], rc_data2 + rc_pos2});
                        for (int j = 0; j < neo_data2s[i].lseq; j++) {
                            rc_data2[rc_pos2++] = rc_gpu_nt2int_mod8[seq2[neo_data2s[i].lseq - 1 - j] & 7];
                        }
                    }
                    time1_1_1 += GetTime() - t_3;
                    real_chunk_num++;
                } else break;
            }
            assert(rc_pos1 <= batch_seq_szie && rc_pos2 <= batch_seq_szie);
//            printf("chunk size %d\n", neo_data1s.size());

            time1_1 += GetTime() - t_2;

            t_2 = GetTime();
            chunk_index = id;
            if (data1s.empty() && res == 0) eof = true;
            if (eof) break;
            // Use chunk index as random seed for reproducibility
            random_engine.seed(chunk_index);
            GPU_align_PE(data1s, data2s,
                         ctx,
                         align_tmp_results,
                         global_hits_num, global_nams_info, global_align_info,
                         index, d_aligner, d_map_param, d_index_para,
                         global_references[gpu_id], d_randstrobes[gpu_id], d_randstrobe_start_indices[gpu_id],
                         global_randstrobes, global_todo_ids, global_randstrobe_sizes, global_hashes_value,
                         global_hits_per_ref0s, global_hits_per_ref1s, global_nams, global_align_res,
                         d_seq, d_len, d_pre_sum, h_seq, h_len, h_pre_sum);
            time1_2 += GetTime() - t_2;
            time1 += GetTime() - t_1;
        }

        //process todo_nams
        {
            // step1 : filter nams and get todo_strings
            t_1 = GetTime();
            for (int i = 0; i < data1s.size(); i++) {
                GPUAlignTmpRes &align_tmp_res = global_align_res[i];
                for (int j = 0; j < align_tmp_res.todo_infos.size(); j++) {
                    TODOInfos& todo_info = align_tmp_res.todo_infos[j];
                    uint32_t info = todo_info.read_info;
                    int is_read1 = (info >> 31) & 0x1;
                    int is_rc    = (info >> 30) & 0x1;
                    int q_begin  = (info >> 15) & 0x7FFF;
                    int q_len    = info & 0x7FFF;
//                    size_t seq_len1, seq_len2;
//                    int s_len = data1s.size();
//                    seq_len1 = h_len[i];
//                    seq_len2 = h_len[i + s_len * 2];
//                    char *seq1, *seq2, *rc1, *rc2;
//                    seq1 = h_seq + h_pre_sum[i + s_len * 0];
//                    rc1  = h_seq + h_pre_sum[i + s_len * 1];
//                    seq2 = h_seq + h_pre_sum[i + s_len * 2];
//                    rc2  = h_seq + h_pre_sum[i + s_len * 3];
//                    const auto& query_seq = is_read1 ? (is_rc ? rc1 : seq1) :
//                                                       (is_rc ? rc2 : seq2);
//                    todo_querys.push_back(std::string_view(todo_info.seq + q_begin, q_len));
//                    const auto& ref_seq = global_references[gpu_id]->sequences[todo_info.ref_id];
//                    todo_refs.push_back(std::string_view(todo_info.ref + todo_info.r_begin, todo_info.r_len));

                    const auto& h_query_seq = is_read1 ? (is_rc ? data1s[i].rc : (char*)data1s[i].read.base + data1s[i].read.pseq) :
                                                   (is_rc ? data2s[i].rc : (char*)data2s[i].read.base + data2s[i].read.pseq);
                    const auto& h_ref_seq = references.sequences[todo_info.ref_id];
//                    h_todo_querys.push_back(std::string(h_query_seq + q_begin, q_len));
//                    h_todo_refs.push_back(std::string(h_ref_seq.c_str() + todo_info.r_begin, todo_info.r_len));
                    todo_querys.push_back(std::string_view(h_query_seq + q_begin, q_len));
                    todo_refs.push_back(std::string_view(h_ref_seq.c_str() + todo_info.r_begin, todo_info.r_len));
                }
            }
//            printf("todo size %d\n", todo_querys.size());
            assert(todo_querys.size() == todo_refs.size());
            time2_1 += GetTime() - t_1;

            // step2 : solve todo_strings -- do ssw on gpu -- key step, need async
            t_1 = GetTime();

            //std::thread gpu_ssw_async;
            //gpu_ssw_async = std::thread([&] (){
                //hipSetDevice(gpu_id);
                for (size_t i = 0; i + STREAM_BATCH_SIZE <= todo_querys.size(); i += STREAM_BATCH_SIZE) {
                    auto query_start = todo_querys.begin() + i;
                    auto query_end = query_start + STREAM_BATCH_SIZE;
                    std::vector<std::string_view> query_batch(query_start, query_end);
                    auto ref_start = todo_refs.begin() + i;
                    auto ref_end = ref_start + STREAM_BATCH_SIZE;
                    std::vector<std::string_view> ref_batch(ref_start, ref_end);
                    solve_ssw_on_gpu2(
                        thread_id, gasal_results_tmp, query_batch, ref_batch, aln_params.match,
                        aln_params.mismatch, aln_params.gap_open, aln_params.gap_extend
                    );
                    gasal_results.insert(gasal_results.end(), gasal_results_tmp.begin(), gasal_results_tmp.end());
                }
                size_t remaining = todo_querys.size() % STREAM_BATCH_SIZE;
                if (remaining > 0) {
                    auto query_start = todo_querys.end() - remaining;
                    std::vector<std::string_view> query_batch(query_start, todo_querys.end());
                    auto ref_start = todo_refs.end() - remaining;
                    std::vector<std::string_view> ref_batch(ref_start, todo_refs.end());
                    solve_ssw_on_gpu2(
                        thread_id, gasal_results_tmp, query_batch, ref_batch, aln_params.match,
                        aln_params.mismatch, aln_params.gap_open, aln_params.gap_extend
                    );
                    gasal_results.insert(gasal_results.end(), gasal_results_tmp.begin(), gasal_results_tmp.end());
                }
            //});
            //gpu_ssw_async.join();

//            printf("GPU2 done\n");
            time2_2 += GetTime() - t_1;
        }


        //post-process ssw results and trans to sam
        {
            // step1 : post-process the gpu results, re-ssw for bad results on cpu
            t_1 = GetTime();
            uint64_t seq_todo_size = 0;
            uint64_t ref_todo_size = 0;
            info_results.resize(todo_querys.size());
            for (size_t i = 0; i < todo_querys.size(); i++) {
                AlignmentInfo info;
//                std::string& todo_q = h_todo_querys[i];
//                std::string& todo_r = h_todo_refs[i];
//                std::string todo_q = std::string(todo_querys[i]);
//                std::string todo_r = std::string(todo_refs[i]);
                const auto& todo_q = todo_querys[i];
                const auto& todo_r = todo_refs[i];
                seq_todo_size += todo_q.length();
                ref_todo_size += todo_r.length();
                if (gasal_fail(todo_q, todo_r, gasal_results[i])) {
//                if (1) {
                    info = aligner.align(todo_q, todo_r);
                } else {
                    info = aligner.align_gpu(todo_q, todo_r, gasal_results[i]);
                }
                info_results[i] = info;
            }
            //printf("chunk todo size %lld %lld\n", seq_todo_size, ref_todo_size);
            time2_3 += GetTime() - t_1;

            // step2 : store ssw results
            t_1 = GetTime();
            int pos = 0;

            for (size_t i = 0; i < data1s.size(); i++) {
                const auto mu = isize_est.mu;
                const auto sigma = isize_est.sigma;
                GPUAlignTmpRes& align_tmp_res = global_align_res[i];
                size_t todo_size = align_tmp_res.todo_nams.size();
                if (align_tmp_res.type == 1 || align_tmp_res.type == 2) {
                    for (size_t j = 0; j < todo_size; j += 2) {
                        if (!align_tmp_res.done_align[j]) {
                            GPU_part2_extend_seed_store_res(
                                align_tmp_res, j, data1s[i], data2s[i], references, info_results[pos++]
                            );
                        }
                        if (!align_tmp_res.done_align[j + 1]) {
                            GPU_part2_rescue_mate_store_res(
                                align_tmp_res, j + 1, data1s[i], data2s[i], references, info_results[pos++], mu, sigma
                            );
                        }
                    }
                } else if (align_tmp_res.type == 3) {
                    if (!align_tmp_res.done_align[0]) {
                        GPU_part2_extend_seed_store_res(
                            align_tmp_res, 0, data1s[i], data2s[i], references, info_results[pos++]
                        );
                    }
                    if (!align_tmp_res.done_align[1]) {
                        GPU_part2_extend_seed_store_res(
                            align_tmp_res, 1, data1s[i], data2s[i], references, info_results[pos++]
                        );
                    }
                } else if (align_tmp_res.type == 4) {
                    for (size_t j = 0; j < todo_size; j++) {
                        if (!align_tmp_res.done_align[j]) {
                            if (align_tmp_res.is_extend_seed[j]) {
                                GPU_part2_extend_seed_store_res(
                                    align_tmp_res, j, data1s[i], data2s[i], references, info_results[pos++]
                                );
                            } else {
                                GPU_part2_rescue_mate_store_res(
                                    align_tmp_res, j, data1s[i], data2s[i], references, info_results[pos++], mu, sigma
                                );
                            }
                        }
                    }
                }
            }
            time2_4 += GetTime() - t_1;

            // step3 : use ssw results to construct sam
            t_1 = GetTime();
            std::string sam_out;
            sam_out.reserve(7 * map_param.r * (data1s.size()));
            Sam sam{sam_out, references, map_param.cigar_ops, read_group_id, map_param.output_unmapped, map_param.details};
            for (size_t i = 0; i < data1s.size(); ++i) {
                GPU_align_PE_read_last(global_align_res[i], data1s[i], data2s[i], sam, sam_out, isize_est, aligner,
                                       map_param, index_parameters, references, index, random_engine
                );
            }
            time3_1 += GetTime() - t_1;

            t_1 = GetTime();
            output_buffer.output_records(std::move(sam_out), chunk_index);
            time3_2 += GetTime() - t_1;
        }

        for(int chunk_id = 0; chunk_id < real_chunk_num; chunk_id++) {
            fastqPool.Release(fqdatachunks[chunk_id]->left_part);
            fastqPool.Release(fqdatachunks[chunk_id]->right_part);
        }
    }
    done = true;

    std::cout << "gpu cost " << gpu_copy1 << " " << gpu_copy2 << " " << gpu_cost1 << " " << gpu_cost2 << " [" << gpu_cost2_1 << " " << gpu_cost2_2 << "] " << gpu_cost3 << " " << gpu_cost4 << std::endl;
    std::cout << gpu_cost5 << " " << gpu_cost6 << " " << gpu_cost7 << " " << gpu_cost8 << " " << gpu_cost9 << " " << gpu_cost10 << std::endl;
    std::cout << "[" << gpu_cost10_0 << " " << gpu_cost10_1 << " " << gpu_cost10_2 << " " << gpu_cost10_3 << " " << gpu_cost10_4 << "]" << std::endl;
    std::cout << "copy data to host cost " << gpu_cost11 << " [" << gpu_cost11_copy1 << ", " << gpu_cost11_copy2 << "]" << std::endl;
    std::cout << "total cost " << tot_cost << std::endl;
    std::cout << "check_sum : " << check_sum << ", size_tot : " << size_tot << std::endl;
    std::cout << "total_hits12 : " << global_hits_num12 << ", nr_good_hits12 : " << global_nams_info12 << std::endl;
    std::cout << "total_hits3 : " << global_hits_num3 << ", nr_good_hits3 : " << global_nams_info3 << std::endl;
    std::cout << "total_align_info123 : " << global_align_info123 << std::endl;

    t_1 = GetTime();
    hipHostFree(d_seq);
    hipHostFree(d_len);
    hipHostFree(d_pre_sum);
    hipFree(d_index_para);
    hipFree(d_randstrobes);
    hipFree(d_randstrobe_start_indices);
    delete h_seq;
    delete h_len;
    delete h_pre_sum;
    delete rc_data1;
    delete rc_data2;
    time4 += GetTime() - t_1;


    time_tot = GetTime() - t_0;
    fprintf(
        stderr, "cost time0:%.2f(%.2f %.2f %.2f %.2f) time1:%.2f(%.2f[%.2f] %.2f %.2f) time2:(%.2f[%.2f] %.2f %.2f %.2f) time3:(%.2f %.2f), time4:%.2f tot time:%.2f\n",
        time0, time0_1, time0_2, time0_3, time0_4,
        time1, time1_1, time1_1_1, time1_2, time1_3,
        time2_1, time2_1_1, time2_2, time2_3, time2_4,
        time3_1, time3_2, time4, time_tot
    );

    hipStreamSynchronize(ctx.stream);
}
