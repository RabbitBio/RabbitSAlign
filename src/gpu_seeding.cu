#include "hip/hip_runtime.h"
#include "gpu_seeding.h"

__device__ inline randstrobe_hash_t gpu_get_hash(const RefRandstrobe *d_randstrobes, size_t d_randstrobes_size, my_bucket_index_t position) {
    if (position < d_randstrobes_size) {
        return d_randstrobes[position].hash;
    } else {
        return static_cast<randstrobe_hash_t>(-1);
    }
}

__device__ inline bool gpu_is_filtered(const RefRandstrobe *d_randstrobes, size_t d_randstrobes_size, my_bucket_index_t position,
                                       unsigned int filter_cutoff) {
    return gpu_get_hash(d_randstrobes, d_randstrobes_size, position) ==
           gpu_get_hash(d_randstrobes, d_randstrobes_size, position + filter_cutoff);
}

__device__ int gpu_get_count(
        const RefRandstrobe *d_randstrobes,
        const my_bucket_index_t *d_randstrobe_start_indices,
        my_bucket_index_t position,
        int bits
) {
    const auto key = d_randstrobes[position].hash;
    const unsigned int top_N = key >> (64 - bits);
    int64_t position_end = d_randstrobe_start_indices[top_N + 1];
    int64_t position_start = position;

    if(position_end == 0) return 0;
    int64_t low = position_start, high = position_end - 1, ans = 0;
    while (low <= high) {
        int64_t mid = (low + high) / 2;
        if (d_randstrobes[mid].hash == key) {
            low = mid + 1;
            ans = mid;
        } else {
            high = mid - 1;
        }
    }
    return ans - position_start + 1;
}

__device__ inline size_t gpu_find(
        const RefRandstrobe *d_randstrobes,
        const my_bucket_index_t *d_randstrobe_start_indices,
        const randstrobe_hash_t key,
        int bits
) {
    const unsigned int top_N = key >> (64 - bits);
    my_bucket_index_t position_start = d_randstrobe_start_indices[top_N];
    my_bucket_index_t position_end = d_randstrobe_start_indices[top_N + 1];
    if(position_end - position_start < 64) {
        for (my_bucket_index_t i = position_start; i < position_end; ++i) {
            if (d_randstrobes[i].hash == key) {
                return i;
            }
        }
        return static_cast<size_t>(-1);
    } else {
        my_bucket_index_t low = position_start, high = position_end;
        while (low < high) {
            my_bucket_index_t mid = low + (high - low) / 2;
            if (d_randstrobes[mid].hash < key) {
                low = mid + 1;
            } else {
                high = mid;
            }
        }
        if (d_randstrobes[low].hash == key) {
            return low;
        } else return static_cast<size_t>(-1);
    }
}

__device__ void add_to_hits_per_ref(
        my_vector<my_pair<int, Hit>>& hits_per_ref,
        int query_start,
        int query_end,
        size_t position,
        const RefRandstrobe *d_randstrobes,
        size_t d_randstrobes_size,
        int k
) {
    int min_diff = 1 << 30;
    for (const auto hash = gpu_get_hash(d_randstrobes, d_randstrobes_size, position); gpu_get_hash(d_randstrobes, d_randstrobes_size, position) == hash; ++position) {
        int ref_start = d_randstrobes[position].position;
        int ref_end = ref_start + d_randstrobes[position].strobe2_offset() + k;
        int diff = std::abs((query_end - query_start) - (ref_end - ref_start));
        if (diff <= min_diff) {
            hits_per_ref.push_back({d_randstrobes[position].reference_index(), Hit{query_start, query_end, ref_start, ref_end}});
            min_diff = diff;
        }
    }
}

__global__ void gpu_get_randstrobes(
        int num_tasks,
        int read_num,
        int base_read_num,
        int *pre_sum,
        int *lens,
        char *all_seqs,
        IndexParameters *index_para,
        int *randstrobe_sizes,
        uint64_t *hashes,
        my_vector<QueryRandstrobe>* global_randstrobes
) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < num_tasks) {
        int read_id = (id + base_read_num) % read_num;
        int is_read2 = (id + base_read_num) / read_num;
        size_t len;
        char *seq, *rc;
        if (is_read2 == 0) {
            len = lens[read_id];
            seq = all_seqs + pre_sum[read_id];
            rc = all_seqs + pre_sum[read_id + read_num];
        } else {
            len = lens[read_id + read_num * 2];
            seq = all_seqs + pre_sum[read_id + read_num * 2];
            rc = all_seqs + pre_sum[read_id + read_num * 3];
        }

        my_vector<Syncmer> syncmers(len);

        const int k = index_para->syncmer.k;
        const int s = index_para->syncmer.s;
        const int t = index_para->syncmer.t_syncmer;

        const uint64_t kmask = (1ULL << 2 * k) - 1;
        const uint64_t smask = (1ULL << 2 * s) - 1;
        const uint64_t kshift = (k - 1) * 2;
        const uint64_t sshift = (s - 1) * 2;
        my_vector<uint64_t> gpu_qs(len * 2);
        int l_pos = 0;
        int r_pos = 0;
        uint64_t qs_min_val = UINT64_MAX;
        int qs_min_pos = -1;
        int l = 0;
        uint64_t xk[2] = {0, 0};
        uint64_t xs[2] = {0, 0};
        for (size_t i = 0; i < len; i++) {
            int c = gpu_seq_nt4_table[(uint8_t) seq[i]];
            if (c < 4) { // not an "N" base
                xk[0] = (xk[0] << 2 | c) & kmask;                  // forward strand
                xk[1] = xk[1] >> 2 | (uint64_t)(3 - c) << kshift;  // reverse strand
                xs[0] = (xs[0] << 2 | c) & smask;                  // forward strand
                xs[1] = xs[1] >> 2 | (uint64_t)(3 - c) << sshift;  // reverse strand
                if (++l < s) {
                    continue;
                }
                // we find an s-mer
                uint64_t ys = xs[0] < xs[1] ? xs[0] : xs[1];
                uint64_t hash_s = gpu_syncmer_smer_hash(ys);
                gpu_qs[r_pos++] = hash_s;
                //assert(r_pos < len * 2);
                // not enough hashes in the queue, yet
                if (r_pos - l_pos < k - s + 1) {
                    continue;
                }
                if (r_pos - l_pos == k - s + 1) { // We are at the last s-mer within the first k-mer, need to decide if we add it
                    for (int j = l_pos; j < r_pos; j++) {
                        if (gpu_qs[j] < qs_min_val) {
                            qs_min_val = gpu_qs[j];
                            qs_min_pos = i - k + j - l_pos + 1;
                        }
                    }
                } else {
                    // update queue and current minimum and position
                    l_pos++;
                    if (qs_min_pos == i - k) { // we popped the previous minimizer, find new brute force
                        qs_min_val = UINT64_MAX;
                        qs_min_pos = i - s + 1;
                        for (int j = r_pos - 1; j >= l_pos; j--) { //Iterate in reverse to choose the rightmost minimizer in a window
                            if (gpu_qs[j] < qs_min_val) {
                                qs_min_val = gpu_qs[j];
                                qs_min_pos = i - k + j - l_pos + 1;
                            }
                        }
                    } else if (hash_s < qs_min_val) { // the new value added to queue is the new minimum
                        qs_min_val = hash_s;
                        qs_min_pos = i - s + 1;
                    }
                }
                if (qs_min_pos == i - k + t) { // occurs at t:th position in k-mer
                    uint64_t yk = xk[0] < xk[1] ? xk[0] : xk[1];
                    syncmers.push_back(Syncmer{gpu_syncmer_kmer_hash(yk), i - k + 1});
                }
            } else {
                // if there is an "N", restart
                qs_min_val = UINT64_MAX;
                qs_min_pos = -1;
                l = xs[0] = xs[1] = xk[0] = xk[1] = 0;
                r_pos = 0;
                l_pos = 0;
            }
        }


        const int w_min = index_para->randstrobe.w_min;
        const int w_max = index_para->randstrobe.w_max;
        const uint64_t q = index_para->randstrobe.q;
        const int max_dist = index_para->randstrobe.max_dist;

        global_randstrobes[id].init((my_max(syncmers.size() - w_min, 0)) * 2);

        for (int strobe1_index = 0; strobe1_index + w_min < syncmers.size(); strobe1_index++) {
            unsigned int w_end = (strobe1_index + w_max < syncmers.size() - 1) ? (strobe1_index + w_max) : syncmers.size() - 1;
            auto strobe1 = syncmers[strobe1_index];
            auto max_position = strobe1.position + max_dist;
            unsigned int w_start = strobe1_index + w_min;
            uint64_t min_val = 0xFFFFFFFFFFFFFFFF;
            Syncmer strobe2 = strobe1;
            for (auto i = w_start; i <= w_end && syncmers[i].position <= max_position; i++) {
                uint64_t hash_diff = (strobe1.hash ^ syncmers[i].hash) & q;
                uint64_t res = __popcll(hash_diff);
                if (res < min_val) {
                    min_val = res;
                    strobe2 = syncmers[i];
                }
            }
            global_randstrobes[id].push_back(
                    QueryRandstrobe{
                            gpu_randstrobe_hash(strobe1.hash, strobe2.hash), static_cast<uint32_t>(strobe1.position),
                            static_cast<uint32_t>(strobe2.position) + index_para->syncmer.k, false
                    }
            );
        }


        for (int i = 0; i < syncmers.size() / 2; i++) {
            my_swap(syncmers[i], syncmers[syncmers.size() - i - 1]);
        }
        for (size_t i = 0; i < syncmers.size(); i++) {
            syncmers[i].position = len - syncmers[i].position - (*index_para).syncmer.k;
        }

        for (int strobe1_index = 0; strobe1_index + w_min < syncmers.size(); strobe1_index++) {
            unsigned int w_end = (strobe1_index + w_max < syncmers.size() - 1) ? (strobe1_index + w_max) : syncmers.size() - 1;
            auto strobe1 = syncmers[strobe1_index];
            auto max_position = strobe1.position + max_dist;
            unsigned int w_start = strobe1_index + w_min;
            uint64_t min_val = 0xFFFFFFFFFFFFFFFF;
            Syncmer strobe2 = strobe1;
            for (auto i = w_start; i <= w_end && syncmers[i].position <= max_position; i++) {
                uint64_t hash_diff = (strobe1.hash ^ syncmers[i].hash) & q;
                uint64_t res = __popcll(hash_diff);
                if (res < min_val) {
                    min_val = res;
                    strobe2 = syncmers[i];
                }
            }
            global_randstrobes[id].push_back(
                    QueryRandstrobe{
                            gpu_randstrobe_hash(strobe1.hash, strobe2.hash), static_cast<uint32_t>(strobe1.position),
                            static_cast<uint32_t>(strobe2.position) + index_para->syncmer.k, true
                    }
            );
        }
    }
}


__global__ void gpu_get_hits_pre(
        int bits,
        unsigned int filter_cutoff,
        int rescue_cutoff,
        const RefRandstrobe *d_randstrobes,
        size_t d_randstrobes_size,
        const my_bucket_index_t *d_randstrobe_start_indices,
        int num_tasks,
        IndexParameters *index_para,
        uint64_t *global_hits_num,
        my_vector<QueryRandstrobe>* global_randstrobes,
        my_vector<my_pair<int, Hit>>* hits_per_ref0s,
        my_vector<my_pair<int, Hit>>* hits_per_ref1s
) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < num_tasks) {
        for (int i = 0; i < global_randstrobes[id].size(); i++) {
            QueryRandstrobe q = global_randstrobes[id][i];
            size_t position = gpu_find(d_randstrobes, d_randstrobe_start_indices, q.hash, bits);
            global_randstrobes[id][i].hash = position;
        }
    }
}

__global__ void gpu_get_hits_after(
        int bits,
        unsigned int filter_cutoff,
        int rescue_cutoff,
        const RefRandstrobe *d_randstrobes,
        size_t d_randstrobes_size,
        const my_bucket_index_t *d_randstrobe_start_indices,
        int num_tasks,
        IndexParameters *index_para,
        uint64_t *global_hits_num,
        my_vector<QueryRandstrobe>* global_randstrobes,
        my_vector<my_pair<int, Hit>>* hits_per_ref0s,
        my_vector<my_pair<int, Hit>>* hits_per_ref1s
) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < num_tasks) {
        uint64_t local_total_hits = 0;
        uint64_t local_nr_good_hits = 0;
        for (int i = 0; i < global_randstrobes[id].size(); i++) {
            QueryRandstrobe q = global_randstrobes[id][i];
            size_t position = q.hash;
            if (position != static_cast<size_t>(-1)) {
                local_total_hits++;
                if (!gpu_is_filtered(d_randstrobes, d_randstrobes_size, position, filter_cutoff)) {
                    local_nr_good_hits++;
                }
            }
        }
        float nonrepetitive_fraction = local_total_hits > 0 ? ((float) local_nr_good_hits) / ((float) local_total_hits) : 1.0;

        if (nonrepetitive_fraction < 0.7) return;

        hits_per_ref0s[id].init(8);
        hits_per_ref1s[id].init(8);
        for (int i = 0; i < global_randstrobes[id].size(); i++) {
            QueryRandstrobe q = global_randstrobes[id][i];
            size_t position = q.hash;
            if (position != static_cast<size_t>(-1)) {
                if (gpu_is_filtered(d_randstrobes, d_randstrobes_size, position, filter_cutoff)) continue;
                if(q.is_reverse) {
                    add_to_hits_per_ref(hits_per_ref1s[id], q.start, q.end, position, d_randstrobes, d_randstrobes_size, index_para->syncmer.k);
                } else {
                    add_to_hits_per_ref(hits_per_ref0s[id], q.start, q.end, position, d_randstrobes, d_randstrobes_size, index_para->syncmer.k);
                }
            }
        }
        global_randstrobes[id].release();
    }
}


__global__ void gpu_rescue_get_hits(
        int bits,
        unsigned int filter_cutoff,
        int rescue_cutoff,
        const RefRandstrobe *d_randstrobes,
        size_t d_randstrobes_size,
        const my_bucket_index_t *d_randstrobe_start_indices,
        int num_tasks,
        IndexParameters *index_para,
        uint64_t *global_hits_num,
        my_vector<QueryRandstrobe>* global_randstrobes,
        my_vector<my_pair<int, Hit>>* hits_per_ref0s,
        my_vector<my_pair<int, Hit>>* hits_per_ref1s,
        int* global_todo_ids,
        int rescue_threshold
)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < num_tasks) {
        int real_id = global_todo_ids[id];
        my_vector<RescueHit> hits_t0;
        my_vector<RescueHit> hits_t1;
        for (int i = 0; i < global_randstrobes[real_id].size(); i++) {
            QueryRandstrobe q = global_randstrobes[real_id][i];
            size_t position = q.hash;
            if (position != static_cast<size_t>(-1)) {
                unsigned int count = gpu_get_count(d_randstrobes, d_randstrobe_start_indices, position, bits);
                RescueHit rh{position, count, q.start, q.end};
                if(q.is_reverse) hits_t1.push_back(rh);
                else hits_t0.push_back(rh);
            }
        }
        global_randstrobes[real_id].release();

        quick_sort(&(hits_t0[0]), hits_t0.size());
        quick_sort(&(hits_t1[0]), hits_t1.size());

        int cnt0 = 0, cnt1 = 0;
        for (int i = 0; i < hits_t0.size(); i++) {
            RescueHit &rh = hits_t0[i];
            if ((rh.count > rescue_cutoff && cnt0 >= 5) || rh.count > rescue_threshold) break;
            cnt0++;
        }
        for (int i = 0; i < hits_t1.size(); i++) {
            RescueHit &rh = hits_t1[i];
            if ((rh.count > rescue_cutoff && cnt1 >= 5) || rh.count > rescue_threshold) break;
            cnt1++;
        }

        quick_sort_iterative(&(hits_t0[0]), 0, cnt0 - 1, [](const RescueHit &r1, const RescueHit &r2) {
            return r1.query_start < r2.query_start;
        });
        quick_sort_iterative(&(hits_t1[0]), 0, cnt1 - 1, [](const RescueHit &r1, const RescueHit &r2) {
            return r1.query_start < r2.query_start;
        });

        for (int i = 0; i < cnt0; i++) {
            RescueHit &rh = hits_t0[i];
            add_to_hits_per_ref(hits_per_ref0s[real_id], rh.query_start, rh.query_end, rh.position, d_randstrobes, d_randstrobes_size, index_para->syncmer.k);
        }
        for (int i = 0; i < cnt1; i++) {
            RescueHit &rh = hits_t1[i];
            add_to_hits_per_ref(hits_per_ref1s[real_id], rh.query_start, rh.query_end, rh.position, d_randstrobes, d_randstrobes_size, index_para->syncmer.k);
        }
    }
}