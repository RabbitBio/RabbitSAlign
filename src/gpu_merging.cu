#include "hip/hip_runtime.h"
#include "gpu_merging.h"
#include <hipcub/hipcub.hpp>

__device__ size_t my_lower_bound(my_pair<int, Hit>* hits, size_t i_start, size_t i_end, int target) {
    size_t left = i_start, right = i_end;
    while (left < right) {
        size_t mid = left + (right - left) / 2;
        if (hits[mid].second.ref_start < target) {
            left = mid + 1;
        } else {
            right = mid;
        }
    }
    return left;
}

__device__ void check_hits(my_vector<my_pair<int, Hit>> &hits_per_ref) {
    if (hits_per_ref.size() < 2) return;
    for(int i = 0; i < hits_per_ref.size() - 1; i++) {
        //if(hits_per_ref[i].first > hits_per_ref[i + 1].first) {
        //    printf("sort error [%d,%d] [%d,%d]\n", hits_per_ref[i].first, hits_per_ref[i].second.query_start, hits_per_ref[i + 1].first, hits_per_ref[i + 1].second.query_start);
        //    assert(false);
        //}
        if(hits_per_ref[i].first == hits_per_ref[i + 1].first && hits_per_ref[i].second.query_start > hits_per_ref[i + 1].second.query_start) {
            printf("sort error [%d,%d] [%d,%d]\n", hits_per_ref[i].first, hits_per_ref[i].second.query_start, hits_per_ref[i + 1].first, hits_per_ref[i + 1].second.query_start);
            assert(false);
        }
        if(hits_per_ref[i].first == hits_per_ref[i + 1].first && hits_per_ref[i].second.query_start == hits_per_ref[i + 1].second.query_start &&
           hits_per_ref[i].second.ref_start > hits_per_ref[i + 1].second.ref_start) {
            printf("sort error [%d,%d,%d] [%d,%d,%d]\n", hits_per_ref[i].first, hits_per_ref[i].second.query_start, hits_per_ref[i].second.ref_start,
                   hits_per_ref[i + 1].first, hits_per_ref[i + 1].second.query_start, hits_per_ref[i + 1].second.ref_start);
            assert(false);
        }
    }
}

__device__ void sort_hits_single(
        my_vector<my_pair<int, Hit>>& hits_per_ref
) {
    quick_sort(&(hits_per_ref[0]), hits_per_ref.size());
}

#define key_mod_val 29

__device__ int find_ref_ids(int ref_id, int* head, ref_ids_edge* edges) {
    int key = ref_id % key_mod_val;
    for (int i = head[key]; i != -1; i = edges[i].pre) {
        if (edges[i].ref_id == ref_id) return i;
    }
    return -1;
}

__device__ size_t indirect_lower_bound(
        const my_vector<my_pair<int, Hit>>& original_hits,
        const int* sorted_indices,
        size_t global_range_start_idx,
        size_t count,
        int value
) {
    size_t start = 0;
    while (count > 0) {
        size_t step = count / 2;
        size_t it = start + step;
        int original_idx = sorted_indices[global_range_start_idx + it];
        if (original_hits.data[original_idx].second.ref_start < value) {
            start = it + 1;
            count -= step + 1;
        } else {
            count = step;
        }
    }
    return start; // Returns a relative index [0, count)
}


__device__ void salign_merge_hits_seg(
        const my_vector<my_pair<int, Hit>>& original_hits,
        const int* sorted_indices,
        int task_start_offset,
        int task_end_offset,
        int k,
        bool is_revcomp,
        my_vector<Nam>& nams
) {
    int num_hits_in_task = task_end_offset - task_start_offset;
    assert(num_hits_in_task == original_hits.size());
    if (num_hits_in_task == 0) return;

    int ref_num = 0;
    my_vector<int> each_ref_size(8);

    int first_hit_original_idx = sorted_indices[task_start_offset];
    int pre_ref_id = original_hits.data[first_hit_original_idx].first;
    int now_ref_num = 1;

    for (int i = 1; i < num_hits_in_task; i++) {
        int global_idx = task_start_offset + i;
        int original_idx = sorted_indices[global_idx];
        int ref_id = original_hits.data[original_idx].first;

        if (ref_id != pre_ref_id) {
            ref_num++;
            pre_ref_id = ref_id;
            each_ref_size.push_back(now_ref_num);
            now_ref_num = 1;
        } else {
            now_ref_num++;
        }
    }
    ref_num++;
    each_ref_size.push_back(now_ref_num);

    // --- Step 2: Apply the original `salign_merge_hits` algorithm logic. ---
    my_vector<Nam> open_nams;
    my_vector<bool> is_added(32);
    int now_vec_pos = 0; // Local offset within this task's hits [0, num_hits_in_task)

    for (int rid = 0; rid < ref_num; rid++) {
        if(rid != 0) now_vec_pos += each_ref_size[rid - 1];

        int first_hit_global_idx = task_start_offset + now_vec_pos;
        int first_hit_original_idx = sorted_indices[first_hit_global_idx];
        int ref_id = original_hits.data[first_hit_original_idx].first;

        open_nams.clear();
        unsigned int prev_q_start = 0;
        size_t hits_size = each_ref_size[rid];

        for (size_t i = 0; i < hits_size; ) {
            int i_start_original_idx = sorted_indices[task_start_offset + now_vec_pos + i];
            int current_query_start = original_hits.data[i_start_original_idx].second.query_start;

            size_t i_start = i;
            size_t i_end = i + 1;
            while(i_end < hits_size) {
                int next_hit_original_idx = sorted_indices[task_start_offset + now_vec_pos + i_end];
                if (original_hits.data[next_hit_original_idx].second.query_start == current_query_start) {
                    i_end++;
                } else {
                    break;
                }
            }
            i = i_end;
            size_t i_size = i_end - i_start;

            is_added.clear();
            for(size_t j = 0; j < i_size; j++) is_added.push_back(false);

            int query_start = current_query_start;
            int cnt_done = 0;
            for (int k = 0; k < open_nams.size(); k++) {
                Nam& o = open_nams[k];
                if ( query_start > o.query_end ) continue;

                size_t global_range_start_idx = task_start_offset + now_vec_pos + i_start;
                size_t range_count = i_end - i_start;

                size_t lower_rel_idx = indirect_lower_bound(original_hits, sorted_indices, global_range_start_idx, range_count, o.ref_prev_hit_startpos + 1);
                size_t upper_rel_idx = indirect_lower_bound(original_hits, sorted_indices, global_range_start_idx, range_count, o.ref_end + 1);

                for (size_t j_rel = lower_rel_idx; j_rel < upper_rel_idx; j_rel++) {
                    size_t j = i_start + j_rel; // Convert relative index to local index
                    if(is_added[j - i_start]) continue;

                    int hit_original_idx = sorted_indices[task_start_offset + now_vec_pos + j];
                    const Hit& h = original_hits.data[hit_original_idx].second;

                    if (o.ref_prev_hit_startpos < h.ref_start && h.ref_start <= o.ref_end) {
                        if ((h.query_end > o.query_end) && (h.ref_end > o.ref_end)) {
                            o.query_end = h.query_end;
                            o.ref_end = h.ref_end;
                            o.query_prev_hit_startpos = h.query_start;
                            o.ref_prev_hit_startpos = h.ref_start;
                            o.n_hits++;
                            is_added[j - i_start] = true;
                            cnt_done++;
                            break;
                        } else if ((h.query_end <= o.query_end) && (h.ref_end <= o.ref_end)) {
                            o.query_prev_hit_startpos = h.query_start;
                            o.ref_prev_hit_startpos = h.ref_start;
                            o.n_hits++;
                            is_added[j - i_start] = true;
                            cnt_done++;
                            break;
                        }
                    }
                }
                if(cnt_done == i_size) break;
            }

            // Add new NAMs for hits not added to existing ones.
            for(size_t j = 0; j < i_size; j++) {
                if (!is_added[j]){
                    int hit_original_idx = sorted_indices[task_start_offset + now_vec_pos + i_start + j];
                    const Hit& h = original_hits.data[hit_original_idx].second;
                    Nam n;
                    n.query_start = h.query_start;
                    n.query_end = h.query_end;
                    n.ref_start = h.ref_start;
                    n.ref_end = h.ref_end;
                    n.ref_id = ref_id;
                    n.query_prev_hit_startpos = h.query_start;
                    n.ref_prev_hit_startpos = h.ref_start;
                    n.n_hits = 1;
                    n.is_rc = is_revcomp;
                    open_nams.push_back(n);
                }
            }


            // Only filter if we have advanced at least k nucleotides
            if (query_start > prev_q_start + k) {

                // Output all NAMs from open_matches to final_nams that the current hit have passed
                for (int k = 0; k < open_nams.size(); k++) {
                    Nam& n = open_nams[k];
                    if (n.query_end < query_start) {
                        int n_max_span = my_max(n.query_span(), n.ref_span());
                        int n_min_span = my_min(n.query_span(), n.ref_span());
                        float n_score;
                        n_score = ( 2*n_min_span -  n_max_span) > 0 ? (float) (n.n_hits * ( 2*n_min_span -  n_max_span) ) : 1;   // this is really just n_hits * ( min_span - (offset_in_span) ) );
                        //                        n_score = n.n_hits * n.query_span();
                        n.score = n_score;
                        n.nam_id = nams.size();
                        nams.push_back(n);
                    }
                }

                // Remove all NAMs from open_matches that the current hit have passed
                auto c = query_start;
                int old_open_size = open_nams.size();
                open_nams.clear();
                for (int in = 0; in < old_open_size; ++in) {
                    if (!(open_nams[in].query_end < c)) {
                        open_nams.push_back(open_nams[in]);
                    }
                }
                prev_q_start = query_start;
            }
        }
        // Add all current open_matches to final NAMs
        for (int k = 0; k < open_nams.size(); k++) {
            Nam& n = open_nams[k];
            int n_max_span = my_max(n.query_span(), n.ref_span());
            int n_min_span = my_min(n.query_span(), n.ref_span());
            float n_score;
            n_score = ( 2*n_min_span -  n_max_span) > 0 ? (float) (n.n_hits * ( 2*n_min_span -  n_max_span) ) : 1;   // this is really just n_hits * ( min_span - (offset_in_span) ) );
            //            n_score = n.n_hits * n.query_span();
            n.score = n_score;
            n.nam_id = nams.size();
            nams.push_back(n);
        }
    }
}


__device__ void salign_merge_hits(
        my_vector<my_pair<int, Hit>>& hits_per_ref,
        int k,
        bool is_revcomp,
        my_vector<Nam>& nams
) {
    if(hits_per_ref.size() == 0) return;
    int ref_num = 0;
    my_vector<int> each_ref_size;
    int pre_ref_id = hits_per_ref[0].first;
    int now_ref_num = 1;
    for(int i = 1; i < hits_per_ref.size(); i++) {
        int ref_id = hits_per_ref[i].first;
        Hit hit = hits_per_ref[i].second;
        if(ref_id != pre_ref_id) {
            ref_num++;
            pre_ref_id = ref_id;
            each_ref_size.push_back(now_ref_num);
            now_ref_num = 1;
        } else {
            now_ref_num++;
        }
    }
    ref_num++;
    each_ref_size.push_back(now_ref_num);

    my_vector<Nam> open_nams;
    my_vector<bool> is_added(32);
    int now_vec_pos = 0;
    for (int rid = 0; rid < ref_num; rid++) {
        if(rid != 0) now_vec_pos += each_ref_size[rid - 1];
        int ref_id = hits_per_ref[now_vec_pos].first;
        open_nams.clear();
        unsigned int prev_q_start = 0;
        size_t hits_size = each_ref_size[rid];
        my_pair<int, Hit>* hits = &(hits_per_ref[now_vec_pos]);
        for (size_t i = 0; i < hits_size; ) {
            size_t i_start = i;
            size_t i_end = i + 1;
            size_t i_size;
            while(i_end < hits_size && hits[i_end].second.query_start == hits[i].second.query_start) i_end++;
            i = i_end;
            i_size = i_end - i_start;
            //for(int j = 0; j < i_size - 1; j++) {
            //    assert(hits[i_start + j].second.ref_start <= hits[i_start + j + 1].second.ref_start);
            //}
            //quick_sort(&(hits[i_start]), i_size);
            is_added.clear();
            for(size_t j = 0; j < i_size; j++) is_added.push_back(false);
            int query_start = hits[i_start].second.query_start;
            int cnt_done = 0;
            for (int k = 0; k < open_nams.size(); k++) {
                Nam& o = open_nams[k];
                if ( query_start > o.query_end ) continue;
                size_t lower = my_lower_bound(hits, i_start, i_end, o.ref_prev_hit_startpos + 1);
                size_t upper = my_lower_bound(hits, i_start, i_end, o.ref_end + 1);
                for (size_t j = lower; j < upper; j++) {
                    if(is_added[j - i_start]) continue;
                    Hit& h = hits[j].second;
                    {
                        if (o.ref_prev_hit_startpos < h.ref_start && h.ref_start <= o.ref_end) {
                            if ((h.query_end > o.query_end) && (h.ref_end > o.ref_end)) {
                                o.query_end = h.query_end;
                                o.ref_end = h.ref_end;
                                //                        o.previous_query_start = h.query_s;
                                //                        o.previous_ref_start = h.ref_s; // keeping track so that we don't . Can be caused by interleaved repeats.
                                o.query_prev_hit_startpos = h.query_start;
                                o.ref_prev_hit_startpos = h.ref_start;
                                o.n_hits++;
                                //                        o.score += (float)1/ (float)h.count;
                                is_added[j - i_start] = true;
                                cnt_done++;
                                break;
                            } else if ((h.query_end <= o.query_end) && (h.ref_end <= o.ref_end)) {
                                //                        o.previous_query_start = h.query_s;
                                //                        o.previous_ref_start = h.ref_s; // keeping track so that we don't . Can be caused by interleaved repeats.
                                o.query_prev_hit_startpos = h.query_start;
                                o.ref_prev_hit_startpos = h.ref_start;
                                o.n_hits++;
                                //                        o.score += (float)1/ (float)h.count;
                                is_added[j - i_start] = true;
                                cnt_done++;
                                break;
                            }
                        }
                    }
                }
                if(cnt_done == i_size) break;
            }

            // Add the hit to open matches
            for(size_t j = 0; j < i_size; j++) {
                if (!is_added[j]){
                    Nam n;
                    n.query_start = hits[i_start + j].second.query_start;
                    n.query_end = hits[i_start + j].second.query_end;
                    n.ref_start = hits[i_start + j].second.ref_start;
                    n.ref_end = hits[i_start + j].second.ref_end;
                    n.ref_id = ref_id;
                    //                n.previous_query_start = h.query_s;
                    //                n.previous_ref_start = h.ref_s;
                    n.query_prev_hit_startpos = hits[i_start + j].second.query_start;
                    n.ref_prev_hit_startpos = hits[i_start + j].second.ref_start;
                    n.n_hits = 1;
                    n.is_rc = is_revcomp;
                    //                n.score += (float)1 / (float)h.count;
                    open_nams.push_back(n);
                }
            }

            // Only filter if we have advanced at least k nucleotides
            if (query_start > prev_q_start + k) {

                // Output all NAMs from open_matches to final_nams that the current hit have passed
                for (int k = 0; k < open_nams.size(); k++) {
                    Nam& n = open_nams[k];
                    if (n.query_end < query_start) {
                        int n_max_span = my_max(n.query_span(), n.ref_span());
                        int n_min_span = my_min(n.query_span(), n.ref_span());
                        float n_score;
                        n_score = ( 2*n_min_span -  n_max_span) > 0 ? (float) (n.n_hits * ( 2*n_min_span -  n_max_span) ) : 1;   // this is really just n_hits * ( min_span - (offset_in_span) ) );
                        //                        n_score = n.n_hits * n.query_span();
                        n.score = n_score;
                        n.nam_id = nams.size();
                        nams.push_back(n);
                    }
                }

                // Remove all NAMs from open_matches that the current hit have passed
                auto c = query_start;
                int old_open_size = open_nams.size();
                open_nams.clear();
                for (int in = 0; in < old_open_size; ++in) {
                    if (!(open_nams[in].query_end < c)) {
                        open_nams.push_back(open_nams[in]);
                    }
                }
                prev_q_start = query_start;
            }
        }
        // Add all current open_matches to final NAMs
        for (int k = 0; k < open_nams.size(); k++) {
            Nam& n = open_nams[k];
            int n_max_span = my_max(n.query_span(), n.ref_span());
            int n_min_span = my_min(n.query_span(), n.ref_span());
            float n_score;
            n_score = ( 2*n_min_span -  n_max_span) > 0 ? (float) (n.n_hits * ( 2*n_min_span -  n_max_span) ) : 1;   // this is really just n_hits * ( min_span - (offset_in_span) ) );
            //            n_score = n.n_hits * n.query_span();
            n.score = n_score;
            n.nam_id = nams.size();
            nams.push_back(n);
        }
    }
}

__device__ void merge_hits_seg(
        const my_vector<my_pair<int, Hit>>& original_hits,
        const int* sorted_indices,
        int task_start_offset,
        int task_end_offset,
        int k,
        bool is_revcomp,
        my_vector<Nam>& nams
) {
    int num_hits_in_task = task_end_offset - task_start_offset;
    assert(num_hits_in_task == original_hits.size());
    if (num_hits_in_task == 0) return;

    // --- Step 1: Group hits by ref_id (same logic as before, but with indirect access) ---
    int ref_num = 0;
    my_vector<int> each_ref_size(8);

    // Get the first hit to initialize the grouping
    int first_hit_original_idx = sorted_indices[task_start_offset];
    int pre_ref_id = original_hits.data[first_hit_original_idx].first;
    int now_ref_num = 1;

    for (int i = 1; i < num_hits_in_task; i++) {
        int global_idx = task_start_offset + i;
        int original_idx = sorted_indices[global_idx];
        int ref_id = original_hits.data[original_idx].first;

        if (ref_id != pre_ref_id) {
            ref_num++;
            pre_ref_id = ref_id;
            each_ref_size.push_back(now_ref_num);
            now_ref_num = 1;
        } else {
            now_ref_num++;
        }
    }
    ref_num++;
    each_ref_size.push_back(now_ref_num);

    // --- Step 2: Iterate through groups and merge hits into NAMs ---
    my_vector<Nam> open_nams;
    int now_vec_pos = 0; // This is a local offset within this task's hits [0, num_hits_in_task)

    for (int i = 0; i < ref_num; i++) {
        if (i != 0) now_vec_pos += each_ref_size[i - 1];

        // Get ref_id for the current group
        int first_hit_global_idx = task_start_offset + now_vec_pos;
        int first_hit_original_idx = sorted_indices[first_hit_global_idx];
        int ref_id = original_hits.data[first_hit_original_idx].first;

        open_nams.clear();
        unsigned int prev_q_start = 0;

        for (int j = 0; j < each_ref_size[i]; j++) {
            // Indirectly access the Hit object in sorted order
            int current_hit_local_idx_in_task = now_vec_pos + j;
            int current_hit_global_idx = task_start_offset + current_hit_local_idx_in_task;
            int original_idx = sorted_indices[current_hit_global_idx];
            const Hit& h = original_hits.data[original_idx].second;
            bool is_added = false;

            for (int k = 0; k < open_nams.size(); k++) {
                Nam& o = open_nams[k];

                // Extend NAM
                if ((o.query_prev_hit_startpos < h.query_start) && (h.query_start <= o.query_end ) && (o.ref_prev_hit_startpos < h.ref_start) && (h.ref_start <= o.ref_end) ){
                    if ( (h.query_end > o.query_end) && (h.ref_end > o.ref_end) ) {
                        o.query_end = h.query_end;
                        o.ref_end = h.ref_end;
                        //                        o.previous_query_start = h.query_s;
                        //                        o.previous_ref_start = h.ref_s; // keeping track so that we don't . Can be caused by interleaved repeats.
                        o.query_prev_hit_startpos = h.query_start;
                        o.ref_prev_hit_startpos = h.ref_start;
                        o.n_hits ++;
                        //                        o.score += (float)1/ (float)h.count;
                        is_added = true;
                        break;
                    }
                    else if ((h.query_end <= o.query_end) && (h.ref_end <= o.ref_end)) {
                        //                        o.previous_query_start = h.query_s;
                        //                        o.previous_ref_start = h.ref_s; // keeping track so that we don't . Can be caused by interleaved repeats.
                        o.query_prev_hit_startpos = h.query_start;
                        o.ref_prev_hit_startpos = h.ref_start;
                        o.n_hits ++;
                        //                        o.score += (float)1/ (float)h.count;
                        is_added = true;
                        break;
                    }
                }

            }

            // Add the hit to open matches
            if (!is_added){
                Nam n;
                n.query_start = h.query_start;
                n.query_end = h.query_end;
                n.ref_start = h.ref_start;
                n.ref_end = h.ref_end;
                n.ref_id = ref_id;
                //                n.previous_query_start = h.query_s;
                //                n.previous_ref_start = h.ref_s;
                n.query_prev_hit_startpos = h.query_start;
                n.ref_prev_hit_startpos = h.ref_start;
                n.n_hits = 1;
                n.is_rc = is_revcomp;
                //                n.score += (float)1 / (float)h.count;
                open_nams.push_back(n);
            }

            // Only filter if we have advanced at least k nucleotides
            if (h.query_start > prev_q_start + k) {
                // Output all NAMs from open_matches to final_nams that the current hit have passed
                for (int k = 0; k < open_nams.size(); k++) {
                    Nam& n = open_nams[k];
                    if (n.query_end < h.query_start) {
                        int n_max_span = my_max(n.query_span(), n.ref_span());
                        int n_min_span = my_min(n.query_span(), n.ref_span());
                        float n_score;
                        n_score = ( 2*n_min_span -  n_max_span) > 0 ? (float) (n.n_hits * ( 2*n_min_span -  n_max_span) ) : 1;   // this is really just n_hits * ( min_span - (offset_in_span) ) );
                        //                        n_score = n.n_hits * n.query_span();
                        n.score = n_score;
                        n.nam_id = nams.size();
                        nams.push_back(n);
                    }
                }

                // Remove all NAMs from open_matches that the current hit have passed
                auto c = h.query_start;
                int old_open_size = open_nams.size();
                open_nams.clear();
                for (int in = 0; in < old_open_size; ++in) {
                    if (!(open_nams[in].query_end < c)) {
                        open_nams.push_back(open_nams[in]);
                    }
                }
                prev_q_start = h.query_start;
            }
        }

        // Add all current open_matches to final NAMs
        for (int k = 0; k < open_nams.size(); k++) {
            Nam& n = open_nams[k];
            int n_max_span = my_max(n.query_span(), n.ref_span());
            int n_min_span = my_min(n.query_span(), n.ref_span());
            float n_score;
            n_score = ( 2*n_min_span -  n_max_span) > 0 ? (float) (n.n_hits * ( 2*n_min_span -  n_max_span) ) : 1;   // this is really just n_hits * ( min_span - (offset_in_span) ) );
            //            n_score = n.n_hits * n.query_span();
            n.score = n_score;
            n.nam_id = nams.size();
            nams.push_back(n);
        }
    }
}

__device__ void merge_hits(
        my_vector<my_pair<int, Hit>>& hits_per_ref,
        int k,
        bool is_revcomp,
        my_vector<Nam>& nams
) {
    if(hits_per_ref.size() == 0) return;
    int num_hits = hits_per_ref.size();

    int ref_num = 0;
    my_vector<int> each_ref_size(8);
    int pre_ref_id = hits_per_ref[0].first;
    int now_ref_num = 1;
    for(int i = 1; i < hits_per_ref.size(); i++) {
        int ref_id = hits_per_ref[i].first;
        Hit hit = hits_per_ref[i].second;
        if(ref_id != pre_ref_id) {
//            assert(ref_id > pre_ref_id);
            ref_num++;
            pre_ref_id = ref_id;
            each_ref_size.push_back(now_ref_num);
            now_ref_num = 1;
        } else {
            now_ref_num++;
        }
    }
    ref_num++;
    each_ref_size.push_back(now_ref_num);

    my_vector<Nam> open_nams;

    int now_vec_pos = 0;
    for (int i = 0; i < ref_num; i++) {

        if(i != 0) now_vec_pos += each_ref_size[i - 1];
        int ref_id = hits_per_ref[now_vec_pos].first;
        open_nams.clear();
        unsigned int prev_q_start = 0;

        for (int j = 0; j < each_ref_size[i]; j++) {
            Hit& h = hits_per_ref[now_vec_pos + j].second;
            bool is_added = false;
            for (int k = 0; k < open_nams.size(); k++) {
                Nam& o = open_nams[k];

                // Extend NAM
                if ((o.query_prev_hit_startpos < h.query_start) && (h.query_start <= o.query_end ) && (o.ref_prev_hit_startpos < h.ref_start) && (h.ref_start <= o.ref_end) ){
                    if ( (h.query_end > o.query_end) && (h.ref_end > o.ref_end) ) {
                        o.query_end = h.query_end;
                        o.ref_end = h.ref_end;
                        //                        o.previous_query_start = h.query_s;
                        //                        o.previous_ref_start = h.ref_s; // keeping track so that we don't . Can be caused by interleaved repeats.
                        o.query_prev_hit_startpos = h.query_start;
                        o.ref_prev_hit_startpos = h.ref_start;
                        o.n_hits ++;
                        //                        o.score += (float)1/ (float)h.count;
                        is_added = true;
                        break;
                    }
                    else if ((h.query_end <= o.query_end) && (h.ref_end <= o.ref_end)) {
                        //                        o.previous_query_start = h.query_s;
                        //                        o.previous_ref_start = h.ref_s; // keeping track so that we don't . Can be caused by interleaved repeats.
                        o.query_prev_hit_startpos = h.query_start;
                        o.ref_prev_hit_startpos = h.ref_start;
                        o.n_hits ++;
                        //                        o.score += (float)1/ (float)h.count;
                        is_added = true;
                        break;
                    }
                }

            }

            // Add the hit to open matches
            if (!is_added){
                Nam n;
                n.query_start = h.query_start;
                n.query_end = h.query_end;
                n.ref_start = h.ref_start;
                n.ref_end = h.ref_end;
                n.ref_id = ref_id;
                //                n.previous_query_start = h.query_s;
                //                n.previous_ref_start = h.ref_s;
                n.query_prev_hit_startpos = h.query_start;
                n.ref_prev_hit_startpos = h.ref_start;
                n.n_hits = 1;
                n.is_rc = is_revcomp;
                //                n.score += (float)1 / (float)h.count;
                open_nams.push_back(n);
            }

            // Only filter if we have advanced at least k nucleotides
            if (h.query_start > prev_q_start + k) {
                // Output all NAMs from open_matches to final_nams that the current hit have passed
                for (int k = 0; k < open_nams.size(); k++) {
                    Nam& n = open_nams[k];
                    if (n.query_end < h.query_start) {
                        int n_max_span = my_max(n.query_span(), n.ref_span());
                        int n_min_span = my_min(n.query_span(), n.ref_span());
                        float n_score;
                        n_score = ( 2*n_min_span -  n_max_span) > 0 ? (float) (n.n_hits * ( 2*n_min_span -  n_max_span) ) : 1;   // this is really just n_hits * ( min_span - (offset_in_span) ) );
                        //                        n_score = n.n_hits * n.query_span();
                        n.score = n_score;
                        n.nam_id = nams.size();
                        nams.push_back(n);
                    }
                }

                // Remove all NAMs from open_matches that the current hit have passed
                auto c = h.query_start;
                int old_open_size = open_nams.size();
                open_nams.clear();
                for (int in = 0; in < old_open_size; ++in) {
                    if (!(open_nams[in].query_end < c)) {
                        open_nams.push_back(open_nams[in]);
                    }
                }
                prev_q_start = h.query_start;
            }
        }

        // Add all current open_matches to final NAMs
        for (int k = 0; k < open_nams.size(); k++) {
            Nam& n = open_nams[k];
            int n_max_span = my_max(n.query_span(), n.ref_span());
            int n_min_span = my_min(n.query_span(), n.ref_span());
            float n_score;
            n_score = ( 2*n_min_span -  n_max_span) > 0 ? (float) (n.n_hits * ( 2*n_min_span -  n_max_span) ) : 1;   // this is really just n_hits * ( min_span - (offset_in_span) ) );
            //            n_score = n.n_hits * n.query_span();
            n.score = n_score;
            n.nam_id = nams.size();
            nams.push_back(n);
        }
    }
}


__device__ void gpu_shuffle_top_nams(my_vector<Nam>& nams) {
#ifdef GPU_ACC_TAG
    return;
#endif
    unsigned int seed = 1234567u;
    if (nams.empty()) {
        return;
    }
    auto best_score = nams[0].score;
    int top_cnt = 1;
    while (top_cnt < nams.size() && nams[top_cnt].score == best_score)
        ++top_cnt;
    auto next_rand = [&seed]() {
        seed = seed * 1664525u + 1013904223u;
        return seed;
    };

    for (int i = top_cnt - 1; i > 0; --i) {
        unsigned int j = next_rand() % (i + 1);
        Nam tmp = nams[i];
        nams[i] = nams[j];
        nams[j] = tmp;
    }
}


__device__ void sort_hits_by_refid(
        my_vector<my_pair<int, Hit>>& hits_per_ref
) {

    int *head = (int*)my_malloc(key_mod_val * sizeof(int));
    my_vector<ref_ids_edge> edges;
    for(int i = 0; i < key_mod_val; i++) head[i] = -1;
    int ref_ids_num = 0;
    for(int i = 0; i < hits_per_ref.size(); i++) {
        int ref_id = hits_per_ref[i].first;
        int find_ref_id_rank = find_ref_ids(ref_id, head, edges.data);
        if (find_ref_id_rank == -1) {
            find_ref_id_rank = ref_ids_num;
            int key = ref_id % key_mod_val;
            edges.push_back({head[key], ref_id});
            head[key] = ref_ids_num++;
        }
    }
    if (ref_ids_num <= 1) {
        my_free(head);
        return;
    }
    my_vector<my_pair<int, my_vector<Hit>*>> all_hits(ref_ids_num);
    all_hits.length = ref_ids_num;
    my_vector<Hit>* all_vecs = (my_vector<Hit>*)my_malloc(ref_ids_num * sizeof(my_vector<Hit>));
    for (int i = 0; i < ref_ids_num; i++) {
        all_hits[i].first = -1;
        all_hits[i].second = &all_vecs[i];
        all_hits[i].second->init();
    }
    for (int i = 0; i < hits_per_ref.size(); i++) {
        int ref_id = hits_per_ref[i].first;
        int find_ref_id_rank = find_ref_ids(ref_id, head, edges.data);
        assert(find_ref_id_rank >= 0 && find_ref_id_rank < ref_ids_num);
        all_hits[find_ref_id_rank].first = ref_id;
        all_hits[find_ref_id_rank].second->push_back(hits_per_ref[i].second);
    }
    hits_per_ref.clear();
    //quick_sort_iterative(&(all_hits[0]), 0, all_hits.size() - 1,
    //                     [](const my_pair<int, my_vector<Hit>*>& a, const my_pair<int, my_vector<Hit>*>& b) {
    //                         return a.first < b.first;
    //                     });
    for(int i = 0; i < all_hits.size(); i++) {
        for(int j = 0; j < all_hits[i].second->size(); j++) {
            hits_per_ref.push_back({all_hits[i].first, (*all_hits[i].second)[j]});
        }
        all_hits[i].second->release();
    }
    my_free(head);
    my_free(all_vecs);
}

__device__ void sort_nams_single_check(
        my_vector<Nam>& nams
) {
    //bubble_sort(&(hits_per_ref[0]), hits_per_ref.size());
    quick_sort_iterative(&(nams[0]), 0, nams.size() - 1, [](const Nam &n1, const Nam &n2) {
        if(n1.score != n2.score) return n1.score > n2.score;
        if(n1.n_hits != n2.n_hits) return n1.n_hits > n2.n_hits;
        if(n1.query_end != n2.query_end) return n1.query_end < n2.query_end;
        if(n1.query_start != n2.query_start) return n1.query_start < n2.query_start;
        if(n1.ref_end != n2.ref_end) return n1.ref_end < n2.ref_end;
        if(n1.ref_start != n2.ref_start) return n1.ref_start < n2.ref_start;
        if(n1.ref_id != n2.ref_id) return n1.ref_id < n2.ref_id;
        return n1.is_rc < n2.is_rc;
    });
}

__device__ void sort_nam_pairs_by_score(my_vector<gpu_NamPair>& joint_nam_scores, int mx_num) {
    int* head = (int*)my_malloc(key_mod_val * sizeof(int));
    my_vector<ref_ids_edge> edges;
    for (int i = 0; i < key_mod_val; i++) head[i] = -1;
    int score_group_num = 0;
    for (int i = 0; i < joint_nam_scores.size(); i++) {
        int score_key = (int)(joint_nam_scores[i].score);
        int score_rank = find_ref_ids(score_key, head, edges.data);
        if (score_rank == -1) {
            score_rank = score_group_num;
            int key = score_key % key_mod_val;
            edges.push_back({head[key], score_key});
            head[key] = score_group_num++;
        }
    }
    if (score_group_num <= 1) {
        my_free(head);
        return;
    }
    my_vector<my_pair<int, my_vector<gpu_NamPair>*>> all_nams(score_group_num);
    all_nams.length = score_group_num;
    my_vector<gpu_NamPair>* all_vecs = (my_vector<gpu_NamPair>*)my_malloc(score_group_num * sizeof(my_vector<gpu_NamPair>));
    for (int i = 0; i < score_group_num; i++) {
        all_nams[i].first = -1;
        all_nams[i].second = &all_vecs[i];
        all_nams[i].second->init();
    }
    for (int i = 0; i < joint_nam_scores.size(); i++) {
        int score_key = (int)(joint_nam_scores[i].score);
        int score_rank = find_ref_ids(score_key, head, edges.data);
        assert(score_rank >= 0 && score_rank < score_group_num);
        all_nams[score_rank].first = score_key;
        all_nams[score_rank].second->push_back(joint_nam_scores[i]);
    }
    joint_nam_scores.clear();
    quick_sort_iterative(&(all_nams[0]), 0, all_nams.size() - 1,
                         [](const my_pair<int, my_vector<gpu_NamPair>*>& a, const my_pair<int, my_vector<gpu_NamPair>*>& b) {
                             return a.first > b.first;
                         });
    for (int i = 0; i < all_nams.size(); i++) {
        for (int j = 0; j < all_nams[i].second->size(); j++) {
            if (joint_nam_scores.size() == mx_num) break;
            joint_nam_scores.push_back((*all_nams[i].second)[j]);
        }
        all_nams[i].second->release();
    }
    my_free(head);
    my_free(all_vecs);
}


__device__ void sort_nams_by_score(my_vector<Nam>& nams, int mx_num) {
    int* head = (int*)my_malloc(key_mod_val * sizeof(int));
    my_vector<ref_ids_edge> edges;
    for (int i = 0; i < key_mod_val; i++) head[i] = -1;
    int score_group_num = 0;
    for (int i = 0; i < nams.size(); i++) {
        int score_key = (int)(nams[i].score);
        int score_rank = find_ref_ids(score_key, head, edges.data);
        if (score_rank == -1) {
            score_rank = score_group_num;
            int key = score_key % key_mod_val;
            edges.push_back({head[key], score_key});
            head[key] = score_group_num++;
        }
    }
    if (score_group_num <= 1) {
        my_free(head);
        return;
    }
    my_vector<my_pair<int, my_vector<Nam>*>> all_nams(score_group_num);
    all_nams.length = score_group_num;
    my_vector<Nam>* all_vecs = (my_vector<Nam>*)my_malloc(score_group_num * sizeof(my_vector<Nam>));
    for (int i = 0; i < score_group_num; i++) {
        all_nams[i].first = -1;
        all_nams[i].second = &all_vecs[i];
        all_nams[i].second->init();
    }
    for (int i = 0; i < nams.size(); i++) {
        int score_key = (int)(nams[i].score);
        int score_rank = find_ref_ids(score_key, head, edges.data);
        assert(score_rank >= 0 && score_rank < score_group_num);
        all_nams[score_rank].first = score_key;
        all_nams[score_rank].second->push_back(nams[i]);
    }
    nams.clear();
    quick_sort_iterative(&(all_nams[0]), 0, all_nams.size() - 1,
                         [](const my_pair<int, my_vector<Nam>*>& a, const my_pair<int, my_vector<Nam>*>& b) {
                             return a.first > b.first;
                         });
    for (int i = 0; i < all_nams.size(); i++) {
        for (int j = 0; j < all_nams[i].second->size(); j++) {
            if (nams.size() == mx_num) break;
            nams.push_back((*all_nams[i].second)[j]);
        }
        all_nams[i].second->release();
    }
    my_free(head);
    my_free(all_vecs);
}

__device__ void sort_nams_by_hits(my_vector<Nam>& nams, int mx_num) {
    int* head = (int*)my_malloc(key_mod_val * sizeof(int));
    my_vector<ref_ids_edge> edges;
    for (int i = 0; i < key_mod_val; i++) head[i] = -1;
    int score_group_num = 0;
    for (int i = 0; i < nams.size(); i++) {
        int score_key = (int)(nams[i].n_hits);
        int score_rank = find_ref_ids(score_key, head, edges.data);
        if (score_rank == -1) {
            score_rank = score_group_num;
            int key = score_key % key_mod_val;
            edges.push_back({head[key], score_key});
            head[key] = score_group_num++;
        }
    }
    if (score_group_num <= 1) {
        my_free(head);
        return;
    }
    my_vector<my_pair<int, my_vector<Nam>*>> all_nams(score_group_num);
    all_nams.length = score_group_num;
    my_vector<Nam>* all_vecs = (my_vector<Nam>*)my_malloc(score_group_num * sizeof(my_vector<Nam>));
    for (int i = 0; i < score_group_num; i++) {
        all_nams[i].first = -1;
        all_nams[i].second = &all_vecs[i];
        all_nams[i].second->init();
    }
    for (int i = 0; i < nams.size(); i++) {
        int score_key = (int)(nams[i].n_hits);
        int score_rank = find_ref_ids(score_key, head, edges.data);
        assert(score_rank >= 0 && score_rank < score_group_num);
        all_nams[score_rank].first = score_key;
        all_nams[score_rank].second->push_back(nams[i]);
    }
    nams.clear();
    quick_sort_iterative(&(all_nams[0]), 0, all_nams.size() - 1,
                         [](const my_pair<int, my_vector<Nam>*>& a, const my_pair<int, my_vector<Nam>*>& b) {
                             return a.first > b.first;
                         });
    for (int i = 0; i < all_nams.size(); i++) {
        for (int j = 0; j < all_nams[i].second->size(); j++) {
            if (nams.size() == mx_num) break;
            nams.push_back((*all_nams[i].second)[j]);
        }
        all_nams[i].second->release();
    }
    my_free(head);
    my_free(all_vecs);
}

__global__ void get_task_sizes_kernel(
        int num_tasks,
        const my_vector<my_pair<int, Hit>>* all_task_vectors,
        const int* global_todo_ids,
        int* out_task_sizes)
{
    int global_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (global_id < num_tasks) {
        int real_id = global_todo_ids[global_id];
        out_task_sizes[global_id] = all_task_vectors[real_id].size();
    }
}

__global__ void marshal_data_for_sort_kernel(
        int num_tasks,
        const my_vector<my_pair<int, Hit>>* all_task_vectors,
        const int* global_todo_ids,
        const int* task_offsets, // Calculated by prefix sum of sizes
        int* out_keys,           // Destination for ref_ids
        int* out_values)         // Destination for original indices
{
    int task_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (task_id < num_tasks) {
        int real_id = global_todo_ids[task_id];
        const my_vector<my_pair<int, Hit>>& current_vector = all_task_vectors[real_id];
        int start_offset = task_offsets[task_id];

        for (int i = 0; i < current_vector.size(); ++i) {
            out_keys[start_offset + i]   = current_vector.data[i].first; // The ref_id
            out_values[start_offset + i] = i;                           // The original index within this task
        }
    }
}

__global__ void reorder_hits_kernel(
        int num_tasks,
        const my_vector<my_pair<int, Hit>>* original_vectors,
        const int* global_todo_ids,
        const int* task_offsets,
        const int* sorted_indices, // This is the d_values array after sorting
        my_pair<int, Hit>* reordered_buffer) // A temporary buffer to write results to
{
    int global_hit_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (global_hit_idx < task_offsets[num_tasks]) {
        int upper_bound_idx = hipcub::UpperBound(task_offsets, num_tasks + 1, global_hit_idx);
        int task_id = upper_bound_idx - 1;

        int real_id = global_todo_ids[task_id];
        const my_vector<my_pair<int, Hit>>& source_vector = original_vectors[real_id];

        // The original index of the hit to copy is given by the sorted_indices array.
        int original_hit_local_index = sorted_indices[global_hit_idx];

        // Copy the original pair to the new sorted location in the buffer.
        reordered_buffer[global_hit_idx] = source_vector.data[original_hit_local_index];
    }
}

__global__ void update_vector_pointers_kernel(
        int num_tasks,
        my_vector<my_pair<int, Hit>>* all_task_vectors,
        const int* global_todo_ids,
        const int* task_offsets,
        my_pair<int, Hit>* reordered_buffer)
{
    int task_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (task_id < num_tasks) {
        int real_id = global_todo_ids[task_id];
        my_vector<my_pair<int, Hit>>& current_vector = all_task_vectors[real_id];
        assert(current_vector.length == task_offsets[task_id + 1] - task_offsets[task_id]);
        current_vector.release();
        current_vector.data = reordered_buffer + task_offsets[task_id];
        current_vector.length = task_offsets[task_id + 1] - task_offsets[task_id];
        current_vector.capacity = current_vector.length;
    }
}

my_pair<int*, int*> sort_all_hits_with_cub_radix(
        int todo_cnt,
        my_vector<my_pair<int, Hit>>* hits_per_refs,
        int* global_todo_ids,
        hipStream_t stream,
        SegSortGpuResources& buffers,
        double *gpu_cost1,
        double *gpu_cost2,
        double *gpu_cost3,
        double *gpu_cost4)
{
    my_pair<int*, int*> res({nullptr, nullptr});
    if (todo_cnt == 0) return res;

    int threads_per_block = 256;
    int blocks_per_grid = (todo_cnt + threads_per_block - 1) / threads_per_block;

    // --- Part 1: Prepare Segment Information with Re-allocation Logic ---
    double t0 = GetTime();

    size_t required_task_sizes_bytes = todo_cnt * sizeof(int);
    if (buffers.task_sizes_bytes < required_task_sizes_bytes) {
        printf("Allocating task sizes buffer: %zu (%zu) bytes\n", required_task_sizes_bytes, buffers.task_sizes_bytes);
        if (buffers.task_sizes_ptr) CUDA_CHECK(hipFreeAsync(buffers.task_sizes_ptr, stream));
        CUDA_CHECK(hipMallocAsync(&buffers.task_sizes_ptr, required_task_sizes_bytes * 2, stream));
        buffers.task_sizes_bytes = required_task_sizes_bytes * 2;
    }
    get_task_sizes_kernel<<<blocks_per_grid, threads_per_block, 0, stream>>>(todo_cnt, hits_per_refs, global_todo_ids, buffers.task_sizes_ptr);

    size_t required_seg_offsets_bytes = (todo_cnt + 1) * sizeof(int);
    if (buffers.seg_offsets_bytes < required_seg_offsets_bytes) {
        printf("Allocating segment offsets buffer: %zu (%zu) bytes\n", required_seg_offsets_bytes, buffers.seg_offsets_bytes);
        if (buffers.seg_offsets_ptr) CUDA_CHECK(hipFreeAsync(buffers.seg_offsets_ptr, stream));
        CUDA_CHECK(hipMallocAsync(&buffers.seg_offsets_ptr, required_seg_offsets_bytes * 2, stream));
        buffers.seg_offsets_bytes = required_seg_offsets_bytes * 2;
    }

    void* d_scan_temp_storage = nullptr;
    size_t scan_temp_storage_bytes = 0;
    hipcub::DeviceScan::ExclusiveSum(d_scan_temp_storage, scan_temp_storage_bytes, buffers.task_sizes_ptr, buffers.seg_offsets_ptr, todo_cnt + 1, stream);

    if (buffers.scan_temp_bytes < scan_temp_storage_bytes) {
        printf("Allocating scan temp storage: %zu (%zu) bytes\n", scan_temp_storage_bytes, buffers.scan_temp_bytes);
        if (buffers.scan_temp_ptr) CUDA_CHECK(hipFreeAsync(buffers.scan_temp_ptr, stream));
        CUDA_CHECK(hipMallocAsync(&buffers.scan_temp_ptr, scan_temp_storage_bytes * 2, stream));
        buffers.scan_temp_bytes = scan_temp_storage_bytes * 2;
    }
    hipcub::DeviceScan::ExclusiveSum(buffers.scan_temp_ptr, scan_temp_storage_bytes, buffers.task_sizes_ptr, buffers.seg_offsets_ptr, todo_cnt + 1, stream);

    int total_hits;
    CUDA_CHECK(hipMemcpyAsync(&total_hits, buffers.seg_offsets_ptr + todo_cnt, sizeof(int), hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(hipStreamSynchronize(stream));
    *gpu_cost1 += GetTime() - t0;

    // --- Part 2: Marshal Data into Pre-allocated Buffers ---
    t0 = GetTime();

    if (buffers.key_value_capacity < total_hits) {
        fprintf(stderr, "FATAL ERROR: Pre-allocated sort buffer is too small. Required: %d, Allocated: %zu\n", total_hits, buffers.key_value_capacity);
        exit(EXIT_FAILURE);
    }

    marshal_data_for_sort_kernel<<<blocks_per_grid, threads_per_block, 0, stream>>>(todo_cnt, hits_per_refs, global_todo_ids, buffers.seg_offsets_ptr, buffers.key_ptr, buffers.value_ptr);
    *gpu_cost2 += GetTime() - t0;

    // --- Part 3: Sort with CUB (Replaced with DeviceSegmentedRadixSort) ---
    t0 = GetTime();


    void* d_sort_temp_storage = nullptr;
    size_t sort_temp_storage_bytes = 0;

    hipcub::DeviceSegmentedRadixSort::SortPairs(
            d_sort_temp_storage, sort_temp_storage_bytes,
            buffers.key_ptr, buffers.key_alt_ptr,         // Keys_in, Keys_out
            buffers.value_ptr, buffers.value_alt_ptr,     // Values_in, Values_out
            total_hits, todo_cnt,
            buffers.seg_offsets_ptr, buffers.seg_offsets_ptr + 1,
            0, sizeof(int) * 8,                           // begin_bit, end_bit for 32-bit int
            stream);

    if (buffers.sort_temp_bytes < sort_temp_storage_bytes) {
        if (buffers.sort_temp_ptr) CUDA_CHECK(hipFreeAsync(buffers.sort_temp_ptr, stream));
        CUDA_CHECK(hipMallocAsync(&buffers.sort_temp_ptr, sort_temp_storage_bytes * 2, stream));
        buffers.sort_temp_bytes = sort_temp_storage_bytes * 2;
    }

    hipcub::DeviceSegmentedRadixSort::SortPairs(
            buffers.sort_temp_ptr, sort_temp_storage_bytes,
            buffers.key_ptr, buffers.key_alt_ptr,         // Keys_in, Keys_out
            buffers.value_ptr, buffers.value_alt_ptr,     // Values_in, Values_out
            total_hits, todo_cnt,
            buffers.seg_offsets_ptr, buffers.seg_offsets_ptr + 1,
            0, sizeof(int) * 8,                           // begin_bit, end_bit
            stream);

    *gpu_cost3 += GetTime() - t0;

    res.first = buffers.seg_offsets_ptr;
    res.second = buffers.value_alt_ptr;
    return res;
}

my_pair<int*, int*> sort_all_hits_with_cub(
        int todo_cnt,
        my_vector<my_pair<int, Hit>>* hits_per_refs,
        int* global_todo_ids,
        hipStream_t stream,
        SegSortGpuResources& buffers,
        double *gpu_cost1,
        double *gpu_cost2,
        double *gpu_cost3,
        double *gpu_cost4)
{
    my_pair<int*, int*> res({nullptr, nullptr});
    if (todo_cnt == 0) return res;

    int threads_per_block = 256;
    int blocks_per_grid = (todo_cnt + threads_per_block - 1) / threads_per_block;

    // --- Part 1: Prepare Segment Information with Re-allocation Logic ---
    double t0 = GetTime();

    size_t required_task_sizes_bytes = todo_cnt * sizeof(int);
    if (buffers.task_sizes_bytes < required_task_sizes_bytes) {
        printf("Allocating task sizes buffer: %zu (%zu) bytes\n", required_task_sizes_bytes, buffers.task_sizes_bytes);
        if (buffers.task_sizes_ptr) CUDA_CHECK(hipFreeAsync(buffers.task_sizes_ptr, stream));
        CUDA_CHECK(hipMallocAsync(&buffers.task_sizes_ptr, required_task_sizes_bytes * 2, stream));
        buffers.task_sizes_bytes = required_task_sizes_bytes * 2;
    }
    get_task_sizes_kernel<<<blocks_per_grid, threads_per_block, 0, stream>>>(todo_cnt, hits_per_refs, global_todo_ids, buffers.task_sizes_ptr);

    size_t required_seg_offsets_bytes = (todo_cnt + 1) * sizeof(int);
    if (buffers.seg_offsets_bytes < required_seg_offsets_bytes) {
        printf("Allocating segment offsets buffer: %zu (%zu) bytes\n", required_seg_offsets_bytes, buffers.seg_offsets_bytes);
        if (buffers.seg_offsets_ptr) CUDA_CHECK(hipFreeAsync(buffers.seg_offsets_ptr, stream));
        CUDA_CHECK(hipMallocAsync(&buffers.seg_offsets_ptr, required_seg_offsets_bytes * 2, stream));
        buffers.seg_offsets_bytes = required_seg_offsets_bytes * 2;
    }

    void* d_scan_temp_storage = nullptr;
    size_t scan_temp_storage_bytes = 0;
    hipcub::DeviceScan::ExclusiveSum(d_scan_temp_storage, scan_temp_storage_bytes, buffers.task_sizes_ptr, buffers.seg_offsets_ptr, todo_cnt + 1, stream);

    if (buffers.scan_temp_bytes < scan_temp_storage_bytes) {
        printf("Allocating scan temp storage: %zu (%zu) bytes\n", scan_temp_storage_bytes, buffers.scan_temp_bytes);
        if (buffers.scan_temp_ptr) CUDA_CHECK(hipFreeAsync(buffers.scan_temp_ptr, stream));
        CUDA_CHECK(hipMallocAsync(&buffers.scan_temp_ptr, scan_temp_storage_bytes * 2, stream));
        buffers.scan_temp_bytes = scan_temp_storage_bytes * 2;
    }
    hipcub::DeviceScan::ExclusiveSum(buffers.scan_temp_ptr, scan_temp_storage_bytes, buffers.task_sizes_ptr, buffers.seg_offsets_ptr, todo_cnt + 1, stream);

    int total_hits;
    CUDA_CHECK(hipMemcpyAsync(&total_hits, buffers.seg_offsets_ptr + todo_cnt, sizeof(int), hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(hipStreamSynchronize(stream));
    *gpu_cost1 += GetTime() - t0;

    // --- Part 2: Marshal Data into Pre-allocated Buffers ---
    t0 = GetTime();

    if (buffers.key_value_capacity < total_hits) {
        fprintf(stderr, "FATAL ERROR: Pre-allocated sort buffer is too small. Required: %d, Allocated: %zu\n", total_hits, buffers.key_value_capacity);
        exit(EXIT_FAILURE);
    }

    marshal_data_for_sort_kernel<<<blocks_per_grid, threads_per_block, 0, stream>>>(todo_cnt, hits_per_refs, global_todo_ids, buffers.seg_offsets_ptr, buffers.key_ptr, buffers.value_ptr);
    *gpu_cost2 += GetTime() - t0;

    // --- Part 3: Sort with CUB ---
    t0 = GetTime();
    hipcub::DoubleBuffer<int> d_keys_buffer(buffers.key_ptr, buffers.key_alt_ptr);
    hipcub::DoubleBuffer<int> d_values_buffer(buffers.value_ptr, buffers.value_alt_ptr);

    void* d_sort_temp_storage = nullptr;
    size_t sort_temp_storage_bytes = 0;
    hipcub::DeviceSegmentedSort::SortPairs(d_sort_temp_storage, sort_temp_storage_bytes, d_keys_buffer, d_values_buffer, total_hits, todo_cnt, buffers.seg_offsets_ptr, buffers.seg_offsets_ptr + 1, stream);

    if (buffers.sort_temp_bytes < sort_temp_storage_bytes) {
        if (buffers.sort_temp_ptr) CUDA_CHECK(hipFreeAsync(buffers.sort_temp_ptr, stream));
        CUDA_CHECK(hipMallocAsync(&buffers.sort_temp_ptr, sort_temp_storage_bytes * 2, stream));
        buffers.sort_temp_bytes = sort_temp_storage_bytes * 2;
    }
    hipcub::DeviceSegmentedSort::SortPairs(buffers.sort_temp_ptr, sort_temp_storage_bytes, d_keys_buffer, d_values_buffer, total_hits, todo_cnt, buffers.seg_offsets_ptr, buffers.seg_offsets_ptr + 1, stream);
    *gpu_cost3 += GetTime() - t0;

    res.first = buffers.seg_offsets_ptr;
    res.second = d_values_buffer.Current();
    return res;
}

__global__ void get_nam_sizes_kernel(int num_tasks, my_vector<Nam>* nams_per_task, int* global_todo_ids, int* task_sizes) {
    int global_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (global_id < num_tasks) {
        int real_id = global_todo_ids[global_id];
        task_sizes[global_id] = nams_per_task[real_id].size();
    }
}

__global__ void marshal_data_for_nam_sort_kernel(
        int num_tasks,
        my_vector<Nam>* nams_per_task,
        int* global_todo_ids,
        const int* seg_offsets,
        int* d_keys,    // Output: scores (as integers)
        int* d_values   // Output: original indices 0, 1, 2...
) {
    int global_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (global_id < num_tasks) {
        int real_id = global_todo_ids[global_id];
        const my_vector<Nam>& nams = nams_per_task[real_id];
        int task_start_offset = seg_offsets[global_id];

        for (int i = 0; i < nams.size(); i++) {
            // Key is the score. Cast to int for sorting. For descending order, negate the score.
            d_keys[task_start_offset + i] = static_cast<int>(-nams.data[i].score);
            // Value is the original index within this specific my_vector<Nam>.
            d_values[task_start_offset + i] = i;
        }
    }
}

__global__ void gather_sorted_nams_kernel(
        int num_tasks,
        const my_vector<Nam>* nams_per_task,
        const int* global_todo_ids,
        const int* seg_offsets,
        const int* sorted_indices,
        Nam* temp_nams_output
) {
    int global_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (global_id >= num_tasks) return;

    int real_id = global_todo_ids[global_id];
    const my_vector<Nam>& nams = nams_per_task[real_id];
    int task_start = seg_offsets[global_id];
    int num_nams_in_task = seg_offsets[global_id + 1] - task_start;

    for (int i = 0; i < num_nams_in_task; ++i) {
        int original_idx = sorted_indices[task_start + i];
        temp_nams_output[task_start + i] = nams.data[original_idx];
    }
}

__global__ void scatter_sorted_nams_kernel(
        int num_tasks,
        const Nam* temp_nams_input,
        const int* global_todo_ids,
        const int* seg_offsets,
        my_vector<Nam>* nams_per_task
) {
    int global_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (global_id >= num_tasks) return;

    int real_id = global_todo_ids[global_id];
    my_vector<Nam>& nams = nams_per_task[real_id];
    int task_start = seg_offsets[global_id];
    int num_nams_in_task = seg_offsets[global_id + 1] - task_start;

    for (int i = 0; i < num_nams_in_task; ++i) {
        nams.data[i] = temp_nams_input[task_start + i];
    }
}

void sort_nams_by_score_in_place_with_cub(
        int todo_cnt,
        my_vector<Nam>* nams_per_task,
        int* global_todo_ids,
        hipStream_t stream,
        SegSortGpuResources& buffers,
        double *gpu_cost1,
        double *gpu_cost2,
        double *gpu_cost3,
        double *gpu_cost4)
{
    if (todo_cnt == 0) return;

    int threads_per_block = 256;
    int blocks_per_grid = (todo_cnt + threads_per_block - 1) / threads_per_block;

    // --- Part 1: Get sizes and offsets for each segment ---
    double t0 = GetTime();

    size_t required_task_sizes_bytes = todo_cnt * sizeof(int);
    if (buffers.task_sizes_bytes < required_task_sizes_bytes) {
        printf("Allocating NAM task sizes buffer: %zu (%zu) bytes\n", required_task_sizes_bytes, buffers.task_sizes_bytes);
        if (buffers.task_sizes_ptr) CUDA_CHECK(hipFreeAsync(buffers.task_sizes_ptr, stream));
        CUDA_CHECK(hipMallocAsync(&buffers.task_sizes_ptr, required_task_sizes_bytes * 2, stream));
        buffers.task_sizes_bytes = required_task_sizes_bytes * 2;
    }
    get_nam_sizes_kernel<<<blocks_per_grid, threads_per_block, 0, stream>>>(todo_cnt, nams_per_task, global_todo_ids, buffers.task_sizes_ptr);

    size_t required_seg_offsets_bytes = (todo_cnt + 1) * sizeof(int);
    if (buffers.seg_offsets_bytes < required_seg_offsets_bytes) {
        printf("Allocating NAM segment offsets buffer: %zu (%zu) bytes\n", required_seg_offsets_bytes, buffers.seg_offsets_bytes);
        if (buffers.seg_offsets_ptr) CUDA_CHECK(hipFreeAsync(buffers.seg_offsets_ptr, stream));
        CUDA_CHECK(hipMallocAsync(&buffers.seg_offsets_ptr, required_seg_offsets_bytes * 2, stream));
        buffers.seg_offsets_bytes = required_seg_offsets_bytes * 2;
    }

    void* d_scan_temp_storage = nullptr;
    size_t scan_temp_storage_bytes = 0;
    hipcub::DeviceScan::ExclusiveSum(d_scan_temp_storage, scan_temp_storage_bytes, buffers.task_sizes_ptr, buffers.seg_offsets_ptr, todo_cnt + 1, stream);

    if (buffers.scan_temp_bytes < scan_temp_storage_bytes) {
        printf("Allocating NAM scan temp storage: %zu (%zu) bytes\n", scan_temp_storage_bytes, buffers.scan_temp_bytes);
        if (buffers.scan_temp_ptr) CUDA_CHECK(hipFreeAsync(buffers.scan_temp_ptr, stream));
        CUDA_CHECK(hipMallocAsync(&buffers.scan_temp_ptr, scan_temp_storage_bytes * 2, stream));
        buffers.scan_temp_bytes = scan_temp_storage_bytes * 2;
    }
    hipcub::DeviceScan::ExclusiveSum(buffers.scan_temp_ptr, scan_temp_storage_bytes, buffers.task_sizes_ptr, buffers.seg_offsets_ptr, todo_cnt + 1, stream);

    int total_nams;
    CUDA_CHECK(hipMemcpyAsync(&total_nams, buffers.seg_offsets_ptr + todo_cnt, sizeof(int), hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(hipStreamSynchronize(stream));
    *gpu_cost1 += GetTime() - t0;

    if (total_nams == 0) {
        return;
    }

    // --- Part 2: Marshal scores (keys) and original indices (values) ---
    t0 = GetTime();

    if (buffers.key_value_capacity < total_nams) {
        printf("Allocating NAM key-value buffers: %d (%zu) bytes\n", total_nams, buffers.key_value_capacity);
        if (buffers.key_ptr)       CUDA_CHECK(hipFreeAsync(buffers.key_ptr, stream));
        if (buffers.value_ptr)     CUDA_CHECK(hipFreeAsync(buffers.value_ptr, stream));
        if (buffers.key_alt_ptr)   CUDA_CHECK(hipFreeAsync(buffers.key_alt_ptr, stream));
        if (buffers.value_alt_ptr) CUDA_CHECK(hipFreeAsync(buffers.value_alt_ptr, stream));
        size_t new_capacity = total_nams * 2;
        size_t new_bytes = new_capacity * sizeof(int);
        CUDA_CHECK(hipMallocAsync(&buffers.key_ptr,       new_bytes, stream));
        CUDA_CHECK(hipMallocAsync(&buffers.value_ptr,     new_bytes, stream));
        CUDA_CHECK(hipMallocAsync(&buffers.key_alt_ptr,   new_bytes, stream));
        CUDA_CHECK(hipMallocAsync(&buffers.value_alt_ptr, new_bytes, stream));
        buffers.key_value_capacity = new_capacity;
    }

    marshal_data_for_nam_sort_kernel<<<blocks_per_grid, threads_per_block, 0, stream>>>(todo_cnt, nams_per_task, global_todo_ids, buffers.seg_offsets_ptr, buffers.key_ptr, buffers.value_ptr);
    CUDA_CHECK(hipStreamSynchronize(stream));
    *gpu_cost2 += GetTime() - t0;

    // --- Part 3: Sort key-value pairs to get the sorted order ---
    t0 = GetTime();
    hipcub::DoubleBuffer<int> d_keys_buffer(buffers.key_ptr, buffers.key_alt_ptr);
    hipcub::DoubleBuffer<int> d_values_buffer(buffers.value_ptr, buffers.value_alt_ptr);

    void* d_sort_temp_storage = nullptr;
    size_t sort_temp_storage_bytes = 0;
    hipcub::DeviceSegmentedSort::SortPairs(d_sort_temp_storage, sort_temp_storage_bytes, d_keys_buffer, d_values_buffer, total_nams, todo_cnt, buffers.seg_offsets_ptr, buffers.seg_offsets_ptr + 1, stream);

    if (buffers.sort_temp_bytes < sort_temp_storage_bytes) {
        printf("Allocating NAM sort temp storage: %zu (%zu) bytes\n", sort_temp_storage_bytes, buffers.sort_temp_bytes);
        if (buffers.sort_temp_ptr) CUDA_CHECK(hipFreeAsync(buffers.sort_temp_ptr, stream));
        CUDA_CHECK(hipMallocAsync(&buffers.sort_temp_ptr, sort_temp_storage_bytes * 2, stream));
        buffers.sort_temp_bytes = sort_temp_storage_bytes * 2;
    }
    hipcub::DeviceSegmentedSort::SortPairs(buffers.sort_temp_ptr, sort_temp_storage_bytes, d_keys_buffer, d_values_buffer, total_nams, todo_cnt, buffers.seg_offsets_ptr, buffers.seg_offsets_ptr + 1, stream);
    const int* sorted_indices = d_values_buffer.Current();
    CUDA_CHECK(hipStreamSynchronize(stream));
    *gpu_cost3 += GetTime() - t0;


    // --- Part 4: Reorder the actual NAM objects in-place ---
    t0 = GetTime();

    if (buffers.nam_temp_capacity < total_nams) {
        printf("Allocating NAM temporary buffer: %d (%zu) bytes\n", total_nams * 2, buffers.nam_temp_capacity);
        if (buffers.nam_temp_ptr) CUDA_CHECK(hipFreeAsync(buffers.nam_temp_ptr, stream));
        CUDA_CHECK(hipMallocAsync(&buffers.nam_temp_ptr, total_nams * 2 * sizeof(Nam), stream));
        buffers.nam_temp_capacity = total_nams * 2;
    }

    // Step A: Gather NAMs into the temporary buffer in sorted order.
    gather_sorted_nams_kernel<<<blocks_per_grid, threads_per_block, 0, stream>>>(
            todo_cnt, nams_per_task, global_todo_ids, buffers.seg_offsets_ptr, sorted_indices, buffers.nam_temp_ptr
    );

    // Step B: Scatter the sorted NAMs from the temp buffer back to the original vectors.
    scatter_sorted_nams_kernel<<<blocks_per_grid, threads_per_block, 0, stream>>>(
            todo_cnt, buffers.nam_temp_ptr, global_todo_ids, buffers.seg_offsets_ptr, nams_per_task
    );
    CUDA_CHECK(hipStreamSynchronize(stream));
    *gpu_cost4 += GetTime() - t0;
}


my_pair<int*, int*> sort_nams_by_score_with_cub(
        int todo_cnt,
        my_vector<Nam>* nams_per_task,
        int* global_todo_ids,
        hipStream_t stream,
        SegSortGpuResources& buffers,
        double *gpu_cost1,
        double *gpu_cost2,
        double *gpu_cost3,
        double *gpu_cost4)
{
    my_pair<int*, int*> res({nullptr, nullptr});
    if (todo_cnt == 0) return res;

    int threads_per_block = 256;
    int blocks_per_grid = (todo_cnt + threads_per_block - 1) / threads_per_block;

    // --- Part 1: Prepare Segment Information with Re-allocation Logic ---
    double t0 = GetTime();

    size_t required_task_sizes_bytes = todo_cnt * sizeof(int);
    if (buffers.task_sizes_bytes < required_task_sizes_bytes) {
        printf("Allocating NAM task sizes buffer: %zu (%zu) bytes\n", required_task_sizes_bytes, buffers.task_sizes_bytes);
        if (buffers.task_sizes_ptr) CUDA_CHECK(hipFreeAsync(buffers.task_sizes_ptr, stream));
        CUDA_CHECK(hipMallocAsync(&buffers.task_sizes_ptr, required_task_sizes_bytes * 2, stream));
        buffers.task_sizes_bytes = required_task_sizes_bytes * 2;
    }
    get_nam_sizes_kernel<<<blocks_per_grid, threads_per_block, 0, stream>>>(todo_cnt, nams_per_task, global_todo_ids, buffers.task_sizes_ptr);

    size_t required_seg_offsets_bytes = (todo_cnt + 1) * sizeof(int);
    if (buffers.seg_offsets_bytes < required_seg_offsets_bytes) {
        printf("Allocating NAM segment offsets buffer: %zu (%zu) bytes\n", required_seg_offsets_bytes, buffers.seg_offsets_bytes);
        if (buffers.seg_offsets_ptr) CUDA_CHECK(hipFreeAsync(buffers.seg_offsets_ptr, stream));
        CUDA_CHECK(hipMallocAsync(&buffers.seg_offsets_ptr, required_seg_offsets_bytes * 2, stream));
        buffers.seg_offsets_bytes = required_seg_offsets_bytes * 2;
    }

    void* d_scan_temp_storage = nullptr;
    size_t scan_temp_storage_bytes = 0;
    hipcub::DeviceScan::ExclusiveSum(d_scan_temp_storage, scan_temp_storage_bytes, buffers.task_sizes_ptr, buffers.seg_offsets_ptr, todo_cnt + 1, stream);

    if (buffers.scan_temp_bytes < scan_temp_storage_bytes) {
        printf("Allocating NAM scan temp storage: %zu (%zu) bytes\n", scan_temp_storage_bytes, buffers.scan_temp_bytes);
        if (buffers.scan_temp_ptr) CUDA_CHECK(hipFreeAsync(buffers.scan_temp_ptr, stream));
        CUDA_CHECK(hipMallocAsync(&buffers.scan_temp_ptr, scan_temp_storage_bytes * 2, stream));
        buffers.scan_temp_bytes = scan_temp_storage_bytes * 2;
    }
    hipcub::DeviceScan::ExclusiveSum(buffers.scan_temp_ptr, scan_temp_storage_bytes, buffers.task_sizes_ptr, buffers.seg_offsets_ptr, todo_cnt + 1, stream);

    int total_nams;
    CUDA_CHECK(hipMemcpyAsync(&total_nams, buffers.seg_offsets_ptr + todo_cnt, sizeof(int), hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(hipStreamSynchronize(stream));
    *gpu_cost1 += GetTime() - t0;

    if (total_nams == 0) {
        res.first = buffers.seg_offsets_ptr;
        return res;
    }

    // --- Part 2: Marshal Data into Pre-allocated Buffers ---
    t0 = GetTime();

    if (buffers.key_value_capacity < total_nams) {
        printf("Allocating NAM key-value buffers: %d (%zu) bytes\n", total_nams, buffers.key_value_capacity);
        if (buffers.key_ptr)       CUDA_CHECK(hipFreeAsync(buffers.key_ptr, stream));
        if (buffers.value_ptr)     CUDA_CHECK(hipFreeAsync(buffers.value_ptr, stream));
        if (buffers.key_alt_ptr)   CUDA_CHECK(hipFreeAsync(buffers.key_alt_ptr, stream));
        if (buffers.value_alt_ptr) CUDA_CHECK(hipFreeAsync(buffers.value_alt_ptr, stream));
        size_t new_capacity = total_nams * 2;
        size_t new_bytes = new_capacity * sizeof(int);
        CUDA_CHECK(hipMallocAsync(&buffers.key_ptr,       new_bytes, stream));
        CUDA_CHECK(hipMallocAsync(&buffers.value_ptr,     new_bytes, stream));
        CUDA_CHECK(hipMallocAsync(&buffers.key_alt_ptr,   new_bytes, stream));
        CUDA_CHECK(hipMallocAsync(&buffers.value_alt_ptr, new_bytes, stream));
        buffers.key_value_capacity = new_capacity;
    }

    marshal_data_for_nam_sort_kernel<<<blocks_per_grid, threads_per_block, 0, stream>>>(todo_cnt, nams_per_task, global_todo_ids, buffers.seg_offsets_ptr, buffers.key_ptr, buffers.value_ptr);
    *gpu_cost2 += GetTime() - t0;

    // --- Part 3: Sort with CUB ---
    t0 = GetTime();
    hipcub::DoubleBuffer<int> d_keys_buffer(buffers.key_ptr, buffers.key_alt_ptr);
    hipcub::DoubleBuffer<int> d_values_buffer(buffers.value_ptr, buffers.value_alt_ptr);

    void* d_sort_temp_storage = nullptr;
    size_t sort_temp_storage_bytes = 0;
    hipcub::DeviceSegmentedSort::SortPairs(d_sort_temp_storage, sort_temp_storage_bytes, d_keys_buffer, d_values_buffer, total_nams, todo_cnt, buffers.seg_offsets_ptr, buffers.seg_offsets_ptr + 1, stream);

    if (buffers.sort_temp_bytes < sort_temp_storage_bytes) {
        printf("Allocating NAM sort temp storage: %zu (%zu) bytes\n", sort_temp_storage_bytes, buffers.sort_temp_bytes);
        if (buffers.sort_temp_ptr) CUDA_CHECK(hipFreeAsync(buffers.sort_temp_ptr, stream));
        CUDA_CHECK(hipMallocAsync(&buffers.sort_temp_ptr, sort_temp_storage_bytes * 2, stream));
        buffers.sort_temp_bytes = sort_temp_storage_bytes * 2;
    }
    hipcub::DeviceSegmentedSort::SortPairs(buffers.sort_temp_ptr, sort_temp_storage_bytes, d_keys_buffer, d_values_buffer, total_nams, todo_cnt, buffers.seg_offsets_ptr, buffers.seg_offsets_ptr + 1, stream);
    *gpu_cost3 += GetTime() - t0;

    // The function does not free any pre-allocated memory.
    res.first = buffers.seg_offsets_ptr;
    res.second = d_values_buffer.Current(); // This is the pointer to the sorted indices.
    return res;
}


__global__ void gpu_sort_hits(
        int num_tasks,
        my_vector<my_pair<int, Hit>>* hits_per_ref0s,
        my_vector<my_pair<int, Hit>>* hits_per_ref1s,
        int* global_todo_ids
) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < num_tasks) {
        int real_id = global_todo_ids[id];
#ifdef GPU_ACC_TAG
        sort_hits_single(hits_per_ref0s[real_id]);
        sort_hits_single(hits_per_ref1s[real_id]);
#else
        sort_hits_by_refid(hits_per_ref0s[real_id]);
        sort_hits_by_refid(hits_per_ref1s[real_id]);
#endif
        //check_hits(hits_per_ref0s[real_id]);
        //check_hits(hits_per_ref1s[real_id]);
    }
}

__global__ void gpu_rescue_sort_hits(
        int num_tasks,
        my_vector<my_pair<int, Hit>>* hits_per_ref0s,
        my_vector<my_pair<int, Hit>>* hits_per_ref1s,
        int* global_todo_ids
) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < num_tasks) {
        int real_id = global_todo_ids[id];
#ifdef GPU_ACC_TAG
        sort_hits_single(hits_per_ref0s[real_id]);
        sort_hits_single(hits_per_ref1s[real_id]);
#else
        sort_hits_by_refid(hits_per_ref0s[real_id]);
        sort_hits_by_refid(hits_per_ref1s[real_id]);
#endif
        //check_hits(hits_per_ref0s[real_id]);
        //check_hits(hits_per_ref1s[real_id]);
    }
}

__global__ void gpu_merge_hits_get_nams_seg(
        int num_tasks,
        IndexParameters *index_para,
        uint64_t *global_nams_info,
        my_vector<my_pair<int, Hit>>* hits_per_ref0s,
        my_vector<my_pair<int, Hit>>* hits_per_ref1s,
        const int* seg_offsets0, const int* sorted_indices0,
        const int* seg_offsets1, const int* sorted_indices1,
        my_vector<Nam> *global_nams,
        int* global_todo_ids
) {
    int global_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (global_id < num_tasks) {
        int real_id = global_todo_ids[global_id];
        global_nams[real_id].init(8);

        // Process hits for read 1 (forward strand)
        const my_vector<my_pair<int, Hit>>& original_hits0 = hits_per_ref0s[real_id];
        int task_start0 = seg_offsets0[global_id];
        int task_end0   = seg_offsets0[global_id + 1];
        merge_hits_seg(original_hits0, sorted_indices0, task_start0, task_end0, index_para->syncmer.k, 0, global_nams[real_id]);

        // Process hits for read 2 (reverse strand)
        const my_vector<my_pair<int, Hit>>& original_hits1 = hits_per_ref1s[real_id];
        int task_start1 = seg_offsets1[global_id];
        int task_end1   = seg_offsets1[global_id + 1];
        merge_hits_seg(original_hits1, sorted_indices1, task_start1, task_end1, index_para->syncmer.k, 1, global_nams[real_id]);

        hits_per_ref0s[real_id].release();
        hits_per_ref1s[real_id].release();
    }
}

__global__ void gpu_merge_hits_get_nams(
        int num_tasks,
        IndexParameters *index_para,
        uint64_t *global_nams_info,
        my_vector<my_pair<int, Hit>>* hits_per_ref0s,
        my_vector<my_pair<int, Hit>>* hits_per_ref1s,
        my_vector<Nam> *global_nams,
        int* global_todo_ids
) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < num_tasks) {
        int real_id = global_todo_ids[id];
        global_nams[real_id].init(8);
        merge_hits(hits_per_ref0s[real_id], index_para->syncmer.k, 0, global_nams[real_id]);
        merge_hits(hits_per_ref1s[real_id], index_para->syncmer.k, 1, global_nams[real_id]);
        hits_per_ref0s[real_id].release();
        hits_per_ref1s[real_id].release();
    }
}

__global__ void gpu_rescue_merge_hits_get_nams_seg(
        int num_tasks,
        IndexParameters *index_para,
        uint64_t *global_nams_info,
        my_vector<my_pair<int, Hit>>* hits_per_ref0s,
        my_vector<my_pair<int, Hit>>* hits_per_ref1s,
        const int* seg_offsets0, const int* sorted_indices0,
        const int* seg_offsets1, const int* sorted_indices1,
        my_vector<Nam> *global_nams,
        int* global_todo_ids
) {
    int global_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (global_id < num_tasks) {
        int real_id = global_todo_ids[global_id];
        global_nams[real_id].init(8);

        // Process hits for read 1 (forward strand)
        const my_vector<my_pair<int, Hit>>& original_hits0 = hits_per_ref0s[real_id];
        int task_start0 = seg_offsets0[global_id];
        int task_end0   = seg_offsets0[global_id + 1];
        salign_merge_hits_seg(original_hits0, sorted_indices0, task_start0, task_end0, index_para->syncmer.k, 0, global_nams[real_id]);

        // Process hits for read 2 (reverse strand)
        const my_vector<my_pair<int, Hit>>& original_hits1 = hits_per_ref1s[real_id];
        int task_start1 = seg_offsets1[global_id];
        int task_end1   = seg_offsets1[global_id + 1];
        salign_merge_hits_seg(original_hits1, sorted_indices1, task_start1, task_end1, index_para->syncmer.k, 1, global_nams[real_id]);

        hits_per_ref0s[real_id].release();
        hits_per_ref1s[real_id].release();
    }
}

__global__ void gpu_rescue_merge_hits_get_nams(
        int num_tasks,
        IndexParameters *index_para,
        uint64_t *global_nams_info,
        my_vector<my_pair<int, Hit>>* hits_per_ref0s,
        my_vector<my_pair<int, Hit>>* hits_per_ref1s,
        my_vector<Nam> *global_nams,
        int* global_todo_ids
)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < num_tasks){
        int real_id = global_todo_ids[id];
        global_nams[real_id].init(8);
        salign_merge_hits(hits_per_ref0s[real_id], index_para->syncmer.k, 0, global_nams[real_id]);
        salign_merge_hits(hits_per_ref1s[real_id], index_para->syncmer.k, 1, global_nams[real_id]);
        hits_per_ref0s[real_id].release();
        hits_per_ref1s[real_id].release();
    }
}


__global__ void gpu_sort_nams(
        int num_tasks,
        my_vector<Nam> *global_nams,
        MappingParameters *mapping_parameters,
        int is_se
) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < num_tasks) {
        int max_tries = mapping_parameters->max_tries;
        if (is_se) {
#ifdef GPU_ACC_TAG
            sort_nams_single_check(global_nams[id]);
#else
            sort_nams_by_score(global_nams[id], max_tries);
            global_nams[id].length = my_min(global_nams[id].length, max_tries);
#endif
        } else {
#ifdef GPU_ACC_TAG
            sort_nams_single_check(global_nams[id]);
#else
//            sort_nams_by_score(global_nams[id], max_tries * 2);
//            global_nams[id].length = my_min(global_nams[id].length, max_tries * 2);
            sort_nams_by_score(global_nams[id], 1e9);
#endif
        }
        gpu_shuffle_top_nams(global_nams[id]);
    }
}
